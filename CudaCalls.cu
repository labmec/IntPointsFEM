#include "CudaCalls.h"
#include "pzreal.h"
#include "pzvec.h"

#ifdef USING_CUDA
#include "SpectralDecompKernels.h"
#include "StressStrainKernels.h"
#include "SigmaProjectionKernels.h"
#include "MatMulKernels.h"
#endif

#define NT 512

CudaCalls::CudaCalls() {
	hipsparse.h = false;
	hipblas.h = false;
}

CudaCalls::~CudaCalls() {
	if(cublas_h == true) {
		hipblasDestroy(handle_cublas);
	}
	if(cusparse_h == true) {
		hipsparseDestroy(handle_cusparse);			
	}
}

void CudaCalls::Multiply(bool trans, int *m, int *n, int *k, REAL *A, int *strideA, 
	REAL *B, int *strideB,  REAL *C, int *strideC, REAL alpha, int nmatrices) {

	MatrixMultiplicationKernel<<<nmatrices,1>>> (trans, m, n, k, A, strideA, B, strideB, C, strideC, alpha, nmatrices);
	hipDeviceSynchronize();
	if (hipGetLastError() != hipSuccess) {
		throw std::runtime_error("failed to perform MatrixMultiplicationKernel kernel");      
	}

}

void CudaCalls::GatherOperation(int n, REAL *x, REAL *y, int *id) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseStatus_t result = hipsparseDgthr(handle_cusparse, n, x, y, id, HIPSPARSE_INDEX_BASE_ZERO);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDgthr");      
	}	
}

void CudaCalls::ScatterOperation(int n, REAL *x, REAL *y, int *id) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseStatus_t result = hipsparseDsctr(handle_cusparse, n, x, id, y, HIPSPARSE_INDEX_BASE_ZERO);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDsctr");      
	}	
}

void CudaCalls::DaxpyOperation(int n, REAL alpha, REAL *x, REAL *y) {
	if(cublas_h == false) {
		hipblas.h = true;
		hipblasStatus_t result = hipblasCreate(&handle_cublas);
		if (result != HIPBLAS_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuBLAS");      
		}			
	}
	hipblasStatus_t result = hipblasDaxpy(handle_cublas, n, &alpha, x, 1., y, 1.);
	if (result != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipblasDaxpy");      
	}	
}







void CudaCalls::ElasticStrain(REAL *delta_strain, REAL *elastic_strain, int64_t n) {
	hipMemcpy(elastic_strain, &delta_strain[0], n * sizeof(REAL), hipMemcpyDeviceToDevice);
	REAL *plastic_strain;
	hipMalloc(&plastic_strain, n * sizeof(REAL));
	hipMemset(plastic_strain, 0, n * sizeof(REAL));

	REAL a = -1.;
	if(cublas_h == false) {
		hipblas.h = true;
		hipblasStatus_t result = hipblasCreate(&handle_cublas);
		if (result != HIPBLAS_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuBLAS");      
		}			
	}
	hipblasStatus_t result = hipblasDaxpy(handle_cublas, n, &a, &plastic_strain[0], 1, &elastic_strain[0], 1);
	if (result != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipblasDaxpy");      
	}
	hipFree(plastic_strain);
}

void CudaCalls::ComputeStress(REAL *elastic_strain, REAL *sigma, int64_t n, REAL mu, REAL lambda) {
	int numBlocks = (n + NT - 1) / NT;

	ComputeStressKernel<<<numBlocks, NT>>>(elastic_strain, sigma, n, mu, lambda);
	hipDeviceSynchronize();
	if (hipGetLastError() != hipSuccess) {
		throw std::runtime_error("failed to perform ComputeStressKernel");      
	}
}

void CudaCalls::SpectralDecomposition(REAL *sigma_trial, REAL *eigenvalues, REAL *eigenvectors, int64_t n) {
	int numBlocks = (n + NT - 1) / NT;

	SpectralDecompositionKernel<<<numBlocks, NT>>>(sigma_trial, eigenvalues, eigenvectors, n);
	hipDeviceSynchronize();
		// if (hipGetLastError() != hipSuccess) {
		// 	throw std::runtime_error("failed to perform SpectralDecompositionKernel");      
		// }
}

void CudaCalls::ProjectSigma(REAL *eigenvalues, REAL *sigma_projected, int64_t n, REAL mc_phi, REAL mc_psi, REAL mc_cohesion, REAL K, REAL G) {
	REAL *m_type;
	hipMalloc((void**) &m_type, n * sizeof(REAL));

	REAL *alpha;
	hipMalloc((void**) &alpha, n * sizeof(REAL));

	int numBlocks = (n + NT - 1) / NT;

	ProjectSigmaKernel<<<numBlocks, NT>>>(eigenvalues, sigma_projected, m_type, alpha, n, mc_phi, mc_psi, mc_cohesion, K, G);	
	hipDeviceSynchronize();
	// if (hipGetLastError() != hipSuccess) {
	// 	throw std::runtime_error("failed to perform ProjectSigmaKernel");      
	// }

}