#include "hip/hip_runtime.h"
#include "CudaCalls.h"
#include "pzreal.h"
#include "pzvec.h"

__global__ void matrixMultiplication (bool trans, int *m, int *n, int *k, REAL *A, int *strideA, REAL *B, int *strideB, REAL *C, int *strideC, REAL alpha, int nmatrices) {

	int imatrix = blockIdx.x;

	if (imatrix < nmatrices) {
		int m_i = m[imatrix];
		int n_i = n[imatrix];
		int k_i = k[imatrix];
		int strideA_i = strideA[imatrix]; 
		int strideB_i = strideB[imatrix]; 
		int strideC_i = strideC[imatrix]; 

		int aux1, aux2;

		if (trans == false) {
			aux1 = m_i;
			aux2 = 1;

		} else {
			aux1 = 1;
			aux2 = m_i;
		}


		for (int i = 0; i < m_i; i++) {
			for (int j = 0; j < n_i; j++) {
				C[j * m_i + i] = 0;
				for (int l = 0; l < k_i; l++) {
					C[j * m_i + i + strideC_i] += alpha * A[l * aux1 + i * aux2 + strideA_i] * B[j * k_i + l + strideB_i];
				}
			}
		}
	}
}

	CudaCalls::CudaCalls() {

	}

	CudaCalls::~CudaCalls() {
		if(handle_cublas) {
			hipblasDestroy(handle_cublas);
		}
		if(handle_cusparse) {
			hipsparseDestroy(handle_cusparse);			
		}
	}

	void CudaCalls::Multiply(bool trans, TPZVecGPU<int> m, TPZVecGPU<int> n, TPZVecGPU<int> k, TPZVecGPU<REAL> A, TPZVecGPU<int> strideA, 
		TPZVecGPU<REAL> B, TPZVecGPU<int> strideB,  TPZVecGPU<REAL> C, TPZVecGPU<int> strideC, REAL alpha, int nmatrices) {

		matrixMultiplication<<<nmatrices,1>>> (trans, m.getData(), n.getData(), k.getData(), A.getData(), strideA.getData(), 
		B.getData(), strideB.getData(), C.getData(), strideC.getData(), alpha, nmatrices);

	}

	void CudaCalls::GatherOperation(int n, TPZVecGPU<REAL> &x, TPZVecGPU<REAL> &y, TPZVecGPU<int> &id) {
		if(!handle_cusparse) {
			hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
			if (result != HIPSPARSE_STATUS_SUCCESS) {
            throw std::runtime_error("failed to initialize cuSparse");      
       		}			
		}
		hipsparseStatus_t result = hipsparseDgthr(handle_cusparse, n, x.getData(), &y.getData()[0], &id.getData()[0], HIPSPARSE_INDEX_BASE_ZERO);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to perform GatherOperation");      
		}	

	}