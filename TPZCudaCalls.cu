#include "TPZCudaCalls.h"
#include "pzreal.h"
#include "pzvec.h"

// #include "MatMulKernels.h"
#include "ComputeSigmaKernel.h"
#include "MatrixAssembleKernel.h"


#define NT          256

// #if __CUDA_ARCH__ >= 200
//     #define MY_KERNEL_MAX_THREADS  (2 * NT)
//     #define MY_KERNEL_MIN_BLOCKS   3
// #else
//     #define MY_KERNEL_MAX_THREADS  NT
//     #define MY_KERNEL_MIN_BLOCKS   2
// #endif

TPZCudaCalls::TPZCudaCalls() {
	hipsparse.h = false;
	hipblas.h = false;
	heap_q = false;
}

TPZCudaCalls::~TPZCudaCalls() {
	if(cublas_h == true) {
		hipblasDestroy(handle_cublas);
	}
	if(cusparse_h == true) {
		hipsparseDestroy(handle_cusparse);			
	}
}

void TPZCudaCalls::Multiply(bool trans, int *m, int *n, int *k, REAL *A, int *strideA, 
	REAL *B, int *strideB,  REAL *C, int *strideC, REAL alpha, int nmatrices) {
	int numBlocks = (nmatrices + NT - 1) / NT;
	MatrixMultiplicationKernel<<<numBlocks,NT>>> (trans, m, n, k, A, strideA, B, strideB, C, strideC, alpha, nmatrices);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::string error_string = hipGetErrorString(error);
		std::string error_message = "failed to perform MatrixMultiplicationKernel: " + error_string;
		throw std::runtime_error(error_message);      
	}

}

void TPZCudaCalls::GatherOperation(int n, REAL *x, REAL *y, int *id) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseStatus_t result = hipsparseDgthr(handle_cusparse, n, x, y, id, HIPSPARSE_INDEX_BASE_ZERO);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDgthr");      
	}	
}

void TPZCudaCalls::ScatterOperation(int n, REAL *x, REAL *y, int *id) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseStatus_t result = hipsparseDsctr(handle_cusparse, n, x, id, y, HIPSPARSE_INDEX_BASE_ZERO);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDsctr");      
	}	
}

void TPZCudaCalls::DaxpyOperation(int n, double alpha, double *x, double *y) {
	if(cublas_h == false) {
		hipblas.h = true;
		hipblasStatus_t result = hipblasCreate(&handle_cublas);
		if (result != HIPBLAS_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuBLAS");      
		}			
	}
	hipblasStatus_t result = hipblasDaxpy(handle_cublas, n, &alpha, x, 1., y, 1.);
	if (result != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipblasDaxpy");      
	}	
}

void TPZCudaCalls::SpMV(int opt, int sym, int m, int k, int nnz, REAL alpha, REAL *csrVal, int *csrRowPtr, int *csrColInd, REAL *B, REAL *C) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    if(sym == 0) {
	   hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);        
    } 
    else {
        hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC); 
    }
    hipsparseOperation_t op;
    if(opt == 0) { 
        op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    } else {
        op = HIPSPARSE_OPERATION_TRANSPOSE;
    }

	REAL beta = 0.;
	hipsparseStatus_t result = hipsparseDcsrmv(handle_cusparse, op, m, k, nnz, &alpha, descr, csrVal, csrRowPtr, csrColInd, B, &beta, C);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDcsrmv");      
	}	
}

void TPZCudaCalls::SpMSpM(int opt, int sym, int m, int n, int k, int nnzA, REAL *csrValA, int *csrRowPtrA, int *csrColIndA, 
	int nnzB, REAL *csrValB, int *csrRowPtrB, int *csrColIndB, 
	int nnzC, REAL *csrValC, int *csrRowPtrC) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}

    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    if(sym == 0) {
       hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);        
    } 
    else {
        hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC); 
    }

    hipsparseOperation_t op;
    if(opt == 0) { 
        op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    } else {
        op = HIPSPARSE_OPERATION_TRANSPOSE;
    }

	int *csrColIndC;
	hipMalloc((void**)&csrColIndC, sizeof(int)*nnzC);

	hipsparseStatus_t result = hipsparseDcsrgemm(handle_cusparse, op, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, 
		descr, nnzA, csrValA, csrRowPtrA, csrColIndA, 
		descr, nnzB, csrValB, csrRowPtrB, csrColIndB,
		descr, csrValC, csrRowPtrC, csrColIndC);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDcsrgemm");      
	}	
}

void TPZCudaCalls::ComputeSigma(bool update_mem, int npts, REAL *glob_delta_strain, REAL *glob_sigma, REAL lambda, REAL mu, REAL mc_phi, REAL mc_psi, REAL mc_cohesion, REAL *dPlasticStrain,  
	REAL *dMType, REAL *dAlpha, REAL *dSigma, REAL *dStrain, REAL *weight) {
	
	int numBlocks = (npts + 256 - 1) / 256;
	ComputeSigmaKernel<<<numBlocks,256>>> (update_mem, npts, glob_delta_strain, glob_sigma, lambda, mu, mc_phi, mc_psi, mc_cohesion, dPlasticStrain, dMType, dAlpha, dSigma, dStrain, weight);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::string error_string = hipGetErrorString(error);
		std::string error_message = "failed to perform ComputeSigmaKernel: " + error_string;
		throw std::runtime_error(error_message);      
	}
}

void TPZCudaCalls::MatrixAssemble(int nnz, REAL *Kg, int first_el, int last_el, int64_t *el_color_index, REAL *weight, int *dof_indexes,
	REAL *storage, int *rowsizes, int *colsizes, int *rowfirstindex, int *colfirstindex, int *matrixposition, int *ia_to_sequence, int *ja_to_sequence) {
	int nel = last_el - first_el;
	int numBlocks = (nel + NT - 1) / NT;

	MatrixAssembleKernel<<<numBlocks,NT>>> (nel, nnz, Kg, first_el, el_color_index, weight, dof_indexes, storage, rowsizes, colsizes, rowfirstindex, colfirstindex, matrixposition, 
		ia_to_sequence, ja_to_sequence);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::string error_string = hipGetErrorString(error);
		std::string error_message = "failed to perform MatrixAssembleKernel: " + error_string;
		throw std::runtime_error(error_message);      
	}
}

void TPZCudaCalls::MatrixAssemble(REAL *K, int nnz, REAL *Kg, int first_el, int last_el, int64_t *el_color_index, int *dof_indexes,
	int *colsizes, int *colfirstindex, int *ia_to_sequence, int *ja_to_sequence) {
	int nel = last_el - first_el;
	int numBlocks = (nel + NT - 1) / NT;

	MatrixAssembleKernel<<<nel,1>>> (K, nel, nnz, Kg, el_color_index, dof_indexes, colsizes, colfirstindex, 
    ia_to_sequence, ja_to_sequence);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::string error_string = hipGetErrorString(error);
		std::string error_message = "failed to perform MatrixAssembleKernel: " + error_string;
		throw std::runtime_error(error_message);      
	}
}

void TPZCudaCalls::SolveCG(int n, int nnzA, REAL *csrValA, int *csrRowPtrA, int *csrColIndA, REAL *r, REAL *x) {
    if(cusparse_h == false) {
        hipsparse.h = true;
        hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
        if (result != HIPSPARSE_STATUS_SUCCESS) {
            throw std::runtime_error("failed to initialize cuSparse");      
        }           
    }

    if(cublas_h == false) {
        hipblas.h = true;
        hipblasStatus_t result = hipblasCreate(&handle_cublas);
        if (result != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("failed to initialize cuBLAS");      
        }           
    }

    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
    hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    REAL alpha = 1.0;
    REAL alpham1 = -1.0;
    REAL beta = 0.0;
    REAL r0 = 0.;
    REAL b;
    REAL r1;
    REAL dot;
    REAL a;
    REAL na;

    REAL *d_Ax;
    REAL *d_p;
    hipMalloc((void **)&d_Ax, n*sizeof(REAL));
    hipMalloc((void **)&d_p, n*sizeof(REAL));

    hipsparseDcsrmv(handle_cusparse,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnzA, &alpha, descr, csrValA, csrRowPtrA, csrColIndA, x, &beta, d_Ax);
    hipblasDaxpy(handle_cublas, n, &alpham1, d_Ax, 1, r, 1);


    hipblasDdot(handle_cublas, n, r, 1, r, 1, &r1);

    const REAL tol = 1.e-5;
    const int max_iter = 10000;
    int k;

    k = 1;

    while (r1 > tol*tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
            hipblasDscal(handle_cublas, n, &b, d_p, 1);
            hipblasDaxpy(handle_cublas, n, &alpha, r, 1, d_p, 1);
        }
        else
        {
            hipblasDcopy(handle_cublas, n, r, 1, d_p, 1);
        }
        hipsparseDcsrmv(handle_cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnzA, &alpha, descr, csrValA, csrRowPtrA, csrColIndA, d_p, &beta, d_Ax);
        hipblasDdot(handle_cublas, n, d_p, 1, d_Ax, 1, &dot);
        a = r1 / dot;

        hipblasDaxpy(handle_cublas, n, &a, d_p, 1, x, 1);
        na = -a;
        hipblasDaxpy(handle_cublas, n, &na, d_Ax, 1, r, 1);

        r0 = r1;
        hipblasDdot(handle_cublas, n, r, 1, r, 1, &r1);
        hipDeviceSynchronize();
        k++;
    }
    hipFree(d_p);
    hipFree(d_Ax);
}

// void assemble_poisson_matrix_coo(std::vector<float>& vals, std::vector<int>& row, std::vector<int>& col,
//                      std::vector<float>& rhs, int Nrows, int Ncols) {

//         //nnz: 5 entries per row (node) for nodes in the interior
//     // 1 entry per row (node) for nodes on the boundary, since we set them explicitly to 1.
//     int nnz = 5*Nrows*Ncols - (2*(Ncols-1) + 2*(Nrows-1))*4;
//     vals.resize(nnz);
//     row.resize(nnz);
//     col.resize(nnz);
//     rhs.resize(Nrows*Ncols);

//     int counter = 0;
//     for(int i = 0; i < Nrows; ++i) {
//         for (int j = 0; j < Ncols; ++j) {
//             int idx = j + Ncols*i;
//             if (i == 0 || j == 0 || j == Ncols-1 || i == Nrows-1) {
//                 vals[counter] = 1.;
//                 row[counter] = idx;
//                 col[counter] = idx;
//                 counter++;
//                 rhs[idx] = 1.;
// //                if (i == 0) {
// //                    rhs[idx] = 3.;
// //                }
//             } else { // -laplace stencil
//                 // above
//                 vals[counter] = -1.;
//                 row[counter] = idx;
//                 col[counter] = idx-Ncols;
//                 counter++;
//                 // left
//                 vals[counter] = -1.;
//                 row[counter] = idx;
//                 col[counter] = idx-1;
//                 counter++;
//                 // center
//                 vals[counter] = 4.;
//                 row[counter] = idx;
//                 col[counter] = idx;
//                 counter++;
//                 // right
//                 vals[counter] = -1.;
//                 row[counter] = idx;
//                 col[counter] = idx+1;
//                 counter++;
//                 // below
//                 vals[counter] = -1.;
//                 row[counter] = idx;
//                 col[counter] = idx+Ncols;
//                 counter++;

//                 rhs[idx] = 0;
//             }
//         }
//     }
// }

// void TPZCudaCalls::Teste() {
// 	    // --- create library handles:
//     hipsolverSpHandle_t cusolver_handle;
//     hipsolverStatus_t cusolver_status;
//     cusolver_status = hipsolverSpCreate(&cusolver_handle);
//     std::cout << "status create cusolver handle: " << cusolver_status << std::endl;

//     hipsparseHandle_t cusparse_handle;
//     hipsparseStatus_t cusparse_status;
//     cusparse_status = hipsparseCreate(&cusparse_handle);
//     std::cout << "status create cusparse handle: " << cusparse_status << std::endl;

//     // --- prepare matrix:
//     int Nrows = 4;
//     int Ncols = 4;
//     std::vector<float> csrVal;
//     std::vector<int> cooRow;
//     std::vector<int> csrColInd;
//     std::vector<float> b;

//     assemble_poisson_matrix_coo(csrVal, cooRow, csrColInd, b, Nrows, Ncols);

//     int nnz = csrVal.size();
//     int m = Nrows * Ncols;
//     std::vector<int> csrRowPtr(m+1);

//     // --- prepare solving and copy to GPU:
//     std::vector<float> x(m);
//     float tol = 1e-5;
//     int reorder = 0;
//     int singularity = 0;

//     float *db, *dcsrVal, *dx;
//     int *dcsrColInd, *dcsrRowPtr, *dcooRow;
//     hipMalloc((void**)&db, m*sizeof(float));
//     hipMalloc((void**)&dx, m*sizeof(float));
//     hipMalloc((void**)&dcsrVal, nnz*sizeof(float));
//     hipMalloc((void**)&dcsrColInd, nnz*sizeof(int));
//     hipMalloc((void**)&dcsrRowPtr, (m+1)*sizeof(int));
//     hipMalloc((void**)&dcooRow, nnz*sizeof(int));

//     hipMemcpy(db, b.data(), b.size()*sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(dcsrVal, csrVal.data(), csrVal.size()*sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(dcsrColInd, csrColInd.data(), csrColInd.size()*sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(dcooRow, cooRow.data(), cooRow.size()*sizeof(int), hipMemcpyHostToDevice);

//     cusparse_status = hipsparseXcoo2csr(cusparse_handle, dcooRow, nnz, m,
//                                        dcsrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
//     std::cout << "status cusparse coo2csr conversion: " << cusparse_status << std::endl;

//     hipDeviceSynchronize(); // matrix format conversion has to be finished!

//     // --- everything ready for computation:

//     hipsparseMatDescr_t descrA;

//     cusparse_status = hipsparseCreateMatDescr(&descrA);
//     std::cout << "status cusparse createMatDescr: " << cusparse_status << std::endl;

//     // optional: print dense matrix that has been allocated on GPU

//     std::vector<float> A(m*m, 0);
//     float *dA;
//     hipMalloc((void**)&dA, A.size()*sizeof(float));

//     hipsparseScsr2dense(cusparse_handle, m, m, descrA, dcsrVal,
//                        dcsrRowPtr, dcsrColInd, dA, m);

//     hipMemcpy(A.data(), dA, A.size()*sizeof(float), hipMemcpyDeviceToHost);
//     std::cout << "A: \n";
//     for (int i = 0; i < m; ++i) {
//         for (int j = 0; j < m; ++j) {
//             std::cout << A[i*m + j] << " ";
//         }
//         std::cout << std::endl;
//     }

//     hipFree(dA);

//     std::cout << "b: \n";
//     hipMemcpy(b.data(), db, (m)*sizeof(int), hipMemcpyDeviceToHost);
//     for (auto a : b) {
//         std::cout << a << ",";
//     }
//     std::cout << std::endl;


//     // --- solving!!!!

// // // does not work:
// //    cusolver_status = hipsolverSpScsrlsvchol(cusolver_handle, m, nnz, descrA, dcsrVal,
// //                       dcsrRowPtr, dcsrColInd, db, tol, reorder, dx,
// //                       &singularity);

//      cusolver_status = hipsolverSpScsrlsvqr(cusolver_handle, m, nnz, descrA, dcsrVal,
//                         dcsrRowPtr, dcsrColInd, db, tol, reorder, dx,
//                         &singularity);

//     hipDeviceSynchronize();

//     std::cout << "singularity (should be -1): " << singularity << std::endl;

//     std::cout << "status cusolver solving (!): " << cusolver_status << std::endl;

//     hipMemcpy(x.data(), dx, m*sizeof(float), hipMemcpyDeviceToHost);

//     cusparse_status = hipsparseDestroy(cusparse_handle);
//     std::cout << "status destroy cusparse handle: " << cusparse_status << std::endl;

//     cusolver_status = hipsolverSpDestroy(cusolver_handle);
//     std::cout << "status destroy cusolver handle: " << cusolver_status << std::endl;

//     for (auto a : x) {
//         std::cout << a << " ";
//     }
//     std::cout << std::endl;


//     hipFree(db);
//     hipFree(dx);
//     hipFree(dcsrVal);
//     hipFree(dcsrColInd);
//     hipFree(dcsrRowPtr);
//     hipFree(dcooRow);


// }