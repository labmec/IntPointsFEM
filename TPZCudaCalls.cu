#include "TPZCudaCalls.h"
#include "pzreal.h"
#include "pzvec.h"

// #include "MatMulKernels.h"
#include "KernelsComputeSigma.h"
#include "KernelsMatMul.h"
#include "KernelsMatrixAssemble.h"

#define NT  64

TPZCudaCalls::TPZCudaCalls() {
	hipsparse.h = false;
	hipblas.h = false;
}

TPZCudaCalls::~TPZCudaCalls() {
	if(cublas_h == true) {
		hipblasDestroy(handle_cublas);
	}
	if(cusparse_h == true) {
		hipsparseDestroy(handle_cusparse);			
	}
}

TPZCudaCalls &TPZCudaCalls::operator=(const TPZCudaCalls &copy) {
	if(&copy == this){
		return *this;
	}
	handle_cusparse = copy.handle_cusparse;
	hipsparse.h = copy.cusparse_h;
	handle_cublas = copy.handle_cublas;
	hipblas.h = copy.cublas_h;

	return *this;
}

void TPZCudaCalls::Multiply(bool trans, int *m, int *n, int *k, REAL *A, int *strideA, 
	REAL *B, int *strideB,  REAL *C, int *strideC, REAL alpha, int nmatrices) {
	int numBlocks = (nmatrices + NT - 1) / NT;
	MatrixMultiplicationKernel<<<numBlocks,NT>>> (trans, m, n, k, A, strideA, B, strideB, C, strideC, alpha, nmatrices);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::string error_string = hipGetErrorString(error);
		std::string error_message = "failed to perform MatrixMultiplicationKernel: " + error_string;
		throw std::runtime_error(error_message);      
	}

}

void TPZCudaCalls::GatherOperation(int n, REAL *x, REAL *y, int *id) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseStatus_t result = hipsparseDgthr(handle_cusparse, n, x, y, id, HIPSPARSE_INDEX_BASE_ZERO);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDgthr");      
	}	
}

void TPZCudaCalls::ScatterOperation(int n, REAL *x, REAL *y, int *id) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseStatus_t result = hipsparseDsctr(handle_cusparse, n, x, id, y, HIPSPARSE_INDEX_BASE_ZERO);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDsctr");      
	}	
}

void TPZCudaCalls::DaxpyOperation(int n, double alpha, double *x, double *y) {
	if(cublas_h == false) {
		hipblas.h = true;
		hipblasStatus_t result = hipblasCreate(&handle_cublas);
		if (result != HIPBLAS_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuBLAS");      
		}			
	}
	hipblasStatus_t result = hipblasDaxpy(handle_cublas, n, &alpha, x, 1., y, 1.);
	if (result != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipblasDaxpy");      
	}	
}

void TPZCudaCalls::SpMV(int opt, int sym, int m, int k, int nnz, REAL alpha, REAL *csrVal, int *csrRowPtr, int *csrColInd, REAL *B, REAL *C) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    if(sym == 0) {
	   hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);        
    } 
    else {
        hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC); 
    }
    hipsparseOperation_t op;
    if(opt == 0) { 
        op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    } else {
        op = HIPSPARSE_OPERATION_TRANSPOSE;
    }

	REAL beta = 0.;
	hipsparseStatus_t result = hipsparseDcsrmv(handle_cusparse, op, m, k, nnz, &alpha, descr, csrVal, csrRowPtr, csrColInd, B, &beta, C);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDcsrmv");      
	}	
}

void TPZCudaCalls::SpMSpM(int opt, int sym, int m, int n, int k, int nnzA, REAL *csrValA, int *csrRowPtrA, int *csrColIndA, 
	int nnzB, REAL *csrValB, int *csrRowPtrB, int *csrColIndB, 
	int nnzC, REAL *csrValC, int *csrRowPtrC) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}

    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    if(sym == 0) {
       hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);        
    } 
    else {
        hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC); 
    }

    hipsparseOperation_t op;
    if(opt == 0) { 
        op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    } else {
        op = HIPSPARSE_OPERATION_TRANSPOSE;
    }

	int *csrColIndC;
	hipMalloc((void**)&csrColIndC, sizeof(int)*nnzC);

	hipsparseStatus_t result = hipsparseDcsrgemm(handle_cusparse, op, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, 
		descr, nnzA, csrValA, csrRowPtrA, csrColIndA, 
		descr, nnzB, csrValB, csrRowPtrB, csrColIndB,
		descr, csrValC, csrRowPtrC, csrColIndC);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDcsrgemm");      
	}	
}

void TPZCudaCalls::ComputeSigma(bool update_mem, int npts, REAL *glob_delta_strain, REAL *glob_sigma, REAL lambda, REAL mu, REAL mc_phi, REAL mc_psi, REAL mc_cohesion, REAL *dPlasticStrain,  
	REAL *dMType, REAL *dAlpha, REAL *dSigma, REAL *dStrain, REAL *weight) {
	
	int numBlocks = (npts + NT - 1) / NT;
	ComputeSigmaKernel<<<numBlocks,NT>>> (update_mem, npts, glob_delta_strain, glob_sigma, lambda, mu, mc_phi, mc_psi, mc_cohesion, dPlasticStrain, dMType, dAlpha, dSigma, dStrain, weight);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::string error_string = hipGetErrorString(error);
		std::string error_message = "failed to perform ComputeSigmaKernel: " + error_string;
		throw std::runtime_error(error_message);      
	}
}

void TPZCudaCalls::ComputeSigmaDep(bool update_mem, int npts, REAL *glob_delta_strain, REAL *glob_sigma, REAL *glob_dep, REAL lambda, REAL mu, REAL mc_phi, REAL mc_psi, REAL mc_cohesion, REAL *dPlasticStrain,
	REAL *dMType, REAL *dAlpha, REAL *dSigma, REAL *dStrain, REAL *weight) {

	int numBlocks = (npts + NT - 1) / NT;
	ComputeSigmaDepKernel<<<numBlocks,NT>>> (update_mem, npts, glob_delta_strain, glob_sigma, glob_dep, lambda, mu, mc_phi, mc_psi, mc_cohesion, dPlasticStrain, dMType, dAlpha, dSigma, dStrain, weight);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::string error_string = hipGetErrorString(error);
		std::string error_message = "failed to perform ComputeSigmaDepKernel: " + error_string;
		throw std::runtime_error(error_message);
	}
}

void TPZCudaCalls::MatrixAssemble(REAL *Kc, REAL *dep, int nel, int64_t *el_color_index, REAL *weight, int *dof_indexes,
	REAL *storage, int *rowsizes, int *colsizes, int *rowfirstindex, int *colfirstindex, int *matrixposition) {
	int numBlocks = (nel + NT_sm - 1) / NT_sm;

	MatrixAssembleKernel<<<numBlocks,NT_sm>>>(nel, Kc, dep, el_color_index, weight, dof_indexes, storage, rowsizes, colsizes, rowfirstindex, colfirstindex, matrixposition);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::string error_string = hipGetErrorString(error);
		std::string error_message = "failed to perform MatrixAssembleKernel: " + error_string;
		throw std::runtime_error(error_message);      
	}
}

void TPZCudaCalls::DeToDevice(REAL lambda, REAL mu) {
		REAL De_host[] = {lambda + 2.0*mu, 0, lambda, 0, mu, 0, lambda, 0, lambda + 2.0*mu};
		hipMemcpyToSymbol(HIP_SYMBOL(De), &De_host, 9 * sizeof(REAL));
	}


void TPZCudaCalls::SolveCG(int n, int nnzA, REAL *csrValA, int *csrRowPtrA, int *csrColIndA, REAL *r, REAL *x) {
    if(cusparse_h == false) {
        hipsparse.h = true;
        hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
        if (result != HIPSPARSE_STATUS_SUCCESS) {
            throw std::runtime_error("failed to initialize cuSparse");      
        }           
    }

    if(cublas_h == false) {
        hipblas.h = true;
        hipblasStatus_t result = hipblasCreate(&handle_cublas);
        if (result != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("failed to initialize cuBLAS");      
        }           
    }

    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
    hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    REAL alpha = 1.0;
    REAL alpham1 = -1.0;
    REAL beta = 0.0;
    REAL r0 = 0.;
    REAL b;
    REAL r1;
    REAL dot;
    REAL a;
    REAL na;

    REAL *d_Ax;
    REAL *d_p;
    hipMalloc((void **)&d_Ax, n*sizeof(REAL));
    hipMalloc((void **)&d_p, n*sizeof(REAL));

    hipsparseDcsrmv(handle_cusparse,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnzA, &alpha, descr, csrValA, csrRowPtrA, csrColIndA, x, &beta, d_Ax);
    hipblasDaxpy(handle_cublas, n, &alpham1, d_Ax, 1, r, 1);


    hipblasDdot(handle_cublas, n, r, 1, r, 1, &r1);

    const REAL tol = 1.e-5;
    const int max_iter = 10000;
    int k;

    k = 1;

    while (r1 > tol*tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
            hipblasDscal(handle_cublas, n, &b, d_p, 1);
            hipblasDaxpy(handle_cublas, n, &alpha, r, 1, d_p, 1);
        }
        else
        {
            hipblasDcopy(handle_cublas, n, r, 1, d_p, 1);
        }
        hipsparseDcsrmv(handle_cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnzA, &alpha, descr, csrValA, csrRowPtrA, csrColIndA, d_p, &beta, d_Ax);
        hipblasDdot(handle_cublas, n, d_p, 1, d_Ax, 1, &dot);
        a = r1 / dot;

        hipblasDaxpy(handle_cublas, n, &a, d_p, 1, x, 1);
        na = -a;
        hipblasDaxpy(handle_cublas, n, &na, d_Ax, 1, r, 1);

        r0 = r1;
        hipblasDdot(handle_cublas, n, r, 1, r, 1, &r1);
        hipDeviceSynchronize();
        k++;
    }
    hipFree(d_p);
    hipFree(d_Ax);
}