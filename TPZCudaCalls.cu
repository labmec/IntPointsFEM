#include "TPZCudaCalls.h"
// #include "pzreal.h"
#include "pzvec.h"

#include "SpectralDecompKernels.h"
#include "StressStrainKernels.h"
#include "SigmaProjectionKernels.h"
#include "MatMulKernels.h"

#define NT 512

TPZCudaCalls::TPZCudaCalls() {
	hipsparse.h = false;
	hipblas.h = false;
}

TPZCudaCalls::~TPZCudaCalls() {
	if(cublas_h == true) {
		hipblasDestroy(handle_cublas);
	}
	if(cusparse_h == true) {
		hipsparseDestroy(handle_cusparse);			
	}
}

void TPZCudaCalls::Multiply(bool trans, int *m, int *n, int *k, double *A, int *strideA, 
	double *B, int *strideB,  double *C, int *strideC, double alpha, int nmatrices) {

	MatrixMultiplicationKernel<<<nmatrices,1>>> (trans, m, n, k, A, strideA, B, strideB, C, strideC, alpha, nmatrices);
	hipDeviceSynchronize();
	if (hipGetLastError() != hipSuccess) {
		throw std::runtime_error("failed to perform MatrixMultiplicationKernel");      
	}

}

void TPZCudaCalls::GatherOperation(int n, double *x, double *y, int *id) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseStatus_t result = hipsparseDgthr(handle_cusparse, n, x, y, id, HIPSPARSE_INDEX_BASE_ZERO);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDgthr");      
	}	
}

void TPZCudaCalls::ScatterOperation(int n, double *x, double *y, int *id) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseStatus_t result = hipsparseDsctr(handle_cusparse, n, x, id, y, HIPSPARSE_INDEX_BASE_ZERO);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDsctr");      
	}	
}

void TPZCudaCalls::DaxpyOperation(int n, double alpha, double *x, double *y) {
	if(cublas_h == false) {
		hipblas.h = true;
		hipblasStatus_t result = hipblasCreate(&handle_cublas);
		if (result != HIPBLAS_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuBLAS");      
		}			
	}
	hipblasStatus_t result = hipblasDaxpy(handle_cublas, n, &alpha, x, 1., y, 1.);
	if (result != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipblasDaxpy");      
	}	
}

void TPZCudaCalls::SpMV(int opt, int m, int k, int nnz, double alpha, double *csrVal, int *csrRowPtr, int *csrColInd, double *B, double *C) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}
	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	double beta = 0.;
	hipsparseStatus_t result;
	if(opt == 0) {
		result = hipsparseDcsrmv(handle_cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, k, nnz, &alpha, descr, csrVal, csrRowPtr, csrColInd, B, &beta, C);
	} else {
		result = hipsparseDcsrmv(handle_cusparse, HIPSPARSE_OPERATION_TRANSPOSE, m, k, nnz, &alpha, descr, csrVal, csrRowPtr, csrColInd, B, &beta, C);
	}
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDcsrmv");      
	}	
}

void TPZCudaCalls::SpMSpM(int opt, int m, int n, int k, int nnzA, double *csrValA, int *csrRowPtrA, int *csrColIndA, 
														int nnzB, double *csrValB, int *csrRowPtrB, int *csrColIndB, 
														int nnzC, double *csrValC, int *csrRowPtrC) {
	if(cusparse_h == false) {
		hipsparse.h = true;
		hipsparseStatus_t result = hipsparseCreate(&handle_cusparse);
		if (result != HIPSPARSE_STATUS_SUCCESS) {
			throw std::runtime_error("failed to initialize cuSparse");      
		}			
	}

	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	hipsparseOperation_t trans;
	if(opt == 0) {
		trans = HIPSPARSE_OPERATION_NON_TRANSPOSE;

	} else {
		trans = HIPSPARSE_OPERATION_TRANSPOSE;
	}

	int *csrColIndC;
	hipMalloc((void**)&csrColIndC, sizeof(int)*nnzC);

	hipsparseStatus_t result = hipsparseDcsrgemm(handle_cusparse, trans, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, 
					descr, nnzA, csrValA, csrRowPtrA, csrColIndA, 
					descr, nnzB, csrValB, csrRowPtrB, csrColIndB,
					descr, csrValC, csrRowPtrC, csrColIndC);
	if (result != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("failed to perform hipsparseDcsrgemm");      
	}	
}