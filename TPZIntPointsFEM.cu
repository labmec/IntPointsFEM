#include "hip/hip_runtime.h"
#include "TPZIntPointsFEM.h"
#include "TPZTensor.h"
#include "pzmatrix.h"
#include <stdlib.h>
#include "TPZTensor.h"
#include "TPZVTKGeoMesh.h"
#include "pzintel.h"
#include "pzskylstrmatrix.h"

#define NT 512

__device__ void Normalize(REAL *sigma, REAL &maxel) {
	maxel = sigma[0];
	for (int i = 1; i < 4; i++) {
		if (fabs(sigma[i]) > fabs(maxel)) {
			maxel = sigma[i];
		}
	}
	for (int i = 0; i < 4; i++) {
		sigma[i] /= maxel;
	}
}

__device__ void Interval(REAL *sigma, REAL *interval) {
	__shared__ REAL lower_vec[3];
	__shared__ REAL upper_vec[3];

	//row 1 |sigma_xx sigma_xy 0|
	lower_vec[0] = sigma[0] - fabs(sigma[3]);
	upper_vec[0] = sigma[0] + fabs(sigma[3]);

	//row 2 |sigma_xy sigma_yy 0|
	lower_vec[1] = sigma[1] - fabs(sigma[3]);
	upper_vec[1] = sigma[1] + fabs(sigma[3]);

	//row 3 |0 0 sigma_zz|
	lower_vec[2] = sigma[2];
	upper_vec[2] = sigma[2];

	interval[0] = upper_vec[0];
	interval[1] = lower_vec[0];

	for (int i = 1; i < 3; i++) {
		if (upper_vec[i] > interval[0]) { //upper interval
			interval[0] = upper_vec[i];
		}

		if (lower_vec[i] < interval[1]) { //lower interval
			interval[1] = lower_vec[i];
		}
	}
}

__device__ void NewtonIterations(REAL *interval, REAL *sigma, REAL *eigenvalues, REAL &maxel) {
	int numiterations = 20;
	REAL tol = 10e-12;

	REAL res, f, df, x;
	int it;

	for (int i = 0; i < 2; i++) {
		x = interval[i];
		it = 0;

		f = sigma[0] * sigma[1] - x * (sigma[0] + sigma[1]) + x * x - sigma[3] * sigma[3];
		res = abs(f);

		while (it < numiterations && res > tol) {
			df = -sigma[0] - sigma[1] + 2 * x;

			x -= f / df;
			f = sigma[0] * sigma[1] - x * (sigma[0] + sigma[1]) + x * x - sigma[3] * sigma[3];
			res = abs(f);
			it++;
		}
		eigenvalues[i] = x;

	}
	eigenvalues[2] = sigma[0] + sigma[1] + sigma[2] - eigenvalues[0] - eigenvalues[1];

	eigenvalues[0] *= maxel;
	eigenvalues[1] *= maxel;
	eigenvalues[2] *= maxel;

	//sorting in descending order
	for (int i = 0; i < 3; ++i) {
		for (int j = i + 1; j < 3; ++j) {
			if (eigenvalues[i] < eigenvalues[j]) {
				REAL a = eigenvalues[i];
				eigenvalues[i] = eigenvalues[j];
				eigenvalues[j] = a;
			}
		}
	}
}

__device__ void Multiplicity1(REAL *sigma, REAL eigenvalue, REAL *eigenvector) {
	__shared__ REAL det[3];
	det[0] = (sigma[0] - eigenvalue) * (sigma[1] - eigenvalue) - sigma[3] * sigma[3];
	det[1] = (sigma[0] - eigenvalue) * (sigma[2] - eigenvalue);
	det[2] = (sigma[1] - eigenvalue) * (sigma[2] - eigenvalue);

	REAL maxdet = fabs(det[0]);
	for (int i = 1; i < 3; i++) {
		if (fabs(det[i]) > fabs(maxdet)) {
			maxdet = fabs(det[i]);
		}
	}
	__shared__ REAL v[3];
	if (maxdet == fabs(det[0])) {
		v[0] = 0;
		v[1] = 0;
		v[2] = 1;

	} else if (maxdet == fabs(det[1])) {
		v[0] = 1 / det[1] * (-(sigma[2] - eigenvalue) * sigma[3]);
		v[1] = 1;
		v[2] = 0;

	} else {
		v[0] = 1;
		v[1] = 1 / det[2] * (-(sigma[2] - eigenvalue) * sigma[3]);
		v[2] = 0;
	}
	REAL norm = sqrt(v[0] * v[0] + v[1] * v[1] + v[2] * v[2]);
	eigenvector[0] = v[0] / norm;
	eigenvector[1] = v[1] / norm;
	eigenvector[2] = v[2] / norm;
}

__device__ void Multiplicity2(REAL *sigma, REAL eigenvalue, REAL *eigenvector1,
		REAL *eigenvector2) {
	__shared__ REAL x[3];
	x[0] = sigma[0] - eigenvalue;
	x[1] = sigma[1] - eigenvalue;
	x[2] = sigma[2] - eigenvalue;

	REAL maxx = fabs(x[0]);
	for (int i = 1; i < 3; i++) {
		if (fabs(x[i]) > fabs(maxx)) {
			maxx = fabs(x[i]);
		}
	}

	__shared__ REAL v1[3];
	__shared__ REAL v2[3];

	if (maxx == fabs(x[0])) {
		v1[0] = -sigma[3] / x[0];
		v1[1] = 1;
		v1[2] = 0;

		v2[0] = 0;
		v2[1] = 0;
		v2[2] = 1;

	} else if (maxx == fabs(x[1])) {
		v1[0] = 1;
		v1[1] = -sigma[3] / x[1];
		v1[2] = 0;

		v2[0] = 0;
		v2[1] = 0;
		v2[2] = 1;

	} else {
		v1[0] = 1;
		v1[1] = 0;
		v1[2] = 0;

		v2[0] = 0;
		v2[1] = 1;
		v2[2] = 0;

	}
	REAL norm1 = sqrt(v1[0] * v1[0] + v1[1] * v1[1] + v1[2] * v1[2]);
	REAL norm2 = sqrt(v2[0] * v2[0] + v2[1] * v1[1] + v2[2] * v2[2]);

	eigenvector1[0] = v1[0] / norm1;
	eigenvector1[1] = v1[1] / norm1;
	eigenvector1[2] = v1[2] / norm1;

	eigenvector2[0] = v2[0] / norm2;
	eigenvector2[1] = v2[1] / norm2;
	eigenvector2[2] = v2[2] / norm2;
}

__device__ void Eigenvectors(REAL *sigma, REAL *eigenvalues, REAL *eigenvectors,
		REAL &maxel) {
	sigma[0] *= maxel;
	sigma[1] *= maxel;
	sigma[2] *= maxel;
	sigma[3] *= maxel;

	if ((eigenvalues[0] == eigenvalues[1])
			&& (eigenvalues[1] == eigenvalues[2])) {
		eigenvectors[0] = 1.;
		eigenvectors[1] = 0.;
		eigenvectors[2] = 0.;

		eigenvectors[3] = 0.;
		eigenvectors[4] = 1.;
		eigenvectors[5] = 0.;

		eigenvectors[6] = 0.;
		eigenvectors[7] = 0.;
		eigenvectors[8] = 1.;
	} else {
		if (eigenvalues[0] != eigenvalues[1] && eigenvalues[0] != eigenvalues[2]) {
			Multiplicity1(sigma, eigenvalues[0], &eigenvectors[0]);
		} else if (eigenvalues[0] == eigenvalues[1]) {
			Multiplicity2(sigma, eigenvalues[0], &eigenvectors[0], &eigenvectors[3]);
		} else if (eigenvalues[0] == eigenvalues[2]) {
			Multiplicity2(sigma, eigenvalues[0], &eigenvectors[0], &eigenvectors[6]);
		}
		if (eigenvalues[1] != eigenvalues[0] && eigenvalues[1] != eigenvalues[2]) {
			Multiplicity1(sigma, eigenvalues[1], &eigenvectors[3]);
		} else if (eigenvalues[1] == eigenvalues[2]) {
			Multiplicity2(sigma, eigenvalues[1], &eigenvectors[3], &eigenvectors[6]);
		}
		if (eigenvalues[2] != eigenvalues[0] && eigenvalues[2] != eigenvalues[1]) {
			Multiplicity1(sigma, eigenvalues[2], &eigenvectors[6]);
		}
	}
}

extern "C" {
__global__ void MatMulcuBLASKernel(hipblasOperation_t trans, int64_t nelem,
		REAL *A, int *rowsizes, int *colsizes, int *matrixpos,
		int *rowfirstindex, int* colfirstindex, int npts, int nphis, REAL *B,
		REAL *C) {

	int iel = blockIdx.x * blockDim.x + threadIdx.x;

	REAL alpha;
	REAL beta;

	int lda, ldb, ldc;
	int Bpos, Cpos;
	int Boffset, Coffset;
	int m, n, k;
	int Apos;

	if (iel < nelem) {
		hipblasHandle_t cnpHandle; //each thread must have its own handle
		hipblasCreate(&cnpHandle);

		Apos = matrixpos[iel];

		if (trans == HIPBLAS_OP_N) {
			m = rowsizes[iel];
			n = 1;
			k = colsizes[iel];

			alpha = 1.;
			beta = 0;

			lda = m;
			ldb = k;
			ldc = m;

			Bpos = colfirstindex[iel];
			Boffset = nphis;

			Cpos = rowfirstindex[iel];
			Coffset = npts;

		} else if (trans == HIPBLAS_OP_T) {
			m = colsizes[iel];
			n = 1;
			k = rowsizes[iel];

			alpha = -1.;
			beta = 0;

			lda = k;
			ldb = k;
			ldc = m;

			Bpos = rowfirstindex[iel];
			Boffset = npts;

			Cpos = colfirstindex[iel];
			Coffset = nphis;
		}
		hipblasDgemm(cnpHandle, trans, HIPBLAS_OP_N, m, n, k, &alpha, &A[Apos],
				lda, &B[Bpos], ldb, &beta, &C[Cpos], ldc);

		hipblasDgemm(cnpHandle, trans, HIPBLAS_OP_N, m, n, k, &alpha, &A[Apos],
				lda, &B[Bpos + Boffset], ldb, &beta, &C[Cpos + Coffset], ldc);

	}
}
}

__global__ void MatMulKernel(bool trans, int64_t nelem, REAL *A, int *rowsizes,
		int *colsizes, int *matrixpos, int *rowfirstindex, int* colfirstindex,
		int npts, int nphis, REAL *B, REAL *C) {
	int iel = blockIdx.x * blockDim.x + threadIdx.x;

	REAL alpha;

	int Bpos, Cpos;
	int Boffset, Coffset;
	int m, k;
	int Apos;
	int aux1;
	int aux2;

	if (iel < nelem) {
		Apos = matrixpos[iel];

		if (trans == false) {
			m = rowsizes[iel];
			k = colsizes[iel];

			aux1 = rowsizes[iel];
			aux2 = 1;

			alpha = 1.;

			Bpos = colfirstindex[iel];
			Boffset = nphis;

			Cpos = rowfirstindex[iel];
			Coffset = npts;

		} else if (trans == true) {
			m = colsizes[iel];
			k = rowsizes[iel];

			aux1 = 1;
			aux2 = rowsizes[iel];

			alpha = -1.;

			Bpos = rowfirstindex[iel];
			Boffset = npts;

			Cpos = colfirstindex[iel];
			Coffset = nphis;
		}
		for (int i = 0; i < m; i++) {
			for (int j = 0; j < k; j++) {
				C[i + Cpos] += alpha * A[j * aux1 + i * aux2 + Apos] * B[j + Bpos];
				C[i + Cpos + Coffset] += alpha * A[j * aux1 + i * aux2 + Apos] * B[j + Bpos + Boffset];
			}
		}
	}
}

__global__ void ComputeStressKernel(int64_t fNpts, int fDim,
		REAL *elastic_strain, REAL *sigma, REAL mu, REAL lambda) {
	int ipts = blockIdx.x * blockDim.x + threadIdx.x;

	if (ipts < fNpts / fDim) {
		sigma[4 * ipts] = elastic_strain[2 * ipts] * (lambda + 2. * mu) + elastic_strain[2 * ipts + fNpts + 1] * lambda; // Sigma xx
		sigma[4 * ipts + 1] = elastic_strain[2 * ipts + fNpts + 1] * (lambda + 2. * mu) + elastic_strain[2 * ipts] * lambda; // Sigma yy
		sigma[4 * ipts + 2] = lambda * (elastic_strain[2 * ipts] + elastic_strain[2 * ipts + fNpts + 1]); // Sigma zz
		sigma[4 * ipts + 3] = mu * (elastic_strain[2 * ipts + 1] + elastic_strain[2 * ipts + fNpts]); // Sigma xy
	}
}

__global__ void ComputeStrainKernel(int64_t fNpts, int fDim, REAL *sigma, REAL *elastic_strain, REAL nu, REAL E, REAL *weight) {
	int ipts = blockIdx.x * blockDim.x + threadIdx.x;

	if (ipts < fNpts / fDim) {
		elastic_strain[2 * ipts + 0] = 1 / weight[ipts] * (1. / E * (sigma[2 * ipts] * (1. - nu * nu) - sigma[2 * ipts + fNpts + 1] * (nu + nu * nu))); //exx
		elastic_strain[2 * ipts + 1] = 1 / weight[ipts] * ((1. + nu) / E * sigma[2 * ipts + 1]); //exy
		elastic_strain[2 * ipts + fNpts + 0] = elastic_strain[2 * ipts + 1]; //exy
		elastic_strain[2 * ipts + fNpts + 1] = 1 / weight[ipts] * (1. / E * (sigma[2 * ipts + fNpts + 1] * (1. - nu * nu) - sigma[2 * ipts] * (nu + nu * nu))); //eyy
	}
}

__global__ void SpectralDecompositionKernel(int64_t fNpts, int fDim, REAL *sigma_trial, REAL *eigenvalues, REAL *eigenvectors) {
	int ipts = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ REAL maxel;
	__shared__ REAL interval[2];
	if (ipts < fNpts / fDim) {
		Normalize(&sigma_trial[4 * ipts], maxel);
		Interval(&sigma_trial[4 * ipts], &interval[0]);
		NewtonIterations(&interval[0], &sigma_trial[4 * ipts], &eigenvalues[3 * ipts], maxel);
		Eigenvectors(&sigma_trial[4 * ipts], &eigenvalues[3 * ipts], &eigenvectors[9 * ipts], maxel);
	}
}

__device__ bool PhiPlane(REAL *eigenvalues, REAL *sigma_projected, REAL mc_phi, REAL mc_cohesion) {
	const REAL sinphi = sin(mc_phi);
	const REAL cosphi = cos(mc_phi);

	REAL phi = eigenvalues[0] - eigenvalues[2]
			+ (eigenvalues[0] + eigenvalues[2]) * sinphi
			- 2. * mc_cohesion * cosphi;

	sigma_projected[0] = eigenvalues[0];
	sigma_projected[1] = eigenvalues[1];
	sigma_projected[2] = eigenvalues[2];

	bool check_validity = (fabs(phi) < 1.e-12) || (phi < 0.0);
	return check_validity;
}

__device__ bool ReturnMappingMainPlane(REAL *eigenvalues, REAL *sigma_projected, REAL &m_hardening, REAL mc_phi, REAL mc_psi, REAL mc_cohesion, REAL K, REAL G) {
	const REAL sinphi = sin(mc_phi);
	const REAL sinpsi = sin(mc_psi);
	const REAL cosphi = cos(mc_phi);
	const REAL sinphi2 = sinphi * sinphi;
	const REAL cosphi2 = 1. - sinphi2;
	const REAL constA = 4. * G * (1. + sinphi * sinpsi / 3.) + 4. * K * sinphi * sinpsi;

	REAL phi = eigenvalues[0] - eigenvalues[2] + (eigenvalues[0] + eigenvalues[2]) * sinphi - 2. * mc_cohesion * cosphi;

	REAL gamma = 0;
	int n_iterations = 30;
	for (int i = 0; i < n_iterations; i++) {
		double jac = -constA - 4. * cosphi2 * 0; // H=0
		double delta_gamma = -phi / jac;
		gamma += delta_gamma;
		phi = eigenvalues[0] - eigenvalues[2] + (eigenvalues[0] + eigenvalues[2]) * sinphi - 2. * mc_cohesion * cosphi - constA * gamma;
		if (fabs(phi) < 1.e-12) {
			break;
		}
	}

	eigenvalues[0] -= (2. * G * (1 + sinpsi / 3.) + 2. * K * sinpsi) * gamma;
	eigenvalues[1] += (4. * G / 3. - K * 2.) * sinpsi * gamma;
	eigenvalues[2] += (2. * G * (1 - sinpsi / 3.) - 2. * K * sinpsi) * gamma;
	sigma_projected[0] = eigenvalues[0];
	sigma_projected[1] = eigenvalues[1];
	sigma_projected[2] = eigenvalues[2];

	m_hardening += gamma * 2. * cosphi;

	bool check_validity = (eigenvalues[0] > eigenvalues[1] || fabs(eigenvalues[0] - eigenvalues[1]) < 1.e-12) && (eigenvalues[1] > eigenvalues[2] || fabs(eigenvalues[1] - eigenvalues[2]) < 1.e-12);
	return check_validity;
}

__device__ bool ReturnMappingRightEdge(REAL *eigenvalues, REAL *sigma_projected, REAL &m_hardening, REAL mc_phi, REAL mc_psi, REAL mc_cohesion, REAL K, REAL G) {
	const REAL sinphi = sin(mc_phi);
	const REAL sinpsi = sin(mc_psi);
	const REAL cosphi = cos(mc_phi);

	__shared__ REAL gamma[2], phi[2], sigma_bar[2], ab[2];

	__shared__ REAL jac[2][2], jac_inv[2][2];

	sigma_bar[0] = eigenvalues[0] - eigenvalues[2] + (eigenvalues[0] + eigenvalues[2]) * sinphi;
	sigma_bar[1] = eigenvalues[0] - eigenvalues[1] + (eigenvalues[0] + eigenvalues[1]) * sinphi;

	phi[0] = sigma_bar[0] - 2. * cosphi * mc_cohesion;
	phi[1] = sigma_bar[1] - 2. * cosphi * mc_cohesion;

	ab[0] = 4. * G * (1 + sinphi * sinpsi / 3.) + 4. * K * sinphi * sinpsi;
	ab[1] = 2. * G * (1. + sinphi + sinpsi - sinphi * sinpsi / 3.) + 4. * K * sinphi * sinpsi;

	int n_iterations = 30;
	for (int i = 0; i < n_iterations; i++) {

		jac[0][0] = -ab[0];
		jac[1][0] = -ab[1];
		jac[0][1] = -ab[1];
		jac[1][1] = -ab[0];

		double det_jac = jac[0][0] * jac[1][1] - jac[0][1] * jac[1][0];

		jac_inv[0][0] = jac[1][1] / det_jac;
		jac_inv[1][0] = -jac[1][0] / det_jac;
		jac_inv[0][1] = -jac[0][1] / det_jac;
		jac_inv[1][1] = jac[0][0] / det_jac;

		gamma[0] -= (jac_inv[0][0] * phi[0] + jac_inv[0][1] * phi[1]);
		gamma[1] -= (jac_inv[1][0] * phi[0] + jac_inv[1][1] * phi[1]);

		phi[0] = sigma_bar[0] - ab[0] * gamma[0] - ab[1] * gamma[1]
				- 2. * cosphi * mc_cohesion;
		phi[1] = sigma_bar[1] - ab[1] * gamma[0] - ab[0] * gamma[1]
				- 2. * cosphi * mc_cohesion;

		double res = (fabs(phi[0]) + fabs(phi[1]));

		if (fabs(res) < 1.e-12) {
			break;
		}
	}

	eigenvalues[0] -= (2. * G * (1 + sinpsi / 3.) + 2. * K * sinpsi) * (gamma[0] + gamma[1]);
	eigenvalues[1] += ((4. * G / 3. - K * 2.) * sinpsi) * gamma[0] + (2. * G * (1. - sinpsi / 3.) - 2. * K * sinpsi) * gamma[1];
	eigenvalues[2] += (2. * G * (1 - sinpsi / 3.) - 2. * K * sinpsi) * gamma[0] + ((4. * G / 3. - 2. * K) * sinpsi) * gamma[1];
	sigma_projected[0] = eigenvalues[0];
	sigma_projected[1] = eigenvalues[1];
	sigma_projected[2] = eigenvalues[2];

	m_hardening += (gamma[0] + gamma[1]) * 2. * cosphi;

	bool check_validity = (eigenvalues[0] > eigenvalues[1] || fabs(eigenvalues[0] - eigenvalues[1]) < 1.e-12)
			&& (eigenvalues[1] > eigenvalues[2] || fabs(eigenvalues[1] - eigenvalues[2]) < 1.e-12);
	return check_validity;
}

__device__ bool ReturnMappingLeftEdge(REAL *eigenvalues, REAL *sigma_projected, REAL &m_hardening, REAL mc_phi, REAL mc_psi, REAL mc_cohesion, REAL K, REAL G) {
	const REAL sinphi = sin(mc_phi);
	const REAL sinpsi = sin(mc_psi);
	const REAL cosphi = cos(mc_phi);
	const REAL sinphi2 = sinphi * sinphi;
	const REAL cosphi2 = 1. - sinphi2;

	__shared__ REAL gamma[2], phi[2], sigma_bar[2], ab[2];

	__shared__ REAL jac[2][2], jac_inv[2][2];

	sigma_bar[0] = eigenvalues[0] - eigenvalues[2] + (eigenvalues[0] + eigenvalues[2]) * sinphi;
	sigma_bar[1] = eigenvalues[1] - eigenvalues[2] + (eigenvalues[1] + eigenvalues[2]) * sinphi;

	ab[0] = 4. * G * (1 + sinphi * sinpsi / 3.) + 4. * K * sinphi * sinpsi;
	ab[1] = 2. * G * (1. - sinphi - sinpsi - sinphi * sinpsi / 3.) + 4. * K * sinphi * sinpsi;

	phi[0] = sigma_bar[0] - 2. * cosphi * mc_cohesion;
	phi[1] = sigma_bar[1] - 2. * cosphi * mc_cohesion;

	int n_iterations = 30;
	for (int i = 0; i < n_iterations; i++) {

		jac[0][0] = -ab[0] - 4. * cosphi2 * 0;
		jac[1][0] = -ab[1] - 4. * cosphi2 * 0;
		jac[0][1] = -ab[1] - 4. * cosphi2 * 0;
		jac[1][1] = -ab[0] - 4. * cosphi2 * 0;

		REAL det_jac = jac[0][0] * jac[1][1] - jac[0][1] * jac[1][0];

		jac_inv[0][0] = jac[1][1] / det_jac;
		jac_inv[1][0] = -jac[1][0] / det_jac;
		jac_inv[0][1] = -jac[0][1] / det_jac;
		jac_inv[1][1] = jac[0][0] / det_jac;

		gamma[0] -= (jac_inv[0][0] * phi[0] + jac_inv[0][1] * phi[1]);
		gamma[1] -= (jac_inv[1][0] * phi[0] + jac_inv[1][1] * phi[1]);

		phi[0] = sigma_bar[0] - ab[0] * gamma[0] - ab[1] * gamma[1] - 2. * cosphi * mc_cohesion;
		phi[1] = sigma_bar[1] - ab[1] * gamma[0] - ab[0] * gamma[1] - 2. * cosphi * mc_cohesion;

		REAL res = (fabs(phi[0]) + fabs(phi[1]));

		if (fabs(res) < 1.e-12) {
			break;
		}
	}

	eigenvalues[0] += -(2. * G * (1 + sinpsi / 3.) + 2. * K * sinpsi) * gamma[0] + ((4. * G / 3. - 2. * K) * sinpsi) * gamma[1];
	eigenvalues[1] += ((4. * G / 3. - K * 2.) * sinpsi) * gamma[0] - (2. * G * (1. + sinpsi / 3.) + 2. * K * sinpsi) * gamma[1];
	eigenvalues[2] += (2. * G * (1 - sinpsi / 3.) - 2. * K * sinpsi) * (gamma[0] + gamma[1]);
	sigma_projected[0] = eigenvalues[0];
	sigma_projected[1] = eigenvalues[1];
	sigma_projected[2] = eigenvalues[2];

	m_hardening += (gamma[0] + gamma[1]) * 2. * cosphi;

	bool check_validity = (eigenvalues[0] > eigenvalues[1] || fabs(eigenvalues[0] - eigenvalues[1]) < 1.e-12) && (eigenvalues[1] > eigenvalues[2] || fabs(eigenvalues[1] - eigenvalues[2]) < 1.e-12);
	return check_validity;
}

__device__ bool ReturnMappingApex(REAL *eigenvalues, REAL *sigma_projected, REAL &m_hardening, REAL mc_phi, REAL mc_psi, REAL mc_cohesion, REAL K) {
	const REAL cotphi = 1. / tan(mc_phi);

	REAL ptrnp1 = 0.;
	for (int i = 0; i < 3; i++) {
		ptrnp1 += eigenvalues[i];
	}
	ptrnp1 /= 3.;

	REAL DEpsPV = 0.;
	REAL alpha = cos(mc_phi) / sin(mc_psi);
	REAL res = mc_cohesion * cotphi - ptrnp1;
	REAL pnp1;

	int n_iterations = 30;
	for (int i = 0; i < n_iterations; i++) {
		const REAL jac = K; //H = 0
		DEpsPV -= res / jac;

		pnp1 = ptrnp1 - K * DEpsPV;
		res = mc_cohesion * cotphi - pnp1;

		if (fabs(res) < 1.e-12) {
			break;
		}
	}

	m_hardening += alpha * DEpsPV;
	for (int i = 0; i < 3; i++) {
		sigma_projected[i] = pnp1;
	}
}

__global__ void ProjectSigmaKernel(int64_t fNpts, int fDim, REAL mc_phi, REAL mc_psi, REAL mc_cohesion, REAL K, REAL G, REAL *eigenvalues, REAL *sigma_projected, REAL *m_type, REAL *alpha) {
	int ipts = blockIdx.x * blockDim.x + threadIdx.x;

	bool check = false;
	if (ipts < fNpts / fDim) {
		m_type[ipts] = 0;
		check = PhiPlane(&eigenvalues[3 * ipts], &sigma_projected[3 * ipts], mc_phi, mc_cohesion); //elastic domain
		if (!check) { //plastic domain
			m_type[ipts] = 1;
			check = ReturnMappingMainPlane(&eigenvalues[3 * ipts], &sigma_projected[3 * ipts], alpha[ipts], mc_phi, mc_psi, mc_cohesion, K, G); //main plane
			if (!check) { //edges or apex
				if (((1 - sin(mc_psi)) * eigenvalues[0 + 3 * ipts] - 2. * eigenvalues[1 + 3 * ipts] + (1 + sin(mc_psi)) * eigenvalues[2 + 3 * ipts]) > 0) { // right edge
					check = ReturnMappingRightEdge(&eigenvalues[3 * ipts], &sigma_projected[3 * ipts], alpha[ipts], mc_phi, mc_psi, mc_cohesion, K, G);
				} else { //left edge
					check = ReturnMappingLeftEdge(&eigenvalues[3 * ipts], &sigma_projected[3 * ipts], alpha[ipts], mc_phi, mc_psi, mc_cohesion, K, G);
				}
				if (!check) { //apex
					m_type[ipts] = -1;
					ReturnMappingApex(&eigenvalues[3 * ipts], &sigma_projected[3 * ipts], alpha[ipts], mc_phi, mc_psi, mc_cohesion, K);
				}
			}
		}
	}
}

__global__ void StressCompleteTensorKernel(int64_t fNpts, int fDim, REAL *sigma_projected, REAL *eigenvectors, REAL *sigma, REAL *weight) {
	int ipts = blockIdx.x * blockDim.x + threadIdx.x;

	if (ipts < fNpts / fDim) {

		sigma[2 * ipts + 0] = weight[ipts] * (sigma_projected[3 * ipts + 0] * eigenvectors[9 * ipts + 0] * eigenvectors[9 * ipts + 0] + sigma_projected[3 * ipts + 1]
								* eigenvectors[9 * ipts + 3] * eigenvectors[9 * ipts + 3] + sigma_projected[3 * ipts + 2] * eigenvectors[9 * ipts + 6] * eigenvectors[9 * ipts + 6]);
		sigma[2 * ipts + 1] = weight[ipts] * (sigma_projected[3 * ipts + 0] * eigenvectors[9 * ipts + 0] * eigenvectors[9 * ipts + 1] + sigma_projected[3 * ipts + 1]
								* eigenvectors[9 * ipts + 3] * eigenvectors[9 * ipts + 4] + sigma_projected[3 * ipts + 2] * eigenvectors[9 * ipts + 6] * eigenvectors[9 * ipts + 7]);
		sigma[2 * ipts + fNpts] = sigma[2 * ipts + 1];
		sigma[2 * ipts + fNpts + 1] = weight[ipts] * (sigma_projected[3 * ipts + 0] * eigenvectors[9 * ipts + 1] * eigenvectors[9 * ipts + 1] + sigma_projected[3 * ipts + 1]
								* eigenvectors[9 * ipts + 4] * eigenvectors[9 * ipts + 4] + sigma_projected[3 * ipts + 2] * eigenvectors[9 * ipts + 7] * eigenvectors[9 * ipts + 7]);
	}
}


TPZIntPointsFEM::TPZIntPointsFEM() :
		fDim(-1), fBoundaryElements(), fCmesh(0), fNpts(-1), fNphis(-1), fElemColor(
				0), fMaterial(0), fRhs(0, 0), fRhsBoundary(0, 0), fSolution(0,
				0), fPlasticStrain(0, 0), fStorage(0), fRowSizes(0), fColSizes(
				0), fMatrixPosition(0), fRowFirstIndex(0), fColFirstIndex(0), fIndexes(
				0), fIndexesColor(0), fWeight() {
//	handle_cusparse = new hipsparseHandle_t;
//	handle_cublas = new hipblasHandle_t;

	dRhs = new REAL[0];
	dRhsBoundary = new REAL[0];
	dSolution = new REAL[0];
	dPlasticStrain = new REAL[0];
	dStorage = new REAL[0];
	dRowSizes = new int[0];
	dColSizes = new int[0];
	dMatrixPosition = new int[0];
	dRowFirstIndex = new int[0];
	dColFirstIndex = new int[0];
	dIndexes = new int[0];
	dIndexesColor = new int[0];
	dWeight = new REAL[0];

}

TPZIntPointsFEM::TPZIntPointsFEM(TPZCompMesh *cmesh, int materialid) :
		fDim(-1), fBoundaryElements(), fCmesh(0), fNpts(-1), fNphis(-1), fElemColor(
				0), fMaterial(0), fRhs(0, 0), fRhsBoundary(0, 0), fSolution(0,
				0), fPlasticStrain(0, 0), fStorage(0), fRowSizes(0), fColSizes(
				0), fMatrixPosition(0), fRowFirstIndex(0), fColFirstIndex(0), fIndexes(
				0), fIndexesColor(0), fWeight() {
	SetCompMesh(cmesh);
	SetMaterialId(materialid);
//	handle_cusparse = new hipsparseHandle_t;
//	handle_cublas = new hipblasHandle_t;
	dRhs = new REAL[0];
	dRhsBoundary = new REAL[0];
	dSolution = new REAL[0];
	dPlasticStrain = new REAL[0];
	dStorage = new REAL[0];
	dRowSizes = new int[0];
	dColSizes = new int[0];
	dMatrixPosition = new int[0];
	dRowFirstIndex = new int[0];
	dColFirstIndex = new int[0];
	dIndexes = new int[0];
	dIndexesColor = new int[0];
	dWeight = new REAL[0];
}

TPZIntPointsFEM::~TPZIntPointsFEM() {
	hipFree(dRhs);
	hipFree(dRhsBoundary);
	hipFree(dSolution);
	hipFree(dPlasticStrain);
	hipFree(dStorage);
	hipFree(dRowSizes);
	hipFree(dColSizes);
	hipFree(dMatrixPosition);
	hipFree(dRowFirstIndex);
	hipFree(dColFirstIndex);
	hipFree(dIndexes);
	hipFree(dIndexesColor);
	hipFree(dWeight);

	hipblasDestroy(handle_cublas);
	hipsparseDestroy(handle_cusparse);
}

TPZIntPointsFEM::TPZIntPointsFEM(const TPZIntPointsFEM &copy) {
	fDim = copy.fDim;
	fBoundaryElements = copy.fBoundaryElements;
	fCmesh = copy.fCmesh;
	fNpts = copy.fNpts;
	fNphis = copy.fNphis;
	fElemColor = copy.fElemColor;
	fMaterial = copy.fMaterial;

	fRhs = copy.fRhs;
	fRhsBoundary = copy.fRhsBoundary;
	fSolution = copy.fSolution;
	fPlasticStrain = copy.fPlasticStrain;
	fStorage = copy.fStorage;
	fColSizes = copy.fColSizes;
	fRowSizes = copy.fRowSizes;
	fMatrixPosition = copy.fMatrixPosition;
	fRowFirstIndex = copy.fRowFirstIndex;
	fColFirstIndex = copy.fColFirstIndex;
	fIndexes = copy.fIndexes;
	fIndexesColor = copy.fIndexesColor;
	fWeight = copy.fWeight;

	handle_cusparse = copy.handle_cusparse;
	handle_cublas = copy.handle_cublas;

	dRhs = copy.dRhs;
	dRhsBoundary = copy.dRhsBoundary;
	dSolution = copy.dSolution;
	dPlasticStrain = copy.dPlasticStrain;
	dStorage = copy.dStorage;
	dRowSizes = copy.dRowSizes;
	dColSizes = copy.dColSizes;
	dMatrixPosition = copy.dMatrixPosition;
	dRowFirstIndex = copy.dRowFirstIndex;
	dColFirstIndex = copy.dColFirstIndex;
	dIndexes = copy.dIndexes;
	dIndexesColor = copy.dIndexesColor;
	dWeight = copy.dWeight;
}

TPZIntPointsFEM &TPZIntPointsFEM::operator=(const TPZIntPointsFEM &copy) {
	if (&copy == this) {
		return *this;
	}

	fDim = copy.fDim;
	fBoundaryElements = copy.fBoundaryElements;
	fCmesh = copy.fCmesh;
	fNpts = copy.fNpts;
	fNphis = copy.fNphis;
	fElemColor = copy.fElemColor;
	fMaterial = copy.fMaterial;

	fRhs = copy.fRhs;
	fRhsBoundary = copy.fRhsBoundary;
	fSolution = copy.fSolution;
	fPlasticStrain = copy.fPlasticStrain;
	fStorage = copy.fStorage;
	fColSizes = copy.fColSizes;
	fRowSizes = copy.fRowSizes;
	fMatrixPosition = copy.fMatrixPosition;
	fRowFirstIndex = copy.fRowFirstIndex;
	fColFirstIndex = copy.fColFirstIndex;
	fIndexes = copy.fIndexes;
	fIndexesColor = copy.fIndexesColor;
	fWeight = copy.fWeight;

	handle_cusparse = copy.handle_cusparse;
	handle_cublas = copy.handle_cublas;

	dRhs = copy.dRhs;
	dRhsBoundary = copy.dRhsBoundary;
	dSolution = copy.dSolution;
	dPlasticStrain = copy.dPlasticStrain;
	dStorage = copy.dStorage;
	dRowSizes = copy.dRowSizes;
	dColSizes = copy.dColSizes;
	dMatrixPosition = copy.dMatrixPosition;
	dRowFirstIndex = copy.dRowFirstIndex;
	dColFirstIndex = copy.dColFirstIndex;
	dIndexes = copy.dIndexes;
	dIndexesColor = copy.dIndexesColor;
	dWeight = copy.dWeight;

	return *this;
}

void TPZIntPointsFEM::SetDataStructure() {

	int dim_mesh = (fCmesh->Reference())->Dimension(); // Mesh dimension
	this->SetMeshDimension(dim_mesh);
	int64_t nelem_c = fCmesh->NElements(); // Number of computational elements
	std::vector<int64_t> cel_indexes;

// Number of domain geometric elements
	for (int64_t i = 0; i < nelem_c; i++) {
		TPZCompEl *cel = fCmesh->Element(i);
		if (!cel)
			continue;
		TPZGeoEl *gel = fCmesh->Element(i)->Reference();
		if (!gel)
			continue;
		if (gel->Dimension() == dim_mesh)
			cel_indexes.push_back(cel->Index());
		if (gel->Dimension() < dim_mesh)
			fBoundaryElements.Push(cel->Index());
	}

	if (cel_indexes.size() == 0) {
		DebugStop();
	}

// RowSizes and ColSizes vectors
	int64_t nelem = cel_indexes.size();
	TPZVec < MKL_INT > rowsizes(nelem);
	TPZVec < MKL_INT > colsizes(nelem);

	int64_t npts_tot = 0;
	int64_t nf_tot = 0;
	int it = 0;
	for (auto iel : cel_indexes) {
		//Verification
		TPZCompEl *cel = fCmesh->Element(iel);

		//Integration rule
		TPZInterpolatedElement *cel_inter =
				dynamic_cast<TPZInterpolatedElement *>(cel);
		if (!cel_inter)
			DebugStop();
		TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());

		int64_t npts = int_rule->NPoints(); // number of integration points of the element
		int64_t dim = cel_inter->Dimension(); //dimension of the element
		int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element

		rowsizes[it] = dim * npts;
		colsizes[it] = nf;

		it++;

		npts_tot += npts;
		nf_tot += nf;
	}
	this->SetNumberofIntPoints(dim_mesh * npts_tot);
	this->SetNumberofPhis(nf_tot);
	this->SetRowandColSizes(rowsizes, colsizes);

// Dphi matrix, weight and indexes vectors
	TPZFMatrix < REAL > elmatrix;
	TPZVec < REAL > weight(npts_tot);
	TPZManVector < MKL_INT > indexes(dim_mesh * nf_tot);

	int64_t cont1 = 0;
	int64_t cont2 = 0;
	it = 0;
	int64_t contw = 0;
	for (auto iel : cel_indexes) {
		//Verification
		TPZCompEl *cel = fCmesh->Element(iel);

		//Integration rule
		TPZInterpolatedElement *cel_inter =
				dynamic_cast<TPZInterpolatedElement *>(cel);
		if (!cel_inter)
			DebugStop();
		TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());

		int64_t npts = int_rule->NPoints(); // number of integration points of the element
		int64_t dim = cel_inter->Dimension(); //dimension of the element
		int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element

		TPZMaterialData data;
		cel_inter->InitMaterialData(data);

		elmatrix.Resize(dim * npts, nf);
		for (int64_t inpts = 0; inpts < npts; inpts++) {
			TPZManVector < REAL > qsi(dim, 1);
			REAL w;
			int_rule->Point(inpts, qsi, w);
			cel_inter->ComputeRequiredData(data, qsi);
//			weight.Push(w * std::abs(data.detjac)); //weight = w * detjac
			weight[contw] = w * std::abs(data.detjac);
			contw++;

			TPZFMatrix < REAL > axes = data.axes;
			TPZFMatrix < REAL > dphix = data.dphix;
			TPZFMatrix < REAL > dphiXY;
			axes.Transpose();
			axes.Multiply(dphix, dphiXY);

			for (int inf = 0; inf < nf; inf++) {
				for (int idim = 0; idim < dim; idim++)
					elmatrix(inpts * dim + idim, inf) = dphiXY(idim, inf);
			}
		}
		this->SetElementMatrix(it, elmatrix);
		it++;

		//Indexes vector
		int64_t ncon = cel->NConnects();
		for (int64_t icon = 0; icon < ncon; icon++) {
			int64_t id = cel->ConnectIndex(icon);
			TPZConnect &df = fCmesh->ConnectVec()[id];
			int64_t conid = df.SequenceNumber();
			if (df.NElConnected() == 0 || conid < 0
					|| fCmesh->Block().Size(conid) == 0)
				continue;
			else {
				int64_t pos = fCmesh->Block().Position(conid);
				int64_t nsize = fCmesh->Block().Size(conid);
				for (int64_t isize = 0; isize < nsize; isize++) {
					if (isize % 2 == 0) {
						indexes[cont1] = pos + isize;
						cont1++;
					} else {
						indexes[cont2 + nf_tot] = pos + isize;
						cont2++;
					}
				}
			}
		}
	}
	this->SetIndexes(indexes);
	this->SetWeightVector(weight);
	this->ColoringElements();
	this->AssembleRhsBoundary();
	this->TransferDataStructure();
}

void TPZIntPointsFEM::ColoringElements() const {
	int64_t nelem_c = fCmesh->NElements();
	int64_t nconnects = fCmesh->NConnects();
	TPZVec<int64_t> connects_vec(nconnects, 0);

	int64_t contcolor = 0;
	bool needstocontinue = true;

	while (needstocontinue) {
		int it = 0;
		needstocontinue = false;
		for (int64_t iel = 0; iel < nelem_c; iel++) {
			TPZCompEl *cel = fCmesh->Element(iel);
			if (!cel || cel->Dimension() != fCmesh->Dimension())
				continue;

			it++;
			if (fElemColor[it - 1] != -1)
				continue;

			TPZStack<int64_t> connectlist;
			fCmesh->Element(iel)->BuildConnectList(connectlist);
			int64_t ncon = connectlist.size();

			int64_t icon;
			for (icon = 0; icon < ncon; icon++) {
				if (connects_vec[connectlist[icon]] != 0)
					break;
			}
			if (icon != ncon) {
				needstocontinue = true;
				continue;
			}
			fElemColor[it - 1] = contcolor;
//            cel->Reference()->SetMaterialId(contcolor);

			for (icon = 0; icon < ncon; icon++) {
				connects_vec[connectlist[icon]] = 1;
			}
		}
		contcolor++;
		connects_vec.Fill(0);
	}
//    ofstream file("colored.vtk");
//    TPZVTKGeoMesh::PrintGMeshVTK(fCmesh->Reference(),file);

	int64_t nelem = fRowSizes.size();
	int64_t neq = fCmesh->NEquations();
	for (int64_t iel = 0; iel < nelem; iel++) {
		int64_t cols = fColSizes[iel];
		int64_t cont_cols = fColFirstIndex[iel];

		for (int64_t icols = 0; icols < cols; icols++) {
			fIndexesColor[cont_cols + icols] = fIndexes[cont_cols + icols]
					+ fElemColor[iel] * neq;
			fIndexesColor[cont_cols + fNphis + icols] = fIndexes[cont_cols
					+ fNphis + icols] + fElemColor[iel] * neq;
		}
	}
}

void TPZIntPointsFEM::AssembleRhsBoundary() {
	int64_t neq = fCmesh->NEquations();
	fRhsBoundary.Resize(neq, 1);
	fRhsBoundary.Zero();

	for (auto iel : fBoundaryElements) {
		TPZCompEl *cel = fCmesh->Element(iel);
		if (!cel)
			continue;
		TPZElementMatrix ef(fCmesh, TPZElementMatrix::EF);
		cel->CalcResidual(ef);
		ef.ComputeDestinationIndices();
		fRhsBoundary.AddFel(ef.fMat, ef.fSourceIndex, ef.fDestinationIndex);
	}
}

void TPZIntPointsFEM::TransferDataStructure() {

	hipblasCreate(&handle_cublas);
	hipsparseCreate(&handle_cusparse);

	int64_t neq = fCmesh->NEquations();
	int64_t nelem = fColSizes.size();

	hipMalloc((void**) &dRhs, neq * sizeof(REAL));
	hipMemset(dRhs, 0, neq * sizeof(REAL));

	hipMalloc((void**) &dRhsBoundary, neq * sizeof(REAL));
	hipMemcpy(dRhsBoundary, &fRhsBoundary(0, 0), neq * sizeof(REAL),
			hipMemcpyHostToDevice);

	hipMalloc((void**) &dSolution, neq * sizeof(REAL));
	hipMemset(dSolution, 0, neq * sizeof(REAL));

	hipMalloc((void**) &dPlasticStrain, fDim * fNpts * sizeof(REAL));
	hipMemset(dPlasticStrain, 0, fDim * fNpts * sizeof(REAL));

	hipMalloc((void**) &dStorage, fStorage.size() * sizeof(REAL));
	hipMemcpy(dStorage, &fStorage[0], fStorage.size() * sizeof(REAL),
			hipMemcpyHostToDevice);

	hipMalloc((void**) &dRowSizes, nelem * sizeof(int));
	hipMemcpy(dRowSizes, &fRowSizes[0], nelem * sizeof(int),
			hipMemcpyHostToDevice);

	hipMalloc((void**) &dColSizes, nelem * sizeof(int));
	hipMemcpy(dColSizes, &fColSizes[0], nelem * sizeof(int),
			hipMemcpyHostToDevice);

	hipMalloc((void**) &dMatrixPosition, nelem * sizeof(int));
	hipMemcpy(dMatrixPosition, &fMatrixPosition[0], nelem * sizeof(int),
			hipMemcpyHostToDevice);

	hipMalloc((void**) &dRowFirstIndex, nelem * sizeof(int));
	hipMemcpy(dRowFirstIndex, &fRowFirstIndex[0], nelem * sizeof(int),
			hipMemcpyHostToDevice);

	hipMalloc((void**) &dColFirstIndex, nelem * sizeof(int));
	hipMemcpy(dColFirstIndex, &fColFirstIndex[0], nelem * sizeof(int),
			hipMemcpyHostToDevice);

	hipMalloc((void**) &dIndexes, fIndexes.size() * sizeof(int));
	hipMemcpy(dIndexes, &fIndexes[0], fIndexes.size() * sizeof(int),
			hipMemcpyHostToDevice);

	hipMalloc((void**) &dIndexesColor, fIndexesColor.size() * sizeof(int));
	hipMemcpy(dIndexesColor, &fIndexesColor[0],
			fIndexesColor.size() * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**) &dWeight, fWeight.size() * sizeof(REAL));
	hipMemcpy(dWeight, &fWeight[0], fWeight.size() * sizeof(REAL),
			hipMemcpyHostToDevice);
}

void TPZIntPointsFEM::AssembleResidual() {
	int64_t ncolor = *std::max_element(fElemColor.begin(), fElemColor.end()) + 1;
	int64_t neq = fCmesh->NEquations();

	REAL *gather_solution;
	hipMalloc((void**) &gather_solution, fNpts * sizeof(REAL));
	hipMemset(gather_solution, 0, fNpts * sizeof(REAL));

	REAL *delta_strain;
	hipMalloc((void**) &delta_strain, fDim * fNpts * sizeof(REAL));
	hipMemset(delta_strain, 0, fDim * fNpts * sizeof(REAL));

	REAL *elastic_strain;
	hipMalloc((void**) &elastic_strain, fDim * fNpts * sizeof(REAL));
	hipMemset(elastic_strain, 0, fDim * fNpts * sizeof(REAL));

	REAL *sigma_trial;
	hipMalloc((void**) &sigma_trial, fDim * fNpts * sizeof(REAL));
	hipMemset(sigma_trial, 0, fDim * fNpts * sizeof(REAL));

	REAL *eigenvalues;
	hipMalloc((void**) &eigenvalues, 3 * fNpts / fDim * sizeof(REAL));
	hipMemset(eigenvalues, 0, 3 * fNpts / fDim * sizeof(REAL));

	REAL *eigenvectors;
	hipMalloc((void**) &eigenvectors, 9 * fNpts / fDim * sizeof(REAL));
	hipMemset(eigenvectors, 0, 9 * fNpts / fDim * sizeof(REAL));

	REAL *sigma_projected;
	hipMalloc((void**) &sigma_projected, 3 * fNpts / fDim * sizeof(REAL));
	hipMemset(sigma_projected, 0, 3 * fNpts / fDim * sizeof(REAL));

	REAL *sigma;
	hipMalloc((void**) &sigma, fDim * fNpts * sizeof(REAL));
	hipMemset(sigma, 0, fDim * fNpts * sizeof(REAL));

	REAL *nodal_forces;
	hipMalloc((void**) &nodal_forces, fDim * fNpts * sizeof(REAL));
	hipMemset(nodal_forces, 0, fDim * fNpts * sizeof(REAL));

	REAL *residual;
	hipMalloc((void**) &residual, neq * ncolor * sizeof(REAL));
	hipMemset(residual, 0, neq * ncolor * sizeof(REAL));

	hipMemcpy(dSolution, &fSolution(0, 0), neq * sizeof(REAL), hipMemcpyHostToDevice);
	GatherSolutionGPU(gather_solution);
	DeltaStrainGPU(gather_solution, delta_strain);
	ElasticStrainGPU(delta_strain, dPlasticStrain, elastic_strain);
	ComputeStressGPU(elastic_strain, sigma_trial);
	SpectralDecompositionGPU(sigma_trial, eigenvalues, eigenvectors);
	ProjectSigmaGPU(eigenvalues, sigma_projected);
	StressCompleteTensorGPU(sigma_projected, eigenvectors, sigma);
	NodalForcesGPU(sigma, nodal_forces);
	ColoredAssembleGPU(nodal_forces, residual);

//update strain
	ComputeStrainGPU(sigma, elastic_strain);
	PlasticStrainGPU(delta_strain, elastic_strain, dPlasticStrain);

	REAL a = 1.;
	hipblasDaxpy(handle_cublas, neq, &a, &dRhsBoundary[0], 1, &residual[0], 1);

	fRhs.Resize(neq, 1);
	hipMemcpy(&fRhs(0,0), residual, neq * sizeof(REAL), hipMemcpyDeviceToHost);

	hipFree(gather_solution);
	hipFree(delta_strain);
	hipFree(elastic_strain);
	hipFree(sigma_trial);
	hipFree(eigenvalues);
	hipFree(eigenvectors);
	hipFree(sigma_projected);
	hipFree(sigma);
	hipFree(nodal_forces);
	hipFree(residual);
}

void TPZIntPointsFEM::GatherSolutionGPU(REAL *gather_solution) {
	hipsparseDgthr(handle_cusparse, fDim * fNphis, dSolution, gather_solution, dIndexes, HIPSPARSE_INDEX_BASE_ZERO);
}

void TPZIntPointsFEM::DeltaStrainGPU(REAL *gather_solution, REAL *delta_strain) {
	int64_t nelem = fRowSizes.size();
	int numBlocks = (nelem + NT - 1) / NT;

#ifdef USING_CUBLAS_MULT //Using cuBLAS matrix-multiplication (each multiplication is done in one thread through cuBLAS library)
	hipblasOperation_t trans = HIPBLAS_OP_N;
	MatMulcuBLASKernel<<<numBlocks, NT>>>(trans, nelem, dStorage, dRowSizes, dColSizes, dMatrixPosition, dRowFirstIndex, dColFirstIndex, fNpts, fNphis, gather_solution, delta_strain);
	hipDeviceSynchronize();
#elif USING_CUSPARSE_MULT
	std::cout << "oi" << std::endl;
#else
	//Using a loop over each line of the matrices
	bool transpose = false;
	MatMulKernel<<<numBlocks, NT>>>(transpose, nelem, dStorage, dRowSizes, dColSizes, dMatrixPosition, dRowFirstIndex, dColFirstIndex, fNpts, fNphis, gather_solution, delta_strain);
	hipDeviceSynchronize();
#endif
}


void TPZIntPointsFEM::ElasticStrainGPU(REAL *delta_strain, REAL *plastic_strain, REAL *elastic_strain) {
	hipMemcpy(elastic_strain, &delta_strain[0], fDim * fNpts * sizeof(REAL), hipMemcpyDeviceToDevice);
	hipMemset(plastic_strain, 0, fDim * fNpts * sizeof(REAL));

	REAL a = -1.;
	hipblasDaxpy(handle_cublas, fDim * fNpts, &a, &plastic_strain[0], 1, &elastic_strain[0], 1);
}

void TPZIntPointsFEM::PlasticStrainGPU(REAL *delta_strain, REAL *elastic_strain, REAL *plastic_strain) {
	hipMemcpy(plastic_strain, &delta_strain[0], fDim * fNpts * sizeof(REAL), hipMemcpyDeviceToDevice);

	REAL a = -1.;
	hipblasDaxpy(handle_cublas, fDim * fNpts, &a, &elastic_strain[0], 1, &plastic_strain[0], 1);
}

//Compute stress
void TPZIntPointsFEM::ComputeStressGPU(REAL *elastic_strain, REAL *sigma) {
	REAL lambda = fMaterial->GetPlasticModel().fER.Lambda();
	REAL mu = fMaterial->GetPlasticModel().fER.Mu();

	int numBlocks = (fNpts / fDim + NT - 1) / NT;
	ComputeStressKernel<<<numBlocks, NT>>>(fNpts, fDim, elastic_strain, sigma, mu, lambda);
	hipDeviceSynchronize();
}

//Compute strain
void TPZIntPointsFEM::ComputeStrainGPU(REAL *sigma, REAL *elastic_strain) {
	REAL E = fMaterial->GetPlasticModel().fER.E();
	REAL nu = fMaterial->GetPlasticModel().fER.Poisson();

	int numBlocks = (fNpts / fDim + NT - 1) / NT;
	ComputeStrainKernel<<<numBlocks, NT>>>(fNpts, fDim, sigma, elastic_strain, nu, E, dWeight);
	hipDeviceSynchronize();
}

void TPZIntPointsFEM::SpectralDecompositionGPU(REAL *sigma_trial, REAL *eigenvalues, REAL *eigenvectors) {
	int numBlocks = (fNpts / fDim + NT - 1) / NT;
	SpectralDecompositionKernel<<<numBlocks, NT>>>(fNpts, fDim, sigma_trial, eigenvalues, eigenvectors);
	hipDeviceSynchronize();
}

void TPZIntPointsFEM::ProjectSigmaGPU(REAL *eigenvalues, REAL *sigma_projected) {

	REAL mc_phi = fMaterial->GetPlasticModel().fYC.Phi();
	REAL mc_psi = fMaterial->GetPlasticModel().fYC.Psi();
	REAL mc_cohesion = fMaterial->GetPlasticModel().fYC.Cohesion();
	REAL K = fMaterial->GetPlasticModel().fER.K();
	REAL G = fMaterial->GetPlasticModel().fER.G();

	REAL *m_type;
	hipMalloc((void**) &m_type, fNpts / fDim * sizeof(REAL));
	hipMemset(m_type, 0, fNpts / fDim * sizeof(REAL));

	REAL *alpha;
	hipMalloc((void**) &alpha, fNpts / fDim * sizeof(REAL));
	hipMemset(alpha, 0, fNpts / fDim * sizeof(REAL));

	int numBlocks = (fNpts / fDim + NT - 1) / NT;
	ProjectSigmaKernel<<<numBlocks, NT>>>(fNpts, fDim, mc_phi, mc_psi, mc_cohesion, K, G, eigenvalues, sigma_projected, m_type, alpha);
	hipDeviceSynchronize();

}

void TPZIntPointsFEM::StressCompleteTensorGPU(REAL *sigma_projected, REAL *eigenvectors, REAL *sigma) {
	int numBlocks = (fNpts / fDim + NT - 1) / NT;
	StressCompleteTensorKernel<<<numBlocks, NT>>>(fNpts, fDim, sigma_projected, eigenvectors, sigma, dWeight);
	hipDeviceSynchronize();


}

void TPZIntPointsFEM::NodalForcesGPU(REAL *sigma, REAL *nodal_forces) {
	int64_t nelem = fRowSizes.size();
	int numBlocks = (nelem + NT - 1) / NT;

#ifdef USING_CUBLAS_MULT //Using cuBLAS matrix-multiplication (each multiplication is done in one thread through cuBLAS library)
	hipblasOperation_t transA = HIPBLAS_OP_T;
	MatMulcuBLASKernel<<<numBlocks, NT>>>(transA, nelem, dStorage, dRowSizes, dColSizes, dMatrixPosition, dRowFirstIndex, dColFirstIndex, fNpts, fNphis, sigma, nodal_forces);
	hipDeviceSynchronize();
#elif USING_CUSPARSE_MULT
	std::cout << "oi" << std::endl;
#else
	//Using a loop over each line of the matrices
	bool transpose = true;
	MatMulKernel<<<numBlocks, NT>>>(transpose, nelem, dStorage, dRowSizes, dColSizes, dMatrixPosition, dRowFirstIndex, dColFirstIndex, fNpts, fNphis, sigma, nodal_forces);
	hipDeviceSynchronize();
#endif


}

void TPZIntPointsFEM::ColoredAssembleGPU(REAL *nodal_forces, REAL *residual) {
	int64_t ncolor = *std::max_element(fElemColor.begin(), fElemColor.end())
			+ 1;
	int64_t sz = fIndexes.size();
	int64_t neq = fCmesh->NEquations();

	hipsparseDsctr(handle_cusparse, sz, &nodal_forces[0], &dIndexesColor[0], &residual[0], HIPSPARSE_INDEX_BASE_ZERO);

	int64_t colorassemb = ncolor / 2.;
	REAL alpha = 1.;
	while (colorassemb > 0) {

		int64_t firsteq = (ncolor - colorassemb) * neq;
		hipblasDaxpy(handle_cublas, colorassemb * neq, &alpha, &residual[firsteq], 1., &residual[0], 1.);

		ncolor -= colorassemb;
		colorassemb = ncolor / 2;
	}
}
