#include "hip/hip_runtime.h"
#include "TPZIntPointsFEM.h"
#include "TPZTensor.h"
#include "pzmatrix.h"
#include <stdlib.h>
#include "TPZTensor.h"
#include "TPZVTKGeoMesh.h"
#include "pzintel.h"
#include "pzskylstrmatrix.h"

#include "SpectralDecompKernels.h"
#include "MatMulKernels.h"
#include "StressStrainKernels.h"
#include "SigmaProjectionKernels.h"

#define NT 512

TPZIntPointsFEM::TPZIntPointsFEM() :
		fDim(-1), fBoundaryElements(), fCmesh(0), fNpts(-1), fNphis(-1), fElemColor(
				0), fMaterial(0), fRhs(0, 0), fRhsBoundary(0, 0), fSolution(0,
				0), fPlasticStrain(0, 0), fStorage(0), fRowSizes(0), fColSizes(
				0), fMatrixPosition(0), fRowFirstIndex(0), fColFirstIndex(0), fIndexes(
				0), fIndexesColor(0), fWeight(0), fRowPtr(0), fColInd(0) {

	dRhs = new REAL[0];
	dRhsBoundary = new REAL[0];
	dSolution = new REAL[0];
	dPlasticStrain = new REAL[0];
	dStorage = new REAL[0];
	dRowSizes = new int[0];
	dColSizes = new int[0];
	dMatrixPosition = new int[0];
	dRowFirstIndex = new int[0];
	dColFirstIndex = new int[0];
	dIndexes = new int[0];
	dIndexesColor = new int[0];
	dWeight = new REAL[0];

	dRowPtr = new int[0];
	dColInd = new int[0];


}

TPZIntPointsFEM::TPZIntPointsFEM(TPZCompMesh *cmesh, int materialid) :
		fDim(-1), fBoundaryElements(), fCmesh(0), fNpts(-1), fNphis(-1), fElemColor(
				0), fMaterial(0), fRhs(0, 0), fRhsBoundary(0, 0), fSolution(0,
				0), fPlasticStrain(0, 0), fStorage(0), fRowSizes(0), fColSizes(
				0), fMatrixPosition(0), fRowFirstIndex(0), fColFirstIndex(0), fIndexes(
				0), fIndexesColor(0), fWeight(0), fRowPtr(0), fColInd(0)  {
	SetCompMesh(cmesh);
	SetMaterialId(materialid);

	dRhs = new REAL[0];
	dRhsBoundary = new REAL[0];
	dSolution = new REAL[0];
	dPlasticStrain = new REAL[0];
	dStorage = new REAL[0];
	dRowSizes = new int[0];
	dColSizes = new int[0];
	dMatrixPosition = new int[0];
	dRowFirstIndex = new int[0];
	dColFirstIndex = new int[0];
	dIndexes = new int[0];
	dIndexesColor = new int[0];
	dWeight = new REAL[0];

	dRowPtr = new int[0];
	dColInd = new int[0];
}

TPZIntPointsFEM::~TPZIntPointsFEM() {
	hipFree(dRhs);
	hipFree(dRhsBoundary);
	hipFree(dSolution);
	hipFree(dPlasticStrain);
	hipFree(dStorage);
	hipFree(dRowSizes);
	hipFree(dColSizes);
	hipFree(dMatrixPosition);
	hipFree(dRowFirstIndex);
	hipFree(dColFirstIndex);
	hipFree(dIndexes);
	hipFree(dIndexesColor);
	hipFree(dWeight);

	hipFree(dRowPtr);
	hipFree(dColInd);

	hipblasDestroy(handle_cublas);
	hipsparseDestroy(handle_cusparse);
}

TPZIntPointsFEM::TPZIntPointsFEM(const TPZIntPointsFEM &copy) {
	fDim = copy.fDim;
	fBoundaryElements = copy.fBoundaryElements;
	fCmesh = copy.fCmesh;
	fNpts = copy.fNpts;
	fNphis = copy.fNphis;
	fElemColor = copy.fElemColor;
	fMaterial = copy.fMaterial;

	fRhs = copy.fRhs;
	fRhsBoundary = copy.fRhsBoundary;
	fSolution = copy.fSolution;
	fPlasticStrain = copy.fPlasticStrain;
	fStorage = copy.fStorage;
	fColSizes = copy.fColSizes;
	fRowSizes = copy.fRowSizes;
	fMatrixPosition = copy.fMatrixPosition;
	fRowFirstIndex = copy.fRowFirstIndex;
	fColFirstIndex = copy.fColFirstIndex;
	fIndexes = copy.fIndexes;
	fIndexesColor = copy.fIndexesColor;
	fWeight = copy.fWeight;

	fRowPtr = copy.fRowPtr;
	fColInd = copy.fColInd;

	handle_cusparse = copy.handle_cusparse;
	handle_cublas = copy.handle_cublas;

	dRhs = copy.dRhs;
	dRhsBoundary = copy.dRhsBoundary;
	dSolution = copy.dSolution;
	dPlasticStrain = copy.dPlasticStrain;
	dStorage = copy.dStorage;
	dRowSizes = copy.dRowSizes;
	dColSizes = copy.dColSizes;
	dMatrixPosition = copy.dMatrixPosition;
	dRowFirstIndex = copy.dRowFirstIndex;
	dColFirstIndex = copy.dColFirstIndex;
	dIndexes = copy.dIndexes;
	dIndexesColor = copy.dIndexesColor;
	dWeight = copy.dWeight;

	dRowPtr = copy.dRowPtr;
	dColInd = copy.dColInd;
}

TPZIntPointsFEM &TPZIntPointsFEM::operator=(const TPZIntPointsFEM &copy) {
	if (&copy == this) {
		return *this;
	}

	fDim = copy.fDim;
	fBoundaryElements = copy.fBoundaryElements;
	fCmesh = copy.fCmesh;
	fNpts = copy.fNpts;
	fNphis = copy.fNphis;
	fElemColor = copy.fElemColor;
	fMaterial = copy.fMaterial;

	fRhs = copy.fRhs;
	fRhsBoundary = copy.fRhsBoundary;
	fSolution = copy.fSolution;
	fPlasticStrain = copy.fPlasticStrain;
	fStorage = copy.fStorage;
	fColSizes = copy.fColSizes;
	fRowSizes = copy.fRowSizes;
	fMatrixPosition = copy.fMatrixPosition;
	fRowFirstIndex = copy.fRowFirstIndex;
	fColFirstIndex = copy.fColFirstIndex;
	fIndexes = copy.fIndexes;
	fIndexesColor = copy.fIndexesColor;
	fWeight = copy.fWeight;

	fRowPtr = copy.fRowPtr;
	fColInd = copy.fColInd;

	handle_cusparse = copy.handle_cusparse;
	handle_cublas = copy.handle_cublas;

	dRhs = copy.dRhs;
	dRhsBoundary = copy.dRhsBoundary;
	dSolution = copy.dSolution;
	dPlasticStrain = copy.dPlasticStrain;
	dStorage = copy.dStorage;
	dRowSizes = copy.dRowSizes;
	dColSizes = copy.dColSizes;
	dMatrixPosition = copy.dMatrixPosition;
	dRowFirstIndex = copy.dRowFirstIndex;
	dColFirstIndex = copy.dColFirstIndex;
	dIndexes = copy.dIndexes;
	dIndexesColor = copy.dIndexesColor;
	dWeight = copy.dWeight;

	dRowPtr = copy.dRowPtr;
	dColInd = copy.dColInd;

	return *this;
}

void TPZIntPointsFEM::TransferDataStructure() {

	std::cout << "Initializing libraries contexts ... " << std::endl;
	hipblasCreate(&handle_cublas);
	hipsparseCreate(&handle_cusparse);

	int64_t neq = fCmesh->NEquations();
	int64_t nelem = fColSizes.size();
    int64_t nnz = fStorage.size();

	std::cout << "Allocating and trasnfering data to GPU ... " << std::endl;
	hipMalloc((void**) &dRhs, neq * sizeof(REAL));

	hipMalloc((void**) &dRhsBoundary, neq * sizeof(REAL));
	hipMemcpy(dRhsBoundary, &fRhsBoundary(0, 0), neq * sizeof(REAL), hipMemcpyHostToDevice);

	hipMalloc((void**) &dSolution, neq * sizeof(REAL));

	hipMalloc((void**) &dPlasticStrain, fDim * fNpts * sizeof(REAL));

	hipMalloc((void**) &dStorage, fStorage.size() * sizeof(REAL));
	hipMemcpy(dStorage, &fStorage[0], fStorage.size() * sizeof(REAL), hipMemcpyHostToDevice);

	hipMalloc((void**) &dIndexes, fIndexes.size() * sizeof(int));
	hipMemcpy(dIndexes, &fIndexes[0], fIndexes.size() * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**) &dIndexesColor, fIndexesColor.size() * sizeof(int));
	hipMemcpy(dIndexesColor, &fIndexesColor[0],
			fIndexesColor.size() * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**) &dWeight, fWeight.size() * sizeof(REAL));
	hipMemcpy(dWeight, &fWeight[0], fWeight.size() * sizeof(REAL), hipMemcpyHostToDevice);

#ifdef USING_CUSPARSE_MULT
	hipMalloc((void**) &dRowPtr, (fNpts + 1) * sizeof(int));
	hipMemcpy(dRowPtr, &fRowPtr[0], (fNpts + 1) * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**) &dColInd, nnz * sizeof(int));
	hipMemcpy(dColInd, &fColInd[0], nnz * sizeof(int), hipMemcpyHostToDevice);
#else
	hipMalloc((void**) &dRowSizes, nelem * sizeof(int));
	hipMemcpy(dRowSizes, &fRowSizes[0], nelem * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**) &dColSizes, nelem * sizeof(int));
	hipMemcpy(dColSizes, &fColSizes[0], nelem * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**) &dMatrixPosition, nelem * sizeof(int));
	hipMemcpy(dMatrixPosition, &fMatrixPosition[0], nelem * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**) &dRowFirstIndex, nelem * sizeof(int));
	hipMemcpy(dRowFirstIndex, &fRowFirstIndex[0], nelem * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**) &dColFirstIndex, nelem * sizeof(int));
	hipMemcpy(dColFirstIndex, &fColFirstIndex[0], nelem * sizeof(int), hipMemcpyHostToDevice);

#endif
	std::cout << "Done . " << std::endl;
}

void TPZIntPointsFEM::GatherSolution(REAL *gather_solution) {
	hipsparseDgthr(handle_cusparse, fDim * fNphis, dSolution, gather_solution, dIndexes, HIPSPARSE_INDEX_BASE_ZERO);
}

void TPZIntPointsFEM::DeltaStrain(REAL *gather_solution, REAL *delta_strain) {
	int64_t nelem = fRowSizes.size();
	int numBlocks = (nelem + NT - 1) / NT;

#ifdef USING_CUBLAS_MULT //Using cuBLAS matrix-multiplication (each multiplication is done in one thread through cuBLAS library)
	hipblasOperation_t trans = HIPBLAS_OP_N;
	MatMulcuBLASKernel<<<numBlocks, NT>>>(trans, nelem, dStorage, dRowSizes, dColSizes, dMatrixPosition, dRowFirstIndex, dColFirstIndex, fNpts, fNphis, gather_solution, delta_strain);
	hipDeviceSynchronize();

#elif USING_CUSPARSE_MULT //Using cuSPARSE Spmv
	int nnz = fStorage.size();
	REAL alpha = 1.;
	REAL beta = 0.;

	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	hipsparseDcsrmv(handle_cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE, fNpts, fNphis, nnz, &alpha, descr, dStorage, dRowPtr, dColInd, &gather_solution[0], &beta, &delta_strain[0]);
	hipsparseDcsrmv(handle_cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE, fNpts, fNphis, nnz, &alpha, descr, dStorage, dRowPtr, dColInd, &gather_solution[fNphis], &beta, &delta_strain[fNpts]);

#else //Using a loop over each line of the matrices
	bool trans = false;
	MatMulKernel<<<numBlocks, NT>>>(trans, nelem, dStorage, dRowSizes, dColSizes, dMatrixPosition, dRowFirstIndex, dColFirstIndex, fNpts, fNphis, gather_solution, delta_strain);
	hipDeviceSynchronize();
#endif
}

void TPZIntPointsFEM::ElasticStrain(REAL *delta_strain, REAL *plastic_strain, REAL *elastic_strain) {
	hipMemcpy(elastic_strain, &delta_strain[0], fDim * fNpts * sizeof(REAL), hipMemcpyDeviceToDevice);
	hipMemset(plastic_strain, 0, fDim * fNpts * sizeof(REAL));

	REAL a = -1.;
	hipblasDaxpy(handle_cublas, fDim * fNpts, &a, &plastic_strain[0], 1, &elastic_strain[0], 1);
}

void TPZIntPointsFEM::PlasticStrain(REAL *delta_strain, REAL *elastic_strain, REAL *plastic_strain) {
	hipMemcpy(plastic_strain, &delta_strain[0], fDim * fNpts * sizeof(REAL), hipMemcpyDeviceToDevice);

	REAL a = -1.;
	hipblasDaxpy(handle_cublas, fDim * fNpts, &a, &elastic_strain[0], 1, &plastic_strain[0], 1);
}

void TPZIntPointsFEM::ComputeStress(REAL *elastic_strain, REAL *sigma) {
	REAL lambda = fMaterial->GetPlasticModel().fER.Lambda();
	REAL mu = fMaterial->GetPlasticModel().fER.Mu();

	int numBlocks = (fNpts / fDim + NT - 1) / NT;
	ComputeStressKernel<<<numBlocks, NT>>>(fNpts, fDim, elastic_strain, sigma, mu, lambda);
	hipDeviceSynchronize();
}

void TPZIntPointsFEM::ComputeStrain(REAL *sigma, REAL *elastic_strain) {
	REAL E = fMaterial->GetPlasticModel().fER.E();
	REAL nu = fMaterial->GetPlasticModel().fER.Poisson();

	int numBlocks = (fNpts / fDim + NT - 1) / NT;
	ComputeStrainKernel<<<numBlocks, NT>>>(fNpts, fDim, sigma, elastic_strain, nu, E, dWeight);
	hipDeviceSynchronize();
}

void TPZIntPointsFEM::SpectralDecomposition(REAL *sigma_trial, REAL *eigenvalues, REAL *eigenvectors) {
	int numBlocks = (fNpts / fDim + NT - 1) / NT;
	SpectralDecompositionKernel<<<numBlocks, NT>>>(fNpts, fDim, sigma_trial, eigenvalues, eigenvectors);
	hipDeviceSynchronize();
}

void TPZIntPointsFEM::ProjectSigma(REAL *eigenvalues, REAL *sigma_projected) {

	REAL mc_phi = fMaterial->GetPlasticModel().fYC.Phi();
	REAL mc_psi = fMaterial->GetPlasticModel().fYC.Psi();
	REAL mc_cohesion = fMaterial->GetPlasticModel().fYC.Cohesion();
	REAL K = fMaterial->GetPlasticModel().fER.K();
	REAL G = fMaterial->GetPlasticModel().fER.G();

	REAL *m_type;
	hipMalloc((void**) &m_type, fNpts / fDim * sizeof(REAL));
//	hipMemset(m_type, 0, fNpts / fDim * sizeof(REAL));

	REAL *alpha;
	hipMalloc((void**) &alpha, fNpts / fDim * sizeof(REAL));
//	hipMemset(alpha, 0, fNpts / fDim * sizeof(REAL));

	int numBlocks = (fNpts / fDim + NT - 1) / NT;
	ProjectSigmaKernel<<<numBlocks, NT>>>(fNpts, fDim, mc_phi, mc_psi, mc_cohesion, K, G, eigenvalues, sigma_projected, m_type, alpha);
	hipDeviceSynchronize();

}

void TPZIntPointsFEM::StressCompleteTensor(REAL *sigma_projected, REAL *eigenvectors, REAL *sigma) {
	int numBlocks = (fNpts / fDim + NT - 1) / NT;
	StressCompleteTensorKernel<<<numBlocks, NT>>>(fNpts, fDim, sigma_projected, eigenvectors, sigma, dWeight);
	hipDeviceSynchronize();
}

void TPZIntPointsFEM::NodalForces(REAL *sigma, REAL *nodal_forces) {
	int64_t nelem = fRowSizes.size();
	int numBlocks = (nelem + NT - 1) / NT;

#ifdef USING_CUBLAS_MULT //Using cuBLAS matrix-multiplication (each multiplication is done in one thread through cuBLAS library)
	hipblasOperation_t transA = HIPBLAS_OP_T;
	MatMulcuBLASKernel<<<numBlocks, NT>>>(transA, nelem, dStorage, dRowSizes, dColSizes, dMatrixPosition, dRowFirstIndex, dColFirstIndex, fNpts, fNphis, sigma, nodal_forces);
	hipDeviceSynchronize();

#elif USING_CUSPARSE_MULT //Using cuSPARSE Spmv
	int nnz = fStorage.size();
	REAL alpha = -1.;
	REAL beta = 0.;

	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	hipsparseDcsrmv(handle_cusparse, HIPSPARSE_OPERATION_TRANSPOSE, fNpts, fNphis, nnz, &alpha, descr, dStorage, dRowPtr, dColInd, &sigma[0], &beta, &nodal_forces[0]);
	hipsparseDcsrmv(handle_cusparse, HIPSPARSE_OPERATION_TRANSPOSE, fNpts, fNphis, nnz, &alpha, descr, dStorage, dRowPtr, dColInd, &sigma[fNpts], &beta, &nodal_forces[fNphis]);

#else //Using a loop over each line of the matrices
	bool trans = true;
	MatMulKernel<<<numBlocks, NT>>>(trans, nelem, dStorage, dRowSizes, dColSizes, dMatrixPosition, dRowFirstIndex, dColFirstIndex, fNpts, fNphis, sigma, nodal_forces);
	hipDeviceSynchronize();
#endif
}

void TPZIntPointsFEM::ColoredAssemble(REAL *nodal_forces, REAL *residual) {
	int64_t ncolor = *std::max_element(fElemColor.begin(), fElemColor.end())
			+ 1;
	int64_t sz = fIndexes.size();
	int64_t neq = fCmesh->NEquations();

	hipsparseDsctr(handle_cusparse, sz, &nodal_forces[0], &dIndexesColor[0], &residual[0], HIPSPARSE_INDEX_BASE_ZERO);

	int64_t colorassemb = ncolor / 2.;
	REAL alpha = 1.;
	while (colorassemb > 0) {

		int64_t firsteq = (ncolor - colorassemb) * neq;
		hipblasDaxpy(handle_cublas, colorassemb * neq, &alpha, &residual[firsteq], 1., &residual[0], 1.);

		ncolor -= colorassemb;
		colorassemb = ncolor / 2;
	}
}

void TPZIntPointsFEM::AssembleResidual() {
	int64_t ncolor = *std::max_element(fElemColor.begin(), fElemColor.end()) + 1;
	int64_t neq = fCmesh->NEquations();

	REAL *gather_solution;
	hipMalloc((void**) &gather_solution, fNpts * sizeof(REAL));
	hipMemset(gather_solution, 0, fNpts * sizeof(REAL));

	REAL *delta_strain;
	hipMalloc((void**) &delta_strain, fDim * fNpts * sizeof(REAL));
	hipMemset(delta_strain, 0, fDim * fNpts * sizeof(REAL));

	REAL *elastic_strain;
	hipMalloc((void**) &elastic_strain, fDim * fNpts * sizeof(REAL));

	REAL *sigma_trial;
	hipMalloc((void**) &sigma_trial, fDim * fNpts * sizeof(REAL));

	REAL *eigenvalues;
	hipMalloc((void**) &eigenvalues, 3 * fNpts / fDim * sizeof(REAL));

	REAL *eigenvectors;
	hipMalloc((void**) &eigenvectors, 9 * fNpts / fDim * sizeof(REAL));

	REAL *sigma_projected;
	hipMalloc((void**) &sigma_projected, 3 * fNpts / fDim * sizeof(REAL));

	REAL *sigma;
	hipMalloc((void**) &sigma, fDim * fNpts * sizeof(REAL));

	REAL *nodal_forces;
	hipMalloc((void**) &nodal_forces, fDim * fNpts * sizeof(REAL));
	hipMemset(nodal_forces, 0, fDim * fNpts * sizeof(REAL));

	REAL *residual;
	hipMalloc((void**) &residual, neq * ncolor * sizeof(REAL));
	hipMemset(residual, 0, neq * ncolor * sizeof(REAL));

	hipMemcpy(dSolution, &fSolution(0, 0), neq * sizeof(REAL), hipMemcpyHostToDevice);
	GatherSolution(gather_solution);
	DeltaStrain(gather_solution, delta_strain);
	ElasticStrain(delta_strain, dPlasticStrain, elastic_strain);
	ComputeStress(elastic_strain, sigma_trial);
	SpectralDecomposition(sigma_trial, eigenvalues, eigenvectors);
	ProjectSigma(eigenvalues, sigma_projected);
	StressCompleteTensor(sigma_projected, eigenvectors, sigma);
	NodalForces(sigma, nodal_forces);
	ColoredAssemble(nodal_forces, residual);

//update strain
	ComputeStrain(sigma, elastic_strain);
	PlasticStrain(delta_strain, elastic_strain, dPlasticStrain);

	REAL a = 1.;
	hipblasDaxpy(handle_cublas, neq, &a, &dRhsBoundary[0], 1, &residual[0], 1);

	fRhs.Resize(neq, 1);
	hipMemcpy(&fRhs(0,0), residual, neq * sizeof(REAL), hipMemcpyDeviceToHost);

	hipFree(gather_solution);
	hipFree(delta_strain);
	hipFree(elastic_strain);
	hipFree(sigma_trial);
	hipFree(eigenvalues);
	hipFree(eigenvectors);
	hipFree(sigma_projected);
	hipFree(sigma);
	hipFree(nodal_forces);
	hipFree(residual);
}

void TPZIntPointsFEM::SetDataStructure() {

	int dim_mesh = (fCmesh->Reference())->Dimension(); // Mesh dimension
	this->SetMeshDimension(dim_mesh);
	int64_t nelem_c = fCmesh->NElements(); // Number of computational elements
	std::vector<int64_t> cel_indexes;

// Number of domain geometric elements
	for (int64_t i = 0; i < nelem_c; i++) {
		TPZCompEl *cel = fCmesh->Element(i);
		if (!cel)
			continue;
		TPZGeoEl *gel = fCmesh->Element(i)->Reference();
		if (!gel)
			continue;
		if (gel->Dimension() == dim_mesh)
			cel_indexes.push_back(cel->Index());
		if (gel->Dimension() < dim_mesh)
			fBoundaryElements.Push(cel->Index());
	}

	if (cel_indexes.size() == 0) {
		DebugStop();
	}

// RowSizes and ColSizes vectors
	int64_t nelem = cel_indexes.size();
	TPZVec < MKL_INT > rowsizes(nelem);
	TPZVec < MKL_INT > colsizes(nelem);

	int64_t npts_tot = 0;
	int64_t nf_tot = 0;
	int it = 0;
	for (auto iel : cel_indexes) {
		//Verification
		TPZCompEl *cel = fCmesh->Element(iel);

		//Integration rule
		TPZInterpolatedElement *cel_inter =
				dynamic_cast<TPZInterpolatedElement *>(cel);
		if (!cel_inter)
			DebugStop();
		TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());

		int64_t npts = int_rule->NPoints(); // number of integration points of the element
		int64_t dim = cel_inter->Dimension(); //dimension of the element
		int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element

		rowsizes[it] = dim * npts;
		colsizes[it] = nf;

		it++;

		npts_tot += npts;
		nf_tot += nf;
	}
	this->SetNumberofIntPoints(dim_mesh * npts_tot);
	this->SetNumberofPhis(nf_tot);
	this->SetRowandColSizes(rowsizes, colsizes);

// Dphi matrix, weight and indexes vectors
	TPZFMatrix < REAL > elmatrix;
	TPZVec < REAL > weight(npts_tot);
	TPZManVector < MKL_INT > indexes(dim_mesh * nf_tot);

	int64_t cont1 = 0;
	int64_t cont2 = 0;
	it = 0;
	int64_t contw = 0;
	for (auto iel : cel_indexes) {
		//Verification
		TPZCompEl *cel = fCmesh->Element(iel);

		//Integration rule
		TPZInterpolatedElement *cel_inter =
				dynamic_cast<TPZInterpolatedElement *>(cel);
		if (!cel_inter)
			DebugStop();
		TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());

		int64_t npts = int_rule->NPoints(); // number of integration points of the element
		int64_t dim = cel_inter->Dimension(); //dimension of the element
		int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element

		TPZMaterialData data;
		cel_inter->InitMaterialData(data);

		elmatrix.Resize(dim * npts, nf);
		for (int64_t inpts = 0; inpts < npts; inpts++) {
			TPZManVector < REAL > qsi(dim, 1);
			REAL w;
			int_rule->Point(inpts, qsi, w);
			cel_inter->ComputeRequiredData(data, qsi);
//			weight.Push(w * std::abs(data.detjac)); //weight = w * detjac
			weight[contw] = w * std::abs(data.detjac);
			contw++;

			TPZFMatrix < REAL > axes = data.axes;
			TPZFMatrix < REAL > dphix = data.dphix;
			TPZFMatrix < REAL > dphiXY;
			axes.Transpose();
			axes.Multiply(dphix, dphiXY);

			for (int inf = 0; inf < nf; inf++) {
				for (int idim = 0; idim < dim; idim++)
					elmatrix(inpts * dim + idim, inf) = dphiXY(idim, inf);
			}
		}
#ifdef USING_CUSPARSE_MULT
		elmatrix.Transpose();
#endif
		this->SetElementMatrix(it, elmatrix);
		it++;

		//Indexes vector
		int64_t ncon = cel->NConnects();
		for (int64_t icon = 0; icon < ncon; icon++) {
			int64_t id = cel->ConnectIndex(icon);
			TPZConnect &df = fCmesh->ConnectVec()[id];
			int64_t conid = df.SequenceNumber();
			if (df.NElConnected() == 0 || conid < 0
					|| fCmesh->Block().Size(conid) == 0)
				continue;
			else {
				int64_t pos = fCmesh->Block().Position(conid);
				int64_t nsize = fCmesh->Block().Size(conid);
				for (int64_t isize = 0; isize < nsize; isize++) {
					if (isize % 2 == 0) {
						indexes[cont1] = pos + isize;
						cont1++;
					} else {
						indexes[cont2 + nf_tot] = pos + isize;
						cont2++;
					}
				}
			}
		}
	}
	this->SetIndexes(indexes);
	this->SetWeightVector(weight);
	this->ColoringElements();
	this->AssembleRhsBoundary();
#ifdef USING_CUSPARSE_MULT
	this->CSRInfo();
#endif
	this->TransferDataStructure();
}

void TPZIntPointsFEM::ColoringElements() const {
	int64_t nelem_c = fCmesh->NElements();
	int64_t nconnects = fCmesh->NConnects();
	TPZVec<int64_t> connects_vec(nconnects, 0);

	int64_t contcolor = 0;
	bool needstocontinue = true;

	while (needstocontinue) {
		int it = 0;
		needstocontinue = false;
		for (int64_t iel = 0; iel < nelem_c; iel++) {
			TPZCompEl *cel = fCmesh->Element(iel);
			if (!cel || cel->Dimension() != fCmesh->Dimension())
				continue;

			it++;
			if (fElemColor[it - 1] != -1)
				continue;

			TPZStack<int64_t> connectlist;
			fCmesh->Element(iel)->BuildConnectList(connectlist);
			int64_t ncon = connectlist.size();

			int64_t icon;
			for (icon = 0; icon < ncon; icon++) {
				if (connects_vec[connectlist[icon]] != 0)
					break;
			}
			if (icon != ncon) {
				needstocontinue = true;
				continue;
			}
			fElemColor[it - 1] = contcolor;
//            cel->Reference()->SetMaterialId(contcolor);

			for (icon = 0; icon < ncon; icon++) {
				connects_vec[connectlist[icon]] = 1;
			}
		}
		contcolor++;
		connects_vec.Fill(0);
	}
//    ofstream file("colored.vtk");
//    TPZVTKGeoMesh::PrintGMeshVTK(fCmesh->Reference(),file);

	int64_t nelem = fRowSizes.size();
	int64_t neq = fCmesh->NEquations();
	for (int64_t iel = 0; iel < nelem; iel++) {
		int64_t cols = fColSizes[iel];
		int64_t cont_cols = fColFirstIndex[iel];

		for (int64_t icols = 0; icols < cols; icols++) {
			fIndexesColor[cont_cols + icols] = fIndexes[cont_cols + icols]
					+ fElemColor[iel] * neq;
			fIndexesColor[cont_cols + fNphis + icols] = fIndexes[cont_cols
					+ fNphis + icols] + fElemColor[iel] * neq;
		}
	}
}

void TPZIntPointsFEM::AssembleRhsBoundary() {
	int64_t neq = fCmesh->NEquations();
	fRhsBoundary.Resize(neq, 1);
	fRhsBoundary.Zero();

	for (auto iel : fBoundaryElements) {
		TPZCompEl *cel = fCmesh->Element(iel);
		if (!cel)
			continue;
		TPZElementMatrix ef(fCmesh, TPZElementMatrix::EF);
		cel->CalcResidual(ef);
		ef.ComputeDestinationIndices();
		fRhsBoundary.AddFel(ef.fMat, ef.fSourceIndex, ef.fDestinationIndex);
	}
}
