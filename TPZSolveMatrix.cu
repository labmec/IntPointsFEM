#include "hip/hip_runtime.h"
#include "TPZSolveMatrix.h"
#include "pzmatrix.h"
#include <stdlib.h>
# include <chrono>
#ifdef USING_MKL
#include <mkl.h>
#include <algorithm>
#endif

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <omp.h>
#include <chrono>

using namespace std::chrono;


//CUDA KERNELS
__global__ void ComputeSigmaKernel(int npts_tot, double *weight, double *result, double *sigma) {
    REAL E = 200000000.;
    REAL nu = 0.30;
    int ipts = blockIdx.x * blockDim.x + threadIdx.x;

    if (ipts < npts_tot / 2) {
        //plane strain
        sigma[2 * ipts] = weight[ipts] * (result[2 * ipts] * E * (1. - nu) / ((1. - 2 * nu) * (1. + nu)) + result[2 * ipts + npts_tot + 1] * E * nu / ((1. - 2 * nu) * (1. + nu))); // Sigma x
        sigma[2 * ipts + 1] = weight[ipts] * E / (2 * (1. + nu)) * (result[2 * ipts + 1] + result[2 * ipts + npts_tot]); // Sigma xy
        sigma[2 * ipts + npts_tot] = sigma[2 * ipts + 1]; //Sigma xy
        sigma[2 * ipts + npts_tot + 1] = weight[ipts] * (result[2 * ipts + npts_tot + 1] * E * (1. - nu) / ((1. - 2 * nu) * (1. + nu)) + result[2 * ipts] * E * nu / ((1. - 2 * nu) * (1. + nu))); // Sigma y
//        sigma[2 * ipts] = weight[ipts] * E / (1. - nu * nu) * (result[2 * ipts] + nu * result[2 * ipts + npts_tot + 1]); // Sigma x
//        sigma[2 * ipts + 1] = weight[ipts] * E / (1. - nu * nu) * (1. - nu) / 2 * (result[2 * ipts + 1] + result[2 * ipts + npts_tot]) * 0.5; // Sigma xy
//        sigma[2 * ipts + npts_tot] = sigma[2 * ipts + 1]; //Sigma xy
//        sigma[2 * ipts + npts_tot + 1] = weight[ipts] * E / (1. - nu * nu) * (result[2 * ipts + npts_tot + 1] + nu * result[2 * ipts]); // Sigma y
    }
}

__global__ void MultiplyInThreadsKernel (double *storage, int *rowsizes, int *colsizes, int *matrixposition, int *rowfirstindex, int *colfirstindex, double *result, double *expandsolution, int nindexes, int nelem) {
    int iel = blockIdx.x * blockDim.x + threadIdx.x;

    if (iel < nelem) {
        for (int i = 0; i < rowsizes[iel]; i++) {
            for (int j = 0; j < 1; j++) {
                for (int k = 0; k < colsizes[iel]; k++) {
                    result[j * rowsizes[iel] + i + rowfirstindex[iel]] += storage[k * rowsizes[iel] + i + matrixposition[iel]] * expandsolution[j * colsizes[iel] + k + colfirstindex[iel]];
                    result[j * rowsizes[iel] + i + rowfirstindex[iel] + nindexes] += storage[k * rowsizes[iel] + i + matrixposition[iel]] * expandsolution[j * colsizes[iel] + k + colfirstindex[iel] + nindexes/2];
                }
            }
        }
    }

}

void TPZSolveMatrix::AllocateMemory(TPZCompMesh *cmesh) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int nelem = fRowSizes.size();
    int nindexes = fIndexes.size();
    int neq = cmesh->NEquations();
    int npts_tot = fRow;
    int64_t ncolor = *std::max_element(fElemColor.begin(), fElemColor.end()) + 1;

    hipMalloc(&dglobal_solution, neq * sizeof(double));
    hipMalloc(&dindexes, nindexes * sizeof(int));
    hipMalloc(&dstorage, nelem*fColSizes[0]*fRowSizes[0] * sizeof(double));
    hipMalloc(&dexpandsolution, nindexes * sizeof(double));
    hipMalloc(&dresult, 2 * nindexes * sizeof(double));
    hipMalloc(&dweight, npts_tot/2 * sizeof(double));
    hipMalloc(&dsigma, 2 * npts_tot * sizeof(double));
    hipMalloc(&dnodal_forces_vec, npts_tot * sizeof(double));
    hipMalloc(&dindexescolor, nindexes * sizeof(int));
    hipMalloc(&dnodal_forces_global, ncolor * neq * sizeof(double));

    hipMalloc(&dfRowSizes, nelem * sizeof(int));
    hipMalloc(&dfColSizes, nelem * sizeof(int));
    hipMalloc(&dfMatrixPosition, nelem * sizeof(int));
    hipMalloc(&dfRowFirstIndex, nelem * sizeof(int));
    hipMalloc(&dfColFirstIndex, nelem * sizeof(int));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Allocate: " << milliseconds/1000 << std::endl;

}


void TPZSolveMatrix::FreeMemory() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipFree(dglobal_solution);
    hipFree(dindexes);
    hipFree(dstorage);
    hipFree(dexpandsolution);
    hipFree(dresult);
    hipFree(dweight);
    hipFree(dsigma);
    hipFree(dnodal_forces_vec);
    hipFree(dindexescolor);
    hipFree(dnodal_forces_global);

    hipFree(dfRowSizes);
    hipFree(dfColSizes);
    hipFree(dfMatrixPosition);
    hipFree(dfRowFirstIndex);
    hipFree(dfColFirstIndex);

    hipblasDestroy(handle_cublas);
    hipsparseDestroy(handle_cusparse);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Free: " << milliseconds/1000 << std::endl;

}

void TPZSolveMatrix::cuSparseHandle() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipsparseCreate (&handle_cusparse);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "cuSPARSE: " << milliseconds/1000 << std::endl;

}

void TPZSolveMatrix::cuBlasHandle() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipblasCreate (&handle_cublas);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "cuBLAS: " << milliseconds/1000 << std::endl;
}

void TPZSolveMatrix::MultiplyInThreadsCUDA(TPZFMatrix<STATE> &global_solution, TPZFMatrix<STATE> &result) {
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int64_t nelem = fRowSizes.size();
    int64_t nindexes = fIndexes.size();
    int64_t cols = fColSizes[0];
    int64_t rows = fRowSizes[0];
    result.Resize(2 * nindexes, 1);
    result.Zero();

    hipEventRecord(start);

    hipMemcpy(dindexes, &fIndexes[0], nindexes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dglobal_solution, &global_solution[0], global_solution.Rows() * sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    hipsparseDgthr(handle_cusparse, nindexes, dglobal_solution, &dexpandsolution[0], &dindexes[0], HIPSPARSE_INDEX_BASE_ZERO);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Gather: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    hipMemcpy(dstorage, &fStorage[0], nelem*fColSizes[0]*fRowSizes[0] * sizeof(double), hipMemcpyHostToDevice);

    int *drowsizes;
    hipMalloc(&drowsizes, nelem * sizeof(int));
    hipMemcpy(drowsizes, &fRowSizes[0], nelem * sizeof(int), hipMemcpyHostToDevice);


    hipMemcpy(dfRowSizes, &fRowSizes[0], nelem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dfColSizes, &fColSizes[0], nelem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dfMatrixPosition, &fMatrixPosition[0], nelem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dfRowFirstIndex, &fRowFirstIndex[0], nelem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dfColFirstIndex, &fColFirstIndex[0], nelem * sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    int numthreads = 32;
    int numblocks = (nelem + numthreads - 1)/numthreads;

    MultiplyInThreadsKernel<<<numblocks, numthreads>>> (dstorage, drowsizes, dfColSizes, dfMatrixPosition, dfRowFirstIndex, dfColFirstIndex, dresult, dexpandsolution, nindexes, nelem);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Multiply: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    hipMemcpy(&result(0, 0), dresult, 2 * nindexes * sizeof(double), hipMemcpyDeviceToHost);
    result.Print(std::cout);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;

}

void TPZSolveMatrix::MultiplyCUDA(const TPZFMatrix<STATE> &global_solution, TPZFMatrix<STATE> &result) const {
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int64_t nelem = fRowSizes.size();
    int64_t nindexes = fIndexes.size();
    int64_t cols = fColSizes[0];
    int64_t rows = fRowSizes[0];
    result.Resize(2 * nindexes, 1);
    result.Zero();

    hipEventRecord(start);

    hipMemcpy(dindexes, &fIndexes[0], nindexes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dglobal_solution, &global_solution[0], global_solution.Rows() * sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    hipsparseDgthr(handle_cusparse, nindexes, dglobal_solution, &dexpandsolution[0], &dindexes[0], HIPSPARSE_INDEX_BASE_ZERO);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Gather: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    hipMemcpy(dstorage, &fStorage[0], nelem*fColSizes[0]*fRowSizes[0] * sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    double alpha = 1.0;
    double beta = 0.0;

    hipblasDgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, rows, 1, cols, &alpha, dstorage, rows, rows*cols, &dexpandsolution[0], cols, cols*1, &beta, &dresult[0], rows, rows*1, nelem);

    hipblasDgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, rows, 1, cols, &alpha, dstorage, rows, rows*cols, &dexpandsolution[fColFirstIndex[nelem]], cols, cols*1, &beta,  &dresult[fRowFirstIndex[nelem]], rows, rows*1, nelem);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Multiply: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    hipMemcpy(&result(0, 0), dresult, 2 * nindexes * sizeof(double), hipMemcpyDeviceToHost);
//    result.Print(std::cout);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;

}

void TPZSolveMatrix::ComputeSigmaCUDA(TPZStack<REAL> &weight, TPZFMatrix<REAL> &result, TPZFMatrix<STATE> &sigma) {
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    REAL E = 200000000.;
    REAL nu = 0.30;
    int npts_tot = fRow;
    int nindexes = fIndexes.size();
    sigma.Resize(2 * npts_tot, 1);
    sigma.Zero();

    hipEventRecord(start);

    hipMemcpy(dweight, &weight[0], weight.size() * sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    dim3 dimGrid(ceil((npts_tot / 2) / 256.0), 1, 1);
    dim3 dimBlock(256, 1, 1);
    ComputeSigmaKernel <<< dimGrid, dimBlock >>> (npts_tot, dweight, dresult, dsigma);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Sigma: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    hipMemcpy(&sigma(0, 0), dsigma, 2 * npts_tot * sizeof(double), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;

}

void TPZSolveMatrix::MultiplyTransposeCUDA(TPZFMatrix<STATE> &sigma, TPZFMatrix<STATE> &nodal_forces_vec) {
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int64_t nelem = fRowSizes.size();
    int64_t npts_tot = fRow;
    nodal_forces_vec.Resize(npts_tot, 1);
    nodal_forces_vec.Zero();

    double alpha = 1.0;
    double beta = 0.;

    int64_t cols = fColSizes[0];
    int64_t rows = fRowSizes[0];

    hipEventRecord(start);

    hipblasDgemmStridedBatched(handle_cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, cols, 1, rows, &alpha, dstorage, rows, 0, &dsigma[0], rows, rows*1, &beta, &dnodal_forces_vec[0], cols, cols*1, nelem);

    hipblasDgemmStridedBatched(handle_cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, cols, 1, rows, &alpha, dstorage, rows, 0, &dsigma[npts_tot], rows, rows*1, &beta,  &dnodal_forces_vec[npts_tot / 2], cols, cols*1, nelem);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Transpose: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);
    hipMemcpy(&nodal_forces_vec(0, 0), dnodal_forces_vec, npts_tot * sizeof(double), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;

}

void TPZSolveMatrix::ColoredAssembleCUDA(TPZFMatrix<STATE> &nodal_forces_vec, TPZFMatrix<STATE> &nodal_forces_global) {
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int64_t ncolor = *std::max_element(fElemColor.begin(), fElemColor.end()) + 1;
    int64_t nindexes = fIndexes.size();
    int64_t neq = nodal_forces_global.Rows();
    int64_t npts_tot = fRow;

    nodal_forces_global.Resize(neq * ncolor, 1);
    nodal_forces_global.Zero();

    hipEventRecord(start);

    hipMemcpy(dindexescolor, &fIndexesColor[0], nindexes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dnodal_forces_global, &nodal_forces_global(0, 0), ncolor * neq * sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    hipsparseDsctr(handle_cusparse, nindexes, dnodal_forces_vec, &dindexescolor[0], &dnodal_forces_global[0], HIPSPARSE_INDEX_BASE_ZERO);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Scatter: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    int64_t colorassemb = ncolor / 2;
    double alpha = 1.;
    while (colorassemb > 0) {

        int64_t firsteq = (ncolor - colorassemb) * neq;

        hipblasDaxpy(handle_cublas, firsteq, &alpha, &dnodal_forces_global[firsteq], 1., &dnodal_forces_global[0], 1.);

        ncolor -= colorassemb;
        colorassemb = ncolor / 2;
    }

    nodal_forces_global.Resize(neq, 1);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Assemble: " << milliseconds/1000 << std::endl;

    hipEventRecord(start);

    hipMemcpy(&nodal_forces_global(0, 0), dnodal_forces_global, neq * sizeof(double), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Copy: " << milliseconds/1000 << std::endl;
}

void TPZSolveMatrix::MultiplyInThreads(TPZFMatrix<STATE> &global_solution, TPZFMatrix<STATE> &result) {
    int64_t nelem = fRowSizes.size();
    int64_t n_globalsol = fIndexes.size();

    result.Resize(2*n_globalsol,1);
    result.Zero();

    TPZVec<REAL> expandsolution(n_globalsol);

/// gather operation
    cblas_dgthr(n_globalsol, global_solution, &expandsolution[0], &fIndexes[0]);

    for (int iel = 0; iel < nelem; iel++) {
        for (int i = 0; i < fRowSizes[iel]; i++) {
            for (int j = 0; j < 1; j++) {
                for (int k = 0; k < fColSizes[iel]; k++) {
                    result(j * fRowSizes[iel] + i + fRowFirstIndex[iel], 0) += fStorage[k * fRowSizes[iel] + i + fMatrixPosition[iel]] * expandsolution[j * fColSizes[iel] + k + fColFirstIndex[iel]];
                    result(j * fRowSizes[iel] + i + fRowFirstIndex[iel] + n_globalsol, 0) += fStorage[k * fRowSizes[iel] + i + fMatrixPosition[iel]] * expandsolution[j * fColSizes[iel] + k + fColFirstIndex[iel] + n_globalsol/2];
                }
            }
        }
    }
}


void TPZSolveMatrix::Multiply(const TPZFMatrix<STATE> &global_solution, TPZFMatrix<STATE> &result) const {
    int64_t nelem = fRowSizes.size();

    int64_t nindexes = fIndexes.size();

    result.Resize(2 * nindexes, 1);
    result.Zero();

    TPZVec<REAL> expandsolution(nindexes);

    high_resolution_clock::time_point t1, t2;
    duration<double> time_span;

    t1 = high_resolution_clock::now();
    cblas_dgthr(nindexes, global_solution, &expandsolution[0], &fIndexes[0]);
    t2 = high_resolution_clock::now();
    time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "Gather: " << time_span.count() << std::endl;

    t1 = high_resolution_clock::now();
    for (int64_t iel = 0; iel < nelem; iel++) {
        int64_t pos = fMatrixPosition[iel];
        int64_t cols = fColSizes[iel];
        int64_t rows = fRowSizes[iel];
        TPZFMatrix<REAL> elmatrix(rows, cols, &fStorage[0], rows * cols);

        int64_t cont_cols = fColFirstIndex[iel];
        int64_t cont_rows = fRowFirstIndex[iel];

        TPZFMatrix<REAL> element_solution_x(cols, 1, &expandsolution[cont_cols], cols);
        TPZFMatrix<REAL> element_solution_y(cols, 1, &expandsolution[cont_cols + fColFirstIndex[nelem]], cols);

        TPZFMatrix<REAL> solx(rows, 1, &result(cont_rows, 0), rows); //du
        TPZFMatrix<REAL> soly(rows, 1, &result(cont_rows + fRowFirstIndex[nelem], 0), rows); //dv

        elmatrix.Multiply(element_solution_x, solx);
        elmatrix.Multiply(element_solution_y, soly);
    }
    t2 = high_resolution_clock::now();
    time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "Multiply: " << time_span.count() << std::endl;

}

void TPZSolveMatrix::ComputeSigma(TPZStack<REAL> &weight, TPZFMatrix<REAL> &result, TPZFMatrix<STATE> &sigma) {
    high_resolution_clock::time_point t1, t2;
    duration<double> time_span;

    t1 = high_resolution_clock::now();

    REAL E = 200000000.;
    REAL nu = 0.30;
    int npts_tot = fRow;
    sigma.Resize(2 * npts_tot, 1);

    for (int64_t ipts = 0; ipts < npts_tot / 2; ipts++) {
        sigma(2 * ipts, 0) = weight[ipts] * E / (1. - nu * nu) *
                             (result(2 * ipts, 0) + nu * result(2 * ipts + npts_tot + 1, 0)); // Sigma x
        sigma(2 * ipts + 1, 0) = weight[ipts] * E / (1. - nu * nu) * (1. - nu) / 2 *
                                 (result(2 * ipts + 1, 0) + result(2 * ipts + npts_tot, 0)) * 0.5; // Sigma xy
        sigma(2 * ipts + npts_tot, 0) = sigma(2 * ipts + 1, 0); //Sigma xy
        sigma(2 * ipts + npts_tot + 1, 0) = weight[ipts] * E / (1. - nu * nu) *
                                            (result(2 * ipts + npts_tot + 1, 0) + nu * result(2 * ipts, 0)); // Sigma y
    }
    t2 = high_resolution_clock::now();
    time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "Sigma: " << time_span.count() << std::endl;


}

void TPZSolveMatrix::MultiplyTranspose(TPZFMatrix<STATE> &intpoint_solution, TPZFMatrix<STATE> &nodal_forces_vec) {
    high_resolution_clock::time_point t1, t2;
    duration<double> time_span;

    t1 = high_resolution_clock::now();

    int64_t nelem = fRowSizes.size();
    int64_t npts_tot = fRow;
    nodal_forces_vec.Resize(npts_tot, 1);
    nodal_forces_vec.Zero();


    for (int64_t iel = 0; iel < nelem; iel++) {
        int64_t pos = fMatrixPosition[iel];
        int64_t rows = fRowSizes[iel];
        int64_t cols = fColSizes[iel];
        int64_t cont_rows = fRowFirstIndex[iel];
        int64_t cont_cols = fColFirstIndex[iel];
        TPZFMatrix<REAL> elmatrix(rows, cols, &fStorage[pos], rows * cols);

        // Nodal forces in x direction
        TPZFMatrix<REAL> fvx(rows, 1, &intpoint_solution(cont_rows, 0), rows);
        TPZFMatrix<STATE> nodal_forcex(cols, 1, &nodal_forces_vec(cont_cols, 0), cols);
        elmatrix.MultAdd(fvx, nodal_forcex, nodal_forcex, 1, 0, 1);

        // Nodal forces in y direction
        TPZFMatrix<REAL> fvy(rows, 1, &intpoint_solution(cont_rows + npts_tot, 0), rows);
        TPZFMatrix<STATE> nodal_forcey(cols, 1, &nodal_forces_vec(cont_cols + npts_tot / 2, 0), cols);
        elmatrix.MultAdd(fvy, nodal_forcey, nodal_forcey, 1, 0, 1);
    }
    t2 = high_resolution_clock::now();
    time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "Transpose: " << time_span.count() << std::endl;


}

void TPZSolveMatrix::ColoredAssemble(TPZFMatrix<STATE> &nodal_forces_vec, TPZFMatrix<STATE> &nodal_forces_global) {
    high_resolution_clock::time_point t1, t2;
    duration<double> time_span;

    t1 = high_resolution_clock::now();


    int64_t ncolor = *std::max_element(fElemColor.begin(), fElemColor.end()) + 1;
    int64_t sz = fIndexes.size();
    int64_t neq = nodal_forces_global.Rows();
    nodal_forces_global.Resize(neq * ncolor, 1);

    cblas_dsctr(sz, nodal_forces_vec, &fIndexesColor[0], &nodal_forces_global(0, 0));

    t2 = high_resolution_clock::now();
    time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "Scatter: " << time_span.count() << std::endl;


    t1 = high_resolution_clock::now();

    int64_t colorassemb = ncolor / 2.;
    while (colorassemb > 0) {

        int64_t firsteq = (ncolor - colorassemb) * neq;

        cblas_daxpy(firsteq, 1., &nodal_forces_global(firsteq, 0), 1., &nodal_forces_global(0, 0), 1.);

        ncolor -= colorassemb;
        colorassemb = ncolor / 2;
    }
    nodal_forces_global.Resize(neq, 1);
    t2 = high_resolution_clock::now();
    time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "Assemble: " << time_span.count() << std::endl;


}

void TPZSolveMatrix::TraditionalAssemble(TPZFMatrix<STATE> &nodal_forces_vec, TPZFMatrix<STATE> &nodal_forces_global) const {
    for (int64_t ir = 0; ir < fRow; ir++) {
        nodal_forces_global(fIndexes[ir], 0) += nodal_forces_vec(ir, 0);
    }
}

void TPZSolveMatrix::ColoringElements(TPZCompMesh *cmesh) const {
    int64_t nelem_c = cmesh->NElements();
    int64_t nconnects = cmesh->NConnects();
    TPZVec<int64_t> connects_vec(nconnects, 0);

    int64_t contcolor = 0;
    bool needstocontinue = true;

    while (needstocontinue) {
        needstocontinue = false;
        for (int64_t iel = 0; iel < nelem_c; iel++) {
            TPZCompEl *cel = cmesh->Element(iel);
            if (!cel || cel->Dimension() != cmesh->Dimension()) continue;

            if (fElemColor[iel] != -1) continue;
            TPZStack<int64_t> connectlist;
            cmesh->Element(iel)->BuildConnectList(connectlist);
            int64_t ncon = connectlist.size();

            int64_t icon;
            for (icon = 0; icon < ncon; icon++) {
                if (connects_vec[connectlist[icon]] != 0) break;
            }
            if (icon != ncon) {
                needstocontinue = true;
                continue;
            }
            fElemColor[iel] = contcolor;

            for (icon = 0; icon < ncon; icon++) {
                connects_vec[connectlist[icon]] = 1;
            }
        }
        contcolor++;
        connects_vec.Fill(0);
    }

    int64_t nelem = fRowSizes.size();
    int64_t neq = cmesh->NEquations();
    for (int64_t iel = 0; iel < nelem; iel++) {
        int64_t cols = fColSizes[iel];
        int64_t cont_cols = fColFirstIndex[iel];

        for (int64_t icols = 0; icols < cols; icols++) {
            fIndexesColor[cont_cols + icols] = fIndexes[cont_cols + icols] + fElemColor[iel] * neq;
            fIndexesColor[cont_cols + fRow / 2 + icols] =
                    fIndexes[cont_cols + fRow / 2 + icols] + fElemColor[iel] * neq;
        }
    }
}
