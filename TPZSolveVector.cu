#include "TPZSolveVector.h"
#include "pzmatrix.h"
#include <mkl.h>
#include <stdlib.h>
# include <chrono>
#ifdef USING_MKL

#include <mkl.h>
#include <algorithm>

#endif

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <omp.h>


void TPZSolveVector::AllocateMemory(TPZCompMesh *cmesh) {
//    hipEvent_t start, stop;
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    hipEventRecord(start);

    int nelem = fRowSizes.size();
    int nindexes = fIndexes.size()/2; //numero real de indices(sem duplicar)
    int neq = cmesh->NEquations();
    int npts_tot = fRow;
    int64_t ncolor = *std::max_element(fElemColor.begin(), fElemColor.end()) + 1;

    hipMalloc(&dglobal_solution, neq * sizeof(double));
    hipMalloc(&dindexes, 2*nindexes * sizeof(int)); //2* pq esta duplicado
    hipMalloc(&dstoragevec, nelem*fColSizes[0]*fRowSizes[0] * sizeof(double));
    hipMalloc(&dexpandsolution, 2*nindexes * sizeof(double)); //sol duplicada
    hipMalloc(&dresult, 2 * nindexes * sizeof(double));
    hipMalloc(&dweight, npts_tot/2 * sizeof(double));
    hipMalloc(&dsigma, 2 * npts_tot * sizeof(double));
    hipMalloc(&dnodal_forces_vec, npts_tot * sizeof(double));
    hipMalloc(&dindexescolor, nindexes * sizeof(int));
    hipMalloc(&dnodal_forces_global, ncolor * neq * sizeof(double));

//    hipEventRecord(stop);
//    hipEventSynchronize(stop);
//    float milliseconds = 0;
//    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Allocate: " << milliseconds/1000 << std::endl;

}

void TPZSolveVector::FreeMemory() {
//    hipEvent_t start, stop;
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    hipEventRecord(start);

    hipFree(dglobal_solution);
    hipFree(dindexes);
    hipFree(dexpandsolution);
    hipFree(dresult);
    hipFree(dweight);
    hipFree(dsigma);
    hipFree(dnodal_forces_vec);
    hipFree(dindexescolor);
    hipFree(dnodal_forces_global);

    hipblasDestroy(handle_cublas);
    hipsparseDestroy(handle_cusparse);

//    hipEventRecord(stop);
//    hipEventSynchronize(stop);
//    float milliseconds = 0;
//    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "Free: " << milliseconds/1000 << std::endl;

}

void TPZSolveVector::cuSparseHandle() {
//    hipEvent_t start, stop;
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    hipEventRecord(start);

    hipsparseCreate (&handle_cusparse);

//    hipEventRecord(stop);
//    hipEventSynchronize(stop);
//    float milliseconds = 0;
//    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "cuSPARSE: " << milliseconds/1000 << std::endl;

}

void TPZSolveVector::cuBlasHandle() {
//    hipEvent_t start, stop;
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    hipEventRecord(start);

    hipblasCreate (&handle_cublas);

//    hipEventRecord(stop);
//    hipEventSynchronize(stop);
//    float milliseconds = 0;
//    hipEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "cuBLAS: " << milliseconds/1000 << std::endl;
}


void TPZSolveVector::Multiply(const TPZFMatrix<STATE> &global_solution, TPZFMatrix<STATE> &result) const {
    int64_t n_globalsol = fIndexes.size()/2; //o vetor de indices esta duplicado
    int64_t nelem = fRowSizes.size();
    int rows = fRowSizes[0];
    int cols = fColSizes[0];

    TPZFMatrix<REAL> expandsolution(2*n_globalsol,1); //vetor solucao duplicado

    cblas_dgthr(2*n_globalsol, global_solution, &expandsolution(0,0), &fIndexes[0]);

    result.Resize(2*n_globalsol,1);
    result.Zero();

    for (int i = 0; i < cols; i++) {
        cblas_dsbmv(CblasColMajor, CblasUpper, nelem * rows / 2, 0, 1., &fStorageVec[i * nelem * rows], 1, &expandsolution(i * nelem, 0), 1, 1., &result(0,0), 1);
        cblas_dsbmv(CblasColMajor, CblasUpper, nelem * rows / 2, 0, 1., &fStorageVec[i * nelem * rows + nelem * rows / 2], 1, &expandsolution(i * nelem, 0), 1, 1., &result(nelem * rows / 2,0), 1);

        cblas_dsbmv(CblasColMajor, CblasUpper, nelem * rows / 2, 0, 1., &fStorageVec[i * nelem * rows], 1, &expandsolution(i * nelem + n_globalsol, 0), 1, 1., &result(n_globalsol,0), 1);
        cblas_dsbmv(CblasColMajor, CblasUpper, nelem * rows / 2, 0, 1., &fStorageVec[i * nelem * rows + nelem * rows / 2], 1, &expandsolution(i * nelem + n_globalsol, 0), 1, 1., &result(n_globalsol + nelem * rows / 2,0), 1);

    }
}

void TPZSolveVector::MultiplyCUDA(const TPZFMatrix<STATE> &global_solution, TPZFMatrix<STATE> &result) const{
    int64_t n_globalsol = fIndexes.size()/2; //o vetor de indices esta duplicado
    int64_t nelem = fRowSizes.size();
    int rows = fRowSizes[0];
    int cols = fColSizes[0];

    hipMemcpy(dindexes, &fIndexes[0], 2*n_globalsol * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dglobal_solution, &global_solution[0], global_solution.Rows() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dstoragevec, &fStorageVec[0], nelem*fColSizes[0]*fRowSizes[0] * sizeof(double), hipMemcpyHostToDevice);

    hipsparseDgthr(handle_cusparse, 2*n_globalsol, dglobal_solution, &dexpandsolution[0], &dindexes[0], HIPSPARSE_INDEX_BASE_ZERO);

    result.Resize(2*n_globalsol,1);
    result.Zero();

    hipMemcpy(dresult, &result(0,0), 2*n_globalsol * sizeof(double), hipMemcpyHostToDevice);

    double alpha = 1.;
    double beta = 1.;

    for (int i = 0; i < cols; i++) {
        hipblasDsbmv(handle_cublas, HIPBLAS_FILL_MODE_LOWER, nelem * rows / 2, 0, &alpha, &dstoragevec[i * nelem * rows], 1, &dexpandsolution[i * nelem], 1, &beta, &dresult[0], 1);
        hipblasDsbmv(handle_cublas, HIPBLAS_FILL_MODE_LOWER, nelem * rows / 2, 0, &alpha, &dstoragevec[i * nelem * rows + nelem * rows / 2], 1, &dexpandsolution[i * nelem], 1, &beta, &dresult[nelem * rows / 2], 1);

        hipblasDsbmv(handle_cublas, HIPBLAS_FILL_MODE_LOWER, nelem * rows / 2, 0, &alpha, &dstoragevec[i * nelem * rows], 1, &dexpandsolution[i * nelem + n_globalsol], 1, &beta, &dresult[n_globalsol], 1);
        hipblasDsbmv(handle_cublas, HIPBLAS_FILL_MODE_LOWER, nelem * rows / 2, 0, &alpha, &dstoragevec[i * nelem * rows + nelem * rows / 2], 1, &dexpandsolution[i * nelem + n_globalsol], 1, &beta, &dresult[n_globalsol + nelem * rows / 2], 1);
    }

    hipMemcpy(&result(0, 0), dresult, 2 * n_globalsol * sizeof(double), hipMemcpyDeviceToHost);
}

void TPZSolveVector::TraditionalAssemble(TPZFMatrix<STATE> &nodal_forces_vec, TPZFMatrix<STATE> &nodal_forces_global) const {
    for (int64_t ir = 0; ir < fRow; ir++) {
        nodal_forces_global(fIndexes[ir], 0) += nodal_forces_vec(ir, 0);
    }
}

void TPZSolveVector::ColoringElements(TPZCompMesh *cmesh) const {
    int64_t nelem_c = cmesh->NElements();
    int64_t nconnects = cmesh->NConnects();
    TPZVec<int64_t> connects_vec(nconnects, 0);

    int64_t contcolor = 0;
    bool needstocontinue = true;

    while (needstocontinue) {
        needstocontinue = false;
        for (int64_t iel = 0; iel < nelem_c; iel++) {
            TPZCompEl *cel = cmesh->Element(iel);
            if (!cel || cel->Dimension() != cmesh->Dimension()) continue;

            if (fElemColor[iel] != -1) continue;
            TPZStack<int64_t> connectlist;
            cmesh->Element(iel)->BuildConnectList(connectlist);
            int64_t ncon = connectlist.size();

            int64_t icon;
            for (icon = 0; icon < ncon; icon++) {
                if (connects_vec[connectlist[icon]] != 0) break;
            }
            if (icon != ncon) {
                needstocontinue = true;
                continue;
            }
            fElemColor[iel] = contcolor;

            for (icon = 0; icon < ncon; icon++) {
                connects_vec[connectlist[icon]] = 1;
            }
        }
        contcolor++;
        connects_vec.Fill(0);
    }

    int64_t nelem = fRowSizes.size();
    int64_t neq = cmesh->NEquations();
    for (int64_t iel = 0; iel < nelem; iel++) {
        int64_t cols = fColSizes[iel];
        int64_t cont_cols = fColFirstIndex[iel];

        for (int64_t icols = 0; icols < cols; icols++) {
            fIndexesColor[cont_cols + icols] = fIndexes[cont_cols + icols] + fElemColor[iel] * neq;
            fIndexesColor[cont_cols + fRow / 2 + icols] =
                    fIndexes[cont_cols + fRow / 2 + icols] + fElemColor[iel] * neq;
        }
    }
}
