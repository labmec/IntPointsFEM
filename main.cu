#include "hip/hip_runtime.h"

#include <iostream>
#include <string.h>
#include <ctime>
#include <algorithm>
#include <iterator>

// Neopz
#include "pzgmesh.h"
#include "pzcmesh.h"
#include "pzgeoelbc.h"
#include "pzbndcond.h"
#include "pzanalysis.h"
#include "pzskylstrmatrix.h"
#include "pzstepsolver.h"
#include "pzinterpolationspace.h"
#include "TPZVTKGeoMesh.h"
#include "pzintel.h"
#include "tpzintpoints.h"
#include "TPZMatElasticity2D.h"
#include "TPZSSpStructMatrix.h"

#include "TPZSolveMatrix.h"
#include "TPZSolveVector.h"

#ifdef USING_TBB
#include "tbb/parallel_for_each.h"
#endif

TPZGeoMesh *geometry_2D(int nelem_x, int nelem_y, REAL len, int ndivide);

TPZCompMesh *cmesh_2D(TPZGeoMesh *gmesh, int pOrder);

TPZCompMesh *cmesh_mat_2D(TPZGeoMesh *gmesh, int pOrder);

void SolMatrix(TPZCompMesh *cmesh);

void SolVector(TPZCompMesh *cmesh);

int main(int argc, char *argv[]) {
        //// ------------------------ DATA INPUT ------------------------------
        //// NUMBER OF ELEMENTS IN X AND Y DIRECTIONS
        int nelem_x = atoi(argv[1]);
        int nelem_y = atoi(argv[1]);

        std::cout << "-------------------------------------------------" << std::endl;
        std::cout << "MESH SIZE: " << nelem_x << "x" << nelem_y << std::endl;

//// DOMAIN LENGTH
        REAL len = 1;

//// COMPUTATIONAL MESH ORDER
        int pOrder = 1;

//// SUBDIVISIONS OF THE ELEMENTS
        int ndivide = 0;

//// ENTER THE FILE NAME
        std::string namefile = "Elasticity_teste";
//// ------------------------------------------------------------------

//// Generating the geometry
        TPZGeoMesh *gmesh = geometry_2D(nelem_x, nelem_y, len, ndivide);
//std::ofstream vtk_file_00(namefile + ".vtk");
//TPZVTKGeoMesh::PrintGMeshVTK(gmesh, vtk_file_00);

//// Creating the computational mesh
        TPZCompMesh *cmesh = cmesh_2D(gmesh, pOrder);
//        TPZCompMesh *cmesh_d = cmesh_mat_2D(gmesh, pOrder);

//// Defining the analysis
        bool optimizeBandwidth = true;
        int n_threads = 16;
        TPZAnalysis an(cmesh, optimizeBandwidth);
//        TPZAnalysis an_d(cmesh_d, optimizeBandwidth);
#ifdef USING_MKL
        TPZSymetricSpStructMatrix strskyl(cmesh);
//        TPZSymetricSpStructMatrix strskyl_d(cmesh_d);
#else
        TPZSkylineStructMatrix strskyl(cmesh);
        TPZSkylineStructMatrix strskyl_d(cmesh_d);
#endif
        strskyl.SetNumThreads(n_threads);
//        strskyl_d.SetNumThreads(n_threads);
        an.SetStructuralMatrix(strskyl);
//        an_d.SetStructuralMatrix(strskyl_d);

//// Solve
        TPZStepSolver<STATE> *direct = new TPZStepSolver<STATE>;
        direct->SetDirect(ECholesky);
        an.SetSolver(*direct);
//        an_d.SetSolver(*direct);
        delete direct;
        an.Assemble();
        an.Solve();

//TPZFMatrix<REAL> sol = cmesh->Solution();
//std::ofstream file("solution.txt");
//for(int i = 0; i < sol.Rows(); i++){
//file << sol(i,0) << std::endl;
//}

// Computing global K
//        an_d.Assemble();
//        TPZFMatrix<STATE> res_d;
// Computing K u
 //       an_d.Solver().Matrix()->Multiply(an.Solution(), res_d);
//Print Rhs without boundary conditions
//res_d.Print("ku = ",std::cout,EMathematicaInput);

//// Post processing in Paraview
//        TPZManVector<std::string> scalarnames(2), vecnames(1);
//        scalarnames[0] = "SigmaX";
//        scalarnames[1] = "SigmaY";
//        vecnames[0] = "Displacement";
//        an.DefineGraphMesh(2, scalarnames, vecnames, namefile + "ElasticitySolutions.vtk");
//        an.PostProcess(0);

//        SolMatrix(cmesh);
        SolVector(cmesh);
    return 0;
}

TPZGeoMesh *geometry_2D(int nelem_x, int nelem_y, REAL len, int ndivide) {
// Creates the geometric mesh
    TPZGeoMesh *gmesh = new TPZGeoMesh();
    int dim = 2;
    gmesh->SetDimension(dim);

// Geometry definitions
    int nnodes_x = nelem_x + 1; //Number of elements in x direction
    int nnodes_y = nelem_y + 1; //Number of elements in x direction
    int64_t nelem = nelem_x * nelem_y; //Total number of elements

// Nodes initialization
// Enumeration: vertical order - from the below to the top, and from the left to the right
    TPZManVector<REAL> coord(3, 0.);
    int64_t id, index;
    for (int i = 0; i < nnodes_x; i++) {
        for (int j = 0; j < nnodes_y; j++) {
            id = i * nnodes_y + j;
            coord[0] = (i) * len / (nnodes_x - 1);
            coord[1] = (j) * len / (nnodes_y - 1);
            index = gmesh->NodeVec().AllocateNewElement();
            gmesh->NodeVec()[index] = TPZGeoNode(id, coord, *gmesh);
        }
    }

// Element connectivities
// Enumeration: vertical order - from the below to the top, and from the left to the right
    TPZManVector<int64_t> connect(4, 0);
    for (int i = 0; i < (nnodes_x - 1); i++) {
        for (int j = 0; j < (nnodes_y - 1); j++) {
            index = (i) * (nnodes_y - 1) + (j);
            connect[0] = (i) * nnodes_y + (j);
            connect[1] = connect[0] + (nnodes_y);
            connect[2] = connect[1] + 1;
            connect[3] = connect[0] + 1;
            gmesh->CreateGeoElement(EQuadrilateral, connect, 1, id); //Allocates and define the geometric element
        }
    }

// Generate neighborhood information
    gmesh->BuildConnectivity();

// Creating the boundary conditions
// Dirichlet
    for (int64_t i = 0; i < nelem_y; i++) {
        TPZGeoEl *gelem = gmesh->Element(i);
        TPZGeoElBC el_boundary(gelem, 7, -1); //Left side of the plane
    }
    for (int64_t i = 0; i < nelem_x; i++) {
        int64_t n = nelem_y * (i + 1) - 1;
        TPZGeoEl *gelem = gmesh->Element(n);
        TPZGeoElBC el_boundary(gelem, 6, -2); //Top side of the plane
    }

// Neumann
    for (int64_t i = nelem - nelem_y; i < nelem; i++) {
        TPZGeoEl *gelem = gmesh->Element(i);
        TPZGeoElBC el_boundary(gelem, 5, -4); //Right side of the plane - tension
    }
    for (int64_t i = 0; i < nelem_x; i++) {
        int64_t n = nelem_y * (i + 1) - (nelem_y);
        TPZGeoEl *gelem = gmesh->Element(n);
        TPZGeoElBC el_boundary(gelem, 4, -3); //Bottom side of the plane - tension
    }

// HP adaptativity
    if (ndivide != 0) {
        // Finding the elements which will be subdivided
        TPZGeoEl *gel; // Defining the element
        TPZVec<REAL> x(3, 0.); // Defining the coordinate at the end of the node
        x[0] = 0;
        x[1] = len;
        TPZVec<REAL> qsi(3, 0.); // Defining the parametric coordinate
        int64_t InitialElIndex = 0;
        int targetDim = 2;
        gel = gmesh->FindElement(x, qsi, InitialElIndex,
                                 targetDim); // Finding the element which is related to the coordinate
        int64_t elid = gel->Index(); // Atention: this procedure catchs the first element which is related to the coordinate

        TPZVec<TPZGeoEl *> subelindex;

        gel = gmesh->Element(elid);
        gel->Divide(subelindex);
        for (int i = 0; i < ndivide - 1; i++) {
            subelindex[3]->Divide(subelindex);
        }

    }
    return gmesh;
}

TPZCompMesh *cmesh_2D(TPZGeoMesh *gmesh, int pOrder) {

// Creating the computational mesh
    TPZCompMesh *cmesh = new TPZCompMesh(gmesh);
    cmesh->SetDefaultOrder(pOrder);

// Creating elasticity material
    TPZMatElasticity2D *mat = new TPZMatElasticity2D(1);
    mat->SetElasticParameters(200000000., 0.3);

// Setting the boundary conditions
    TPZMaterial *bcBottom, *bcRight, *bcTop, *bcLeft;
    TPZFMatrix<REAL> val1(2, 1, 0.);
    TPZFMatrix<REAL> val2(2, 1, 0.);

    bcLeft = mat->CreateBC(mat, -1, 7, val1, val2); // X displacement = 0
    bcTop = mat->CreateBC(mat, -2, 8, val1, val2); // Y displacement = 0

    val2(1, 0) = -1000000.;
    bcBottom = mat->CreateBC(mat, -3, 1, val1, val2); // Tension in y

    val2(0, 0) = 1000000.;
    val2(1, 0) = 0.0;
    bcRight = mat->CreateBC(mat, -4, 1, val1, val2); // Tension in x

    cmesh->InsertMaterialObject(mat);

    cmesh->InsertMaterialObject(bcBottom);
    cmesh->InsertMaterialObject(bcRight);
    cmesh->InsertMaterialObject(bcTop);
    cmesh->InsertMaterialObject(bcLeft);

    cmesh->SetAllCreateFunctionsContinuous();
    cmesh->AutoBuild();
    cmesh->AdjustBoundaryElements();
    cmesh->CleanUpUnconnectedNodes();

    return cmesh;
}

TPZCompMesh *cmesh_mat_2D(TPZGeoMesh *gmesh, int pOrder) {

    // Creating the computational mesh
    TPZCompMesh *cmesh = new TPZCompMesh(gmesh);
    cmesh->SetDefaultOrder(pOrder);

    // Creating elasticity material
    TPZMatElasticity2D *mat = new TPZMatElasticity2D(1);
    mat->SetElasticParameters(200000000., 0.3);
    cmesh->InsertMaterialObject(mat);

    cmesh->SetAllCreateFunctionsContinuous();
    cmesh->AutoBuild();
    return cmesh;
}

void SolVector(TPZCompMesh *cmesh) {

    int dim_mesh = (cmesh->Reference())->Dimension(); // Mesh dimension
    int64_t nelem_c = cmesh->NElements(); // Number of computational elements
    std::vector<int64_t> cel_indexes;

//// -------------------------------------------------------------------------------
//// NUMBER OF DOMAIN GEOMETRIC ELEMENTS
    for (int64_t i = 0; i < nelem_c; i++) {
        TPZCompEl *cel = cmesh->Element(i);
        if (!cel) continue;
        TPZGeoEl *gel = cmesh->Element(i)->Reference();
        if (!gel || gel->Dimension() != dim_mesh) continue;
        cel_indexes.push_back(cel->Index());
    }
//// -------------------------------------------------------------------------------

    if (cel_indexes.size() == 0) {
        DebugStop();
    }

//// ROWSIZES AND COLSIZES VECTORS--------------------------------------------------
    int64_t nelem = cel_indexes.size(); // Number of domain geometric elements
    TPZVec<int64_t> rowsizes(nelem);
    TPZVec<int64_t> colsizes(nelem);

    int64_t npts_tot = 0;
    int64_t nf_tot = 0;

    for (auto iel : cel_indexes) {
        //Verification
        TPZCompEl *cel = cmesh->Element(iel);

        //Integration rule
        TPZInterpolatedElement *cel_inter = dynamic_cast<TPZInterpolatedElement * >(cel);
        if (!cel_inter) DebugStop();
        TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());

        int64_t npts = int_rule->NPoints(); // number of integration points of the element
        int64_t dim = cel_inter->Dimension(); //dimension of the element
        int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element

        rowsizes[iel] = dim * npts;
        colsizes[iel] = nf;

        npts_tot += npts;
        nf_tot += nf;
    }

    TPZSolveVector *SolVec = new TPZSolveVector(dim_mesh * npts_tot, nf_tot, rowsizes, colsizes);
//// -------------------------------------------------------------------------------

//// DPHI MATRIX FOR EACH ELEMENT, WEIGHT AND INDEXES VECTORS-----------------------
    TPZFMatrix<REAL> elmatrix;
    TPZVec<REAL> weight(npts_tot);
    TPZManVector<MKL_INT> indexes(2*dim_mesh * nf_tot);
    int cont = 0;
    for (auto iel : cel_indexes) {
        int64_t cont1 = 0;
        int64_t cont2 = 0;
        //Verification
        TPZCompEl *cel = cmesh->Element(iel);

        //Integration rule
        TPZInterpolatedElement *cel_inter = dynamic_cast<TPZInterpolatedElement * >(cel);
        if (!cel_inter) DebugStop();
        TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());

        int64_t npts = int_rule->NPoints(); // number of integration points of the element
        int64_t dim = cel_inter->Dimension(); //dimension of the element
        int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element

        TPZMaterialData data;
        cel_inter->InitMaterialData(data);

        elmatrix.Resize(dim * npts, nf);
        for (int64_t inpts = 0; inpts < npts; inpts++) {
            TPZManVector<REAL> qsi(dim, 1);
            REAL w;
            int_rule->Point(inpts, qsi, w);
            cel_inter->ComputeRequiredData(data, qsi);
            weight[iel + nelem*inpts] = w * std::abs(data.detjac);

            TPZFMatrix<REAL> &dphix = data.dphix;
            for (int inf = 0; inf < nf; inf++) {
                for (int idim = 0; idim < dim; idim++)
                    elmatrix(inpts * dim + idim, inf) = dphix(idim, inf);
            }
        }
        SolVec->SetElementMatrix(iel, elmatrix);

        //Indexes vector
        int64_t ncon = cel->NConnects();
        for (int64_t icon = 0; icon < ncon; icon++) {
            int64_t id = cel->ConnectIndex(icon);
            TPZConnect &df = cmesh->ConnectVec()[id];
            int64_t conid = df.SequenceNumber();
            if (df.NElConnected() == 0 || conid < 0 || cmesh->Block().Size(conid) == 0) continue;
            else {
                int64_t pos = cmesh->Block().Position(conid);
                int64_t nsize = cmesh->Block().Size(conid);
                for (int64_t isize = 0; isize < nsize; isize++) {
                    if (isize % 2 == 0) {
                        indexes[cont1*nelem + cont] = pos + isize;
                        indexes[cont1*nelem + nf_tot + cont] = pos + isize; //para indices duplicados
                        cont1++;
                    } else {
                        indexes[cont2*nelem + 2*nf_tot + cont] = pos + isize; //2*nf_tot para indices duplicados
                        indexes[cont2*nelem + 3*nf_tot + cont] = pos + isize; //para indices duplicados
                        cont2++;
                    }
                }
            }
        }
        cont++;
    }
    SolVec->SetIndexes(indexes);
    SolVec->ColoringElements(cmesh);

    TPZFMatrix<REAL> coef_sol = cmesh->Solution();
    int neq = cmesh->NEquations();
    TPZFMatrix<REAL> nodal_forces_global1(neq, 1, 0.);
    TPZFMatrix<REAL> nodal_forces_global2(neq, 1, 0.);
    TPZFMatrix<REAL> nodal_forces_global3(neq, 1, 0.);
    TPZFMatrix<REAL> result;
    TPZFMatrix<REAL> sigma;
    TPZFMatrix<REAL> nodal_forces_vec;

#ifdef __HIPCC__
    std::cout << "\n\nSOLVING WITH GPU" << std::endl;
    SolVec->AllocateMemory(cmesh);
    SolVec->MultiplyCUDA(coef_sol,result);
    SolVec->ComputeSigmaCUDA(weight, result, sigma);    
    SolVec->MultiplyTransposeCUDA(sigma,nodal_forces_vec);
    SolVec->ColoredAssembleCUDA(nodal_forces_vec,nodal_forces_global3);
    SolVec->FreeMemory();

#endif

    std::cout << "\n\nSOLVING WITH CPU" << std::endl;
    SolVec->Multiply(coef_sol, result);
    SolVec->ComputeSigma(weight, result, sigma);
    SolVec->MultiplyTranspose(sigma,nodal_forces_vec);
    SolVec->ColoredAssemble(nodal_forces_vec,nodal_forces_global2);

    //Check result
    SolVec->TraditionalAssemble(nodal_forces_vec, nodal_forces_global1); // ok
    int rescpu = Norm(nodal_forces_global1 - nodal_forces_global2);
    if(rescpu == 0){
        std::cout << "\nAssemble done in the CPU is ok." << std::endl;
    } else {
        std::cout << "\nAssemble done in the CPU is not ok." << std::endl;
    }

#ifdef __HIPCC__
    int resgpu = Norm(nodal_forces_global1 - nodal_forces_global3);
    if(resgpu == 0){
        std::cout << "\nAssemble done in the GPU is ok." << std::endl;
    } else {
        std::cout << "\nAssemble done in the GPU is not ok." << std::endl;
    }
#endif
}

void SolMatrix(TPZCompMesh *cmesh) {

    int dim_mesh = (cmesh->Reference())->Dimension(); // Mesh dimension
    int64_t nelem_c = cmesh->NElements(); // Number of computational elements
    std::vector<int64_t> cel_indexes;

//// -------------------------------------------------------------------------------
//// NUMBER OF DOMAIN GEOMETRIC ELEMENTS
    for (int64_t i = 0; i < nelem_c; i++) {
        TPZCompEl *cel = cmesh->Element(i);
        if (!cel) continue;
        TPZGeoEl *gel = cmesh->Element(i)->Reference();
        if (!gel || gel->Dimension() != dim_mesh) continue;
        cel_indexes.push_back(cel->Index());
    }
//// -------------------------------------------------------------------------------

    if (cel_indexes.size() == 0) {
        DebugStop();
    }

//// ROWSIZES AND COLSIZES VECTORS--------------------------------------------------
    int64_t nelem = cel_indexes.size(); // Number of domain geometric elements
    TPZVec<int64_t> rowsizes(nelem);
    TPZVec<int64_t> colsizes(nelem);

    int64_t npts_tot = 0;
    int64_t nf_tot = 0;

    for (auto iel : cel_indexes) {
        //Verification
        TPZCompEl *cel = cmesh->Element(iel);

        //Integration rule
        TPZInterpolatedElement *cel_inter = dynamic_cast<TPZInterpolatedElement * >(cel);
        if (!cel_inter) DebugStop();
        TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());

        int64_t npts = int_rule->NPoints(); // number of integration points of the element
        int64_t dim = cel_inter->Dimension(); //dimension of the element
        int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element

        rowsizes[iel] = dim * npts;
        colsizes[iel] = nf;

        npts_tot += npts;
        nf_tot += nf;
    }

    TPZSolveMatrix *SolMat = new TPZSolveMatrix(dim_mesh * npts_tot, nf_tot, rowsizes, colsizes);
//// -------------------------------------------------------------------------------

//// DPHI MATRIX FOR EACH ELEMENT, WEIGHT AND INDEXES VECTORS-----------------------
    TPZFMatrix<REAL> elmatrix;
    TPZStack<REAL> weight;
    TPZManVector<MKL_INT> indexes(dim_mesh * nf_tot);

    int64_t cont1 = 0;
    int64_t cont2 = 0;

    for (auto iel : cel_indexes) {
        //Verification
        TPZCompEl *cel = cmesh->Element(iel);

        //Integration rule
        TPZInterpolatedElement *cel_inter = dynamic_cast<TPZInterpolatedElement * >(cel);
        if (!cel_inter) DebugStop();
        TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());

        int64_t npts = int_rule->NPoints(); // number of integration points of the element
        int64_t dim = cel_inter->Dimension(); //dimension of the element
        int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element

        TPZMaterialData data;
        cel_inter->InitMaterialData(data);

        elmatrix.Resize(dim * npts, nf);
        for (int64_t inpts = 0; inpts < npts; inpts++) {
            TPZManVector<REAL> qsi(dim, 1);
            REAL w;
            int_rule->Point(inpts, qsi, w);
            cel_inter->ComputeRequiredData(data, qsi);
            weight.Push(w * std::abs(data.detjac)); //weight = w * detjac

            TPZFMatrix<REAL> &dphix = data.dphix;
            for (int inf = 0; inf < nf; inf++) {
                for (int idim = 0; idim < dim; idim++)
                    elmatrix(inpts * dim + idim, inf) = dphix(idim, inf);
            }
        }
        SolMat->SetElementMatrix(iel, elmatrix);

        //Indexes vector
        int64_t ncon = cel->NConnects();
        for (int64_t icon = 0; icon < ncon; icon++) {
            int64_t id = cel->ConnectIndex(icon);
            TPZConnect &df = cmesh->ConnectVec()[id];
            int64_t conid = df.SequenceNumber();
            if (df.NElConnected() == 0 || conid < 0 || cmesh->Block().Size(conid) == 0) continue;
            else {
                int64_t pos = cmesh->Block().Position(conid);
                int64_t nsize = cmesh->Block().Size(conid);
                for (int64_t isize = 0; isize < nsize; isize++) {
                    if (isize % 2 == 0) {
                        indexes[cont1] = pos + isize;
                        cont1++;
                    } else {
                        indexes[cont2 + nf_tot] = pos + isize;
                        cont2++;
                    }
                }
            }
        }
    }
    SolMat->SetIndexes(indexes);
    SolMat->ColoringElements(cmesh);

    //TPZFMatrix<REAL> coef_sol = cmesh->Solution();
    int neq = cmesh->NEquations();
    TPZFMatrix<REAL> nodal_forces_global1(neq, 1, 0.);
    TPZFMatrix<REAL> nodal_forces_global2(neq, 1, 0.);
    TPZFMatrix<REAL> nodal_forces_global3(neq, 1, 0.);
    TPZFMatrix<REAL> result;
    TPZFMatrix<REAL> sigma;
    TPZFMatrix<REAL> nodal_forces_vec;

        std::ifstream input("/home/nataliarvboas/TesteCPU/IntegrationPointExperiments/IntegrationPointExperiments/solution.txt");
        if(!input) {
            std::cout  << "Failed to open file ";
        }

        TPZFMatrix<REAL> coef_sol(cmesh->NEquations(),1);
        int k = 0;
        double val;
        while (input >> val) {
            coef_sol(k,0) = val;
            k++;
        }

    #ifdef __HIPCC__
    std::cout << "\n\nSOLVING WITH GPU" << std::endl;
    SolMat->AllocateMemory(cmesh);
    SolMat->MultiplyCUDA(coef_sol, result);
    SolMat->ComputeSigmaCUDA(weight, result, sigma);
    SolMat->MultiplyTransposeCUDA(sigma, nodal_forces_vec);
    SolMat->ColoredAssembleCUDA(nodal_forces_vec, nodal_forces_global3);
    SolMat->FreeMemory();
    #endif

    std::cout << "\n\nSOLVING WITH CPU" << std::endl;
    SolMat->Multiply(coef_sol, result);
    SolMat->ComputeSigma(weight, result, sigma);
    SolMat->MultiplyTranspose(sigma, nodal_forces_vec);
    SolMat->ColoredAssemble(nodal_forces_vec, nodal_forces_global2);

    //Check Result
    SolMat->TraditionalAssemble(nodal_forces_vec, nodal_forces_global1); // ok
    int rescpu = Norm(nodal_forces_global1 - nodal_forces_global2);
    if(rescpu == 0){
        std::cout << "\nAssemble done in the CPU is ok." << std::endl;
    } else {
        std::cout << "\nAssemble done in the CPU is not ok." << std::endl;
    }

    #ifdef __HIPCC__
    int resgpu = Norm(nodal_forces_global1 - nodal_forces_global3);
    if(resgpu == 0){
        std::cout << "\nAssemble done in the GPU is ok." << std::endl;
    } else {
        std::cout << "\nAssemble done in the GPU is not ok." << std::endl;
    }
    #endif
}

