#include "hip/hip_runtime.h"

#include <iostream>
#include <string.h>
#include <ctime>
#include <algorithm>
#include <iterator>

// Neopz
#include "pzgmesh.h"
#include "pzcmesh.h"
#include "pzgeoelbc.h"
#include "pzbndcond.h"
#include "pzanalysis.h"
#include "pzskylstrmatrix.h"
#include "pzstepsolver.h"
#include "pzinterpolationspace.h"
#include "TPZVTKGeoMesh.h"
#include "pzintel.h"
#include "tpzintpoints.h"
#include "TPZMatElasticity2D.h"
#include "TPZSSpStructMatrix.h"

#include "TPZMatElastoPlastic2D.h"
#include "TPZMatElastoPlastic.h"
#include "TPZElastoPlasticMem.h"
#include "TPZElasticCriterion.h"
#include "TPZPlasticStepPV.h"
#include "TPZSandlerExtended.h"
#include "TPZYCMohrCoulombPV.h"
#include "pzstepsolver.h"

#include "TPZSolveMatrix.h"
//#include "TPZSolveVector.h"

#ifdef USING_TBB
#include "tbb/parallel_for_each.h"
#endif

TPZGeoMesh *Geometry2D(int nelem_x, int nelem_y, REAL len, int ndivide);

TPZCompMesh *CmeshElasticity(TPZGeoMesh *gmesh, int pOrder);
TPZCompMesh *CmeshElasticityNoBoundary(TPZGeoMesh *gmesh, int pOrder);

TPZCompMesh *CmeshElastoplasticity(TPZGeoMesh *gmesh, int pOrder);
TPZCompMesh *CmeshElastoplasticityNoBoundary(TPZGeoMesh *gmesh, int pOrder);

void SolMatrix(TPZFMatrix<REAL> residual, TPZCompMesh *cmesh);
void SolVector(TPZFMatrix<REAL> residual, TPZCompMesh *cmesh);

TPZFMatrix<REAL>  Residual(TPZCompMesh *cmesh, TPZCompMesh *cmesh_noboundary);

int main(int argc, char *argv[]) {
//// ------------------------ DATA INPUT ------------------------------
    int nelem_x = atoi(argv[1]); // Number of elements in x direction
    int nelem_y = atoi(argv[1]); // Number of elements in y direction
    REAL len = 1; // Domain length
    int pOrder = 2; // Computational mesh order
    int ndivide = 0; // Subdivision of elements

// Generates the geometry
    TPZGeoMesh *gmesh = Geometry2D(nelem_x, nelem_y, len, ndivide);
// Creates the computational mesh
    TPZCompMesh *cmesh = CmeshElasticity(gmesh, pOrder);
    TPZCompMesh *cmesh_noboundary = CmeshElasticityNoBoundary(gmesh, pOrder);

// Defines the analysis
    bool optimizeBandwidth = true;
    int n_threads = 16;
    TPZAnalysis an(cmesh, optimizeBandwidth);
    TPZSymetricSpStructMatrix strskyl(cmesh);
    strskyl.SetNumThreads(n_threads);
    an.SetStructuralMatrix(strskyl);

// Solve
    TPZStepSolver<STATE> step;
    step.SetDirect(ELDLt);
    an.SetSolver(step);
    an.Assemble();
//    an.Solver().Matrix()->Print("K = ", std::cout, EMathematicaInput);
    an.Solve();
//    an.Solution().Print("U = ", std::cout, EMathematicaInput);


// Post process
//    TPZManVector<std::string> scalarnames(0), vecnames(1);
//    scalarnames[0] = "SigmaX";
//    scalarnames[1] = "SigmaY";
//    scalarnames[0] = "StressI1";
//    vecnames[0] = "Displacement";
//    std::string namefile = "Elasticity_teste";
//    an.DefineGraphMesh(2, scalarnames, vecnames, namefile + "ElasticitySolutions.vtk");
//    an.PostProcess(0,2);

// Calculates residual without boundary conditions
    TPZFMatrix<REAL> residual = Residual(cmesh, cmesh_noboundary);

// Calculates residual using matrix operations and check if the result is ok
    SolMatrix(residual, cmesh);
//    SolVector(residual, cmesh);
    return 0;
}

TPZGeoMesh *Geometry2D(int nelem_x, int nelem_y, REAL len, int ndivide) {
// Creates the geometric mesh
    TPZGeoMesh *gmesh = new TPZGeoMesh();
    int dim = 2;
    gmesh->SetDimension(dim);

// Geometry definitions
    int nnodes_x = nelem_x + 1; //Number of nodes in x direction
    int nnodes_y = nelem_y + 1; //Number of nodes in x direction
    int64_t nelem = nelem_x * nelem_y; //Total number of elements

// Nodes initialization
// Enumeration: vertical order - from the below to the top, and from the left to the right
    TPZManVector<REAL> coord(3, 0.);
    int64_t id, index;
    for (int i = 0; i < nnodes_x; i++) {
        for (int j = 0; j < nnodes_y; j++) {
            id = i * nnodes_y + j;
            coord[0] = (i) * len / (nnodes_x - 1);
            coord[1] = (j) * len / (nnodes_y - 1);
            index = gmesh->NodeVec().AllocateNewElement();
            gmesh->NodeVec()[index] = TPZGeoNode(id, coord, *gmesh);
        }
    }

// Element connectivities
// Enumeration: vertical order - from the below to the top, and from the left to the right
    TPZManVector<int64_t> connect(4, 0);
    for (int i = 0; i < (nnodes_x - 1); i++) {
        for (int j = 0; j < (nnodes_y - 1); j++) {
            index = (i) * (nnodes_y - 1) + (j);
            connect[0] = (i) * nnodes_y + (j);
            connect[1] = connect[0] + (nnodes_y);
            connect[2] = connect[1] + 1;
            connect[3] = connect[0] + 1;
            gmesh->CreateGeoElement(EQuadrilateral, connect, 1, id); //Allocates and define the geometric element
        }
    }

// Generates neighborhood information
    gmesh->BuildConnectivity();

// Creates the boundary conditions
// Dirichlet
    for (int64_t i = 0; i < nelem_y; i++) {
        TPZGeoEl *gelem = gmesh->Element(i);
        TPZGeoElBC el_boundary(gelem, 7, -1); //Left side of the plane
    }
    for (int64_t i = 0; i < nelem_x; i++) {
        int64_t n = nelem_y * (i + 1) - 1;
        TPZGeoEl *gelem = gmesh->Element(n);
        TPZGeoElBC el_boundary(gelem, 6, -2); //Top side of the plane
    }

// Neumann
    for (int64_t i = 0; i < nelem_x; i++) {
        int64_t n = nelem_y * (i + 1) - (nelem_y);
        TPZGeoEl *gelem = gmesh->Element(n);
        TPZGeoElBC el_boundary(gelem, 4, -3); //Bottom side of the plane - tension
    }
    for (int64_t i = nelem - nelem_y; i < nelem; i++) {
        TPZGeoEl *gelem = gmesh->Element(i);
        TPZGeoElBC el_boundary(gelem, 5, -4); //Right side of the plane - tension
    }

// HP adaptativity
    if (ndivide != 0) {
        // Finding the elements which will be subdivided
        TPZGeoEl *gel; // Defining the element
        TPZVec<REAL> x(3, 0.); // Defining the coordinate at the end of the node
        x[0] = 0;
        x[1] = len;
        TPZVec<REAL> qsi(3, 0.); // Defining the parametric coordinate
        int64_t InitialElIndex = 0;
        int targetDim = 2;
        gel = gmesh->FindElement(x, qsi, InitialElIndex,
                                 targetDim); // Finding the element which is related to the coordinate
        int64_t elid = gel->Index(); // Atention: this procedure catchs the first element which is related to the coordinate

        TPZVec<TPZGeoEl *> subelindex;

        gel = gmesh->Element(elid);
        gel->Divide(subelindex);
        for (int i = 0; i < ndivide - 1; i++) {
            subelindex[3]->Divide(subelindex);
        }

    }
    return gmesh;
}

TPZCompMesh *CmeshElasticity(TPZGeoMesh *gmesh, int pOrder) {

    // Creating the computational mesh
    TPZCompMesh *cmesh = new TPZCompMesh(gmesh);
    cmesh->SetDefaultOrder(pOrder);

// Creating elasticity material
    TPZMatElasticity2D *mat = new TPZMatElasticity2D(1);
    mat->SetElasticParameters(200000000., 0.3);
    mat->SetPlaneStrain();

// Setting the boundary conditions
    TPZMaterial *bcBottom, *bcRight, *bcTop, *bcLeft;
    TPZFMatrix<REAL> val1(2, 1, 0.);
    TPZFMatrix<REAL> val2(2, 1, 0.);

    bcLeft = mat->CreateBC(mat, -1, 7, val1, val2); // X displacement = 0
    bcTop = mat->CreateBC(mat, -2, 8, val1, val2); // Y displacement = 0

    val2(1, 0) = -1000000.;
    bcBottom = mat->CreateBC(mat, -3, 1, val1, val2); // Tension in y

    val2(0, 0) = 1000000.;
    val2(1, 0) = 0.0;
    bcRight = mat->CreateBC(mat, -4, 1, val1, val2); // Tension in x

    cmesh->InsertMaterialObject(mat);

    cmesh->InsertMaterialObject(bcBottom);
    cmesh->InsertMaterialObject(bcRight);
    cmesh->InsertMaterialObject(bcTop);
    cmesh->InsertMaterialObject(bcLeft);

    cmesh->SetAllCreateFunctionsContinuous();
    cmesh->AutoBuild();
    cmesh->AdjustBoundaryElements();
    cmesh->CleanUpUnconnectedNodes();

    return cmesh;


//// Creates the computational mesh
//    TPZCompMesh *cmesh = new TPZCompMesh(gmesh);
//    cmesh->SetDefaultOrder(pOrder);
//
//// Creates elastic material
//    TPZMatElasticity2D *material = new TPZMatElasticity2D(1);
//    material->SetElasticParameters(200000., 0.3);
//    material->SetPlaneStrain();
//
//// Set the boundary conditions
//    TPZMaterial *bcBottom, *bcRight, *bcTop, *bcLeft;
//    TPZFMatrix<REAL> val1(2, 2), val2(2, 2);
//
//    val2(0, 0) = 0;
//    val2(1, 0) = 0;
//    bcLeft = material->CreateBC(material, -1, 0, val1, val2); // X displacement = 0
//
//    val2(0,0) = 0;
//    val2(1,0) = 0;
//    bcTop = material->CreateBC(material, -2, 0, val1, val2); // Y displacement = 0
//
//    val2(0, 0) = 0.0;
//    val2(1, 0) = -1000.;
//    bcBottom = material->CreateBC(material, -3, 1, val1, val2); // Tension in y
//
//    val2(0, 0) = 0.0;
//    val2(1, 0) = 0.0;
//    bcRight = material->CreateBC(material, -4, 0, val1, val2); // Tension in x
//
//    cmesh->InsertMaterialObject(material);
//    cmesh->InsertMaterialObject(bcBottom);
//    cmesh->InsertMaterialObject(bcRight);
//    cmesh->InsertMaterialObject(bcTop);
//    cmesh->InsertMaterialObject(bcLeft);
//
//    cmesh->SetAllCreateFunctionsContinuous();
//    cmesh->AutoBuild();
//    cmesh->AdjustBoundaryElements();
//    cmesh->CleanUpUnconnectedNodes();
//
//    return cmesh;
}

TPZCompMesh *CmeshElasticityNoBoundary(TPZGeoMesh *gmesh, int pOrder) {

    // Creating the computational mesh
    TPZCompMesh *cmesh = new TPZCompMesh(gmesh);
    cmesh->SetDefaultOrder(pOrder);

    // Creating elasticity material
    TPZMatElasticity2D *mat = new TPZMatElasticity2D(1);
    mat->SetElasticParameters(200000000., 0.3);
    mat->SetPlaneStrain();
    cmesh->InsertMaterialObject(mat);

    cmesh->SetAllCreateFunctionsContinuous();
    cmesh->AutoBuild();
    return cmesh;

//    // Creating the computational mesh
//    TPZCompMesh *cmesh = new TPZCompMesh(gmesh);
//    cmesh->SetDefaultOrder(pOrder);
//
//    // Creating elasticity material
//    TPZMatElasticity2D *mat = new TPZMatElasticity2D(1);
//    mat->SetElasticParameters(200000000., 0.3);
//    cmesh->InsertMaterialObject(mat);
//
//    cmesh->SetAllCreateFunctionsContinuous();
//    cmesh->AutoBuild();
//    return cmesh;
}

TPZCompMesh *CmeshElastoplasticity(TPZGeoMesh * gmesh, int p_order) {

// Creates the computational mesh
    TPZCompMesh * cmesh = new TPZCompMesh(gmesh);
    cmesh->SetDefaultOrder(p_order);

// Mohr Coulomb data
    REAL mc_cohesion    = 100.0;
    REAL mc_phi         = (50.0*M_PI/180);
    REAL mc_psi         = mc_phi;

// ElastoPlastic Material using Mohr Coulomb
// Elastic predictor
    TPZElasticResponse ER;
    REAL nu = 0.3;
    REAL E = 200000;

    TPZPlasticStepPV<TPZYCMohrCoulombPV, TPZElasticResponse> LEMC;
    ER.SetUp(E, nu);
    LEMC.SetElasticResponse(ER);
    LEMC.fYC.SetUp(mc_phi, mc_psi, mc_cohesion, ER);
    int PlaneStrain = 1;
    int matid = 1;

// Creates elastoplatic material
    TPZMatElastoPlastic2D < TPZPlasticStepPV<TPZYCMohrCoulombPV, TPZElasticResponse>, TPZElastoPlasticMem > * material = new TPZMatElastoPlastic2D < TPZPlasticStepPV<TPZYCMohrCoulombPV, TPZElasticResponse>, TPZElastoPlasticMem >(matid,PlaneStrain);
    material->SetPlasticityModel(LEMC);

// Set the boundary conditions
    TPZMaterial *bcBottom, *bcRight, *bcTop, *bcLeft;
    TPZFMatrix<REAL> val1(2,2), val2(2,2);

    val2(0,0) = 0;
    val2(1,0) = 0;
    bcLeft = material->CreateBC(material, -1, 0, val1, val2);

    val2(0,0) = 0;
    val2(1,0) = 0;
    bcTop = material->CreateBC(material, -2, 0, val1, val2);

    val2(0,0) = 0;
    val2(1,0) = -1000;
    bcBottom = material->CreateBC(material, -3, 1, val1, val2);

    val2(0,0) = 0;
    val2(1,0) = 0;
    bcRight = material->CreateBC(material, -4, 0, val1, val2);

    cmesh->InsertMaterialObject(material);
    cmesh->InsertMaterialObject(bcBottom);
    cmesh->InsertMaterialObject(bcTop);
    cmesh->InsertMaterialObject(bcLeft);
    cmesh->InsertMaterialObject(bcRight);

    cmesh->SetAllCreateFunctionsContinuousWithMem();
    cmesh->AutoBuild();

    return cmesh;
}

TPZCompMesh *CmeshElastoplasticityNoBoundary(TPZGeoMesh * gmesh, int p_order) {

// Creates the computational mesh
    TPZCompMesh * cmesh = new TPZCompMesh(gmesh);
    cmesh->SetDefaultOrder(p_order);

// Mohr Coulomb data
    REAL mc_cohesion    = 10.0;
    REAL mc_phi         = (20.0*M_PI/180);
    REAL mc_psi         = mc_phi;

// ElastoPlastic Material using Mohr Coulomb
// Elastic predictor
    TPZElasticResponse ER;
    REAL G = 400*mc_cohesion;
    REAL nu = 0.3;
    REAL E = 2.0*G*(1+nu);

    TPZPlasticStepPV<TPZYCMohrCoulombPV, TPZElasticResponse> LEMC;
    ER.SetUp(E, nu);
    LEMC.SetElasticResponse(ER);
    LEMC.fYC.SetUp(mc_phi, mc_psi, mc_cohesion, ER);
    int PlaneStrain = 1;
    int matid = 1;

// Creates elastoplatic material
    TPZMatElastoPlastic2D < TPZPlasticStepPV<TPZYCMohrCoulombPV, TPZElasticResponse>, TPZElastoPlasticMem > * material = new TPZMatElastoPlastic2D < TPZPlasticStepPV<TPZYCMohrCoulombPV, TPZElasticResponse>, TPZElastoPlasticMem >(matid,PlaneStrain);
    material->SetPlasticityModel(LEMC);

    cmesh->InsertMaterialObject(material);
    cmesh->SetAllCreateFunctionsContinuousWithMem();
    cmesh->AutoBuild();

    return cmesh;
}

void SolVector(TPZFMatrix<REAL> residual, TPZCompMesh *cmesh) {

//    int dim_mesh = (cmesh->Reference())->Dimension(); // Mesh dimension
//    int64_t nelem_c = cmesh->NElements(); // Number of computational elements
//    std::vector<int64_t> cel_indexes;
//
//// Number of domain geometric elements
//    for (int64_t i = 0; i < nelem_c; i++) {
//        TPZCompEl *cel = cmesh->Element(i);
//        if (!cel) continue;
//        TPZGeoEl *gel = cmesh->Element(i)->Reference();
//        if (!gel || gel->Dimension() != dim_mesh) continue;
//        cel_indexes.push_back(cel->Index());
//    }
//
//    if (cel_indexes.size() == 0) {
//        DebugStop();
//    }
//
//// RowSizes and ColSizes vectors
//    int64_t nelem = cel_indexes.size();
//    TPZVec<int64_t> rowsizes(nelem);
//    TPZVec<int64_t> colsizes(nelem);
//
//    int64_t npts_tot = 0;
//    int64_t nf_tot = 0;
//
//    for (auto iel : cel_indexes) {
//        //Verification
//        TPZCompEl *cel = cmesh->Element(iel);
//
//        //Integration rule
//        TPZInterpolatedElement *cel_inter = dynamic_cast<TPZInterpolatedElement * >(cel);
//        if (!cel_inter) DebugStop();
//        TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());
//
//        int64_t npts = int_rule->NPoints(); // number of integration points of the element
//        int64_t dim = cel_inter->Dimension(); //dimension of the element
//        int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element
//
//        rowsizes[iel] = dim * npts;
//        colsizes[iel] = nf;
//
//        npts_tot += npts;
//        nf_tot += nf;
//    }
//
//    TPZSolveVector *SolVec = new TPZSolveVector(dim_mesh * npts_tot, nf_tot, rowsizes, colsizes);
//
//// Dphi matrix, weight and indexes vectors
//    TPZFMatrix<REAL> elmatrix;
//    TPZVec<REAL> weight(npts_tot);
//    TPZManVector<MKL_INT> indexes(dim_mesh * nf_tot);
//    int cont = 0;
//    for (auto iel : cel_indexes) {
//        int64_t cont1 = 0;
//        int64_t cont2 = 0;
//        //Verification
//        TPZCompEl *cel = cmesh->Element(iel);
//
//        //Integration rule
//        TPZInterpolatedElement *cel_inter = dynamic_cast<TPZInterpolatedElement * >(cel);
//        if (!cel_inter) DebugStop();
//        TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());
//
//        int64_t npts = int_rule->NPoints(); // number of integration points of the element
//        int64_t dim = cel_inter->Dimension(); //dimension of the element
//        int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element
//
//        TPZMaterialData data;
//        cel_inter->InitMaterialData(data);
//
//        elmatrix.Resize(dim * npts, nf);
//        for (int64_t inpts = 0; inpts < npts; inpts++) {
//            TPZManVector<REAL> qsi(dim, 1);
//            REAL w;
//            int_rule->Point(inpts, qsi, w);
//            cel_inter->ComputeRequiredData(data, qsi);
//            weight[iel + nelem*inpts] = w * std::abs(data.detjac);
//
//            TPZFMatrix<REAL> &dphix = data.dphix;
//            for (int inf = 0; inf < nf; inf++) {
//                for (int idim = 0; idim < dim; idim++)
//                    elmatrix(inpts * dim + idim, inf) = dphix(idim, inf);
//            }
//        }
//        SolVec->SetElementMatrix(iel, elmatrix);
//
//        int64_t ncon = cel->NConnects();
//        for (int64_t icon = 0; icon < ncon; icon++) {
//            int64_t id = cel->ConnectIndex(icon);
//            TPZConnect &df = cmesh->ConnectVec()[id];
//            int64_t conid = df.SequenceNumber();
//            if (df.NElConnected() == 0 || conid < 0 || cmesh->Block().Size(conid) == 0) continue;
//            else {
//                int64_t pos = cmesh->Block().Position(conid);
//                int64_t nsize = cmesh->Block().Size(conid);
//                for (int64_t isize = 0; isize < nsize; isize++) {
//                    if (isize % 2 == 0) {
//                        indexes[cont1*nelem + cont] = pos + isize;
//                        cont1++;
//                    } else {
//                        indexes[cont2*nelem + nf_tot + cont] = pos + isize;
//                        cont2++;
//                    }
//                }
//            }
//        }
//        cont++;
//    }
//    SolVec->SetIndexes(indexes);
//    SolVec->ColoringElements(cmesh);
//
//    TPZFMatrix<REAL> coef_sol = cmesh->Solution();
//    int neq = cmesh->NEquations();
//
//    TPZFMatrix<REAL> nodal_forces_global1(neq, 1, 0.);
//    TPZFMatrix<REAL> nodal_forces_global2(neq, 1, 0.);
//    TPZFMatrix<REAL> result;
//    TPZFMatrix<REAL> sigma;
//    TPZFMatrix<REAL> nodal_forces_vec;
//
//#ifdef __HIPCC__
//    std::cout << "\n\nSOLVING WITH GPU" << std::endl;
//    SolVec->AllocateMemory(cmesh);
//    SolVec->MultiplyCUDA(coef_sol,result);
//    SolVec->ComputeSigmaCUDA(weight, result, sigma);
//    SolVec->MultiplyTransposeCUDA(sigma,nodal_forces_vec);
//    SolVec->ColoredAssembleCUDA(nodal_forces_vec,nodal_forces_global1);
//    SolVec->FreeMemory();
//
//#endif
//
//    std::cout << "\n\nSOLVING WITH CPU" << std::endl;
//    SolVec->Multiply(coef_sol, result);
//    SolVec->ComputeSigma(weight, result, sigma);
//    SolVec->MultiplyTranspose(sigma,nodal_forces_vec);
//    SolVec->ColoredAssemble(nodal_forces_vec,nodal_forces_global2);
//
//    //Check the result
//    int rescpu = Norm(nodal_forces_global2 - residual);
//    if(rescpu == 0){
//        std::cout << "\nAssemble done in the CPU is ok." << std::endl;
//    } else {
//        std::cout << "\nAssemble done in the CPU is not ok." << std::endl;
//    }
//
//#ifdef __HIPCC__
//    int resgpu = Norm(nodal_forces_global1 - residual);
//    if(resgpu == 0){
//        std::cout << "\nAssemble done in the GPU is ok." << std::endl;
//    } else {
//        std::cout << "\nAssemble done in the GPU is not ok." << std::endl;
//    }
//#endif
}

void SolMatrix(TPZFMatrix<REAL> residual, TPZCompMesh *cmesh) {

    int dim_mesh = (cmesh->Reference())->Dimension(); // Mesh dimension
    int64_t nelem_c = cmesh->NElements(); // Number of computational elements
    std::vector<int64_t> cel_indexes;

// Number of domain geometric elements
    for (int64_t i = 0; i < nelem_c; i++) {
        TPZCompEl *cel = cmesh->Element(i);
        if (!cel) continue;
        TPZGeoEl *gel = cmesh->Element(i)->Reference();
        if (!gel || gel->Dimension() != dim_mesh) continue;
        cel_indexes.push_back(cel->Index());
    }

    if (cel_indexes.size() == 0) {
        DebugStop();
    }

// RowSizes and ColSizes vectors
    int64_t nelem = cel_indexes.size();
    TPZVec<int> rowsizes(nelem);
    TPZVec<int> colsizes(nelem);

    int64_t npts_tot = 0;
    int64_t nf_tot = 0;

    for (auto iel : cel_indexes) {
        //Verification
        TPZCompEl *cel = cmesh->Element(iel);

        //Integration rule
        TPZInterpolatedElement *cel_inter = dynamic_cast<TPZInterpolatedElement * >(cel);
        if (!cel_inter) DebugStop();
        TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());

        int64_t npts = int_rule->NPoints(); // number of integration points of the element
        int64_t dim = cel_inter->Dimension(); //dimension of the element
        int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element

        rowsizes[iel] = dim * npts;
        colsizes[iel] = nf;

        npts_tot += npts;
        nf_tot += nf;
    }

    TPZSolveMatrix *SolMat = new TPZSolveMatrix(dim_mesh * npts_tot, nf_tot, rowsizes, colsizes);

// Dphi matrix, weight and indexes vectors
    TPZFMatrix<REAL> elmatrix;
    TPZStack<REAL> weight;
    TPZManVector<MKL_INT> indexes(dim_mesh * nf_tot);

    int64_t cont1 = 0;
    int64_t cont2 = 0;

    for (auto iel : cel_indexes) {
        //Verification
        TPZCompEl *cel = cmesh->Element(iel);

        //Integration rule
        TPZInterpolatedElement *cel_inter = dynamic_cast<TPZInterpolatedElement * >(cel);
        if (!cel_inter) DebugStop();
        TPZIntPoints *int_rule = &(cel_inter->GetIntegrationRule());

        int64_t npts = int_rule->NPoints(); // number of integration points of the element
        int64_t dim = cel_inter->Dimension(); //dimension of the element
        int64_t nf = cel_inter->NShapeF(); // number of shape functions of the element

        TPZMaterialData data;
        cel_inter->InitMaterialData(data);

        elmatrix.Resize(dim * npts, nf);
        for (int64_t inpts = 0; inpts < npts; inpts++) {
            TPZManVector<REAL> qsi(dim, 1);
            REAL w;
            int_rule->Point(inpts, qsi, w);
            cel_inter->ComputeRequiredData(data, qsi);
            weight.Push(w * std::abs(data.detjac)); //weight = w * detjac

            TPZFMatrix<REAL> &dphix = data.dphix;
            for (int inf = 0; inf < nf; inf++) {
                for (int idim = 0; idim < dim; idim++)
                    elmatrix(inpts * dim + idim, inf) = dphix(idim, inf);
            }
        }
        SolMat->SetElementMatrix(iel, elmatrix);

        //Indexes vector
        int64_t ncon = cel->NConnects();
        for (int64_t icon = 0; icon < ncon; icon++) {
            int64_t id = cel->ConnectIndex(icon);
            TPZConnect &df = cmesh->ConnectVec()[id];
            int64_t conid = df.SequenceNumber();
            if (df.NElConnected() == 0 || conid < 0 || cmesh->Block().Size(conid) == 0) continue;
            else {
                int64_t pos = cmesh->Block().Position(conid);
                int64_t nsize = cmesh->Block().Size(conid);
                for (int64_t isize = 0; isize < nsize; isize++) {
                    if (isize % 2 == 0) {
                        indexes[cont1] = pos + isize;
                        cont1++;
                    } else {
                        indexes[cont2 + nf_tot] = pos + isize;
                        cont2++;
                    }
                }
            }
        }
    }
    SolMat->SetIndexes(indexes);
//    SolMat->ColoringElements(cmesh);

    TPZFMatrix<REAL> coef_sol = cmesh->Solution();
    int neq = cmesh->NEquations();

    TPZFMatrix<REAL> nodal_forces_global1(neq, 1, 0.);
    TPZFMatrix<REAL> nodal_forces_global2(neq, 1, 0.);
    TPZFMatrix<REAL> result;
    TPZFMatrix<REAL> sigma;
    TPZFMatrix<REAL> nodal_forces_vec;


    #ifdef __HIPCC__
    std::cout << "\n\nSOLVING WITH GPU" << std::endl;
    SolMat->AllocateMemory(cmesh);
    SolMat->MultiplyInThreadsCUDA(coef_sol, result);
//    SolMat->MultiplyCUDA(coef_sol, result);
//    SolMat->ComputeSigmaCUDA(weight, result, sigma);
//    SolMat->MultiplyTransposeCUDA(sigma, nodal_forces_vec);
//    SolMat->ColoredAssembleCUDA(nodal_forces_vec, nodal_forces_global1);
    SolMat->FreeMemory();
    #endif

//    std::cout << "\n\nSOLVING WITH CPU" << std::endl;
////    SolMat->MultiplyInThreads(coef_sol, result);
//    SolMat->Multiply(coef_sol, result);
//    SolMat->ComputeSigma(weight, result, sigma);
//    SolMat->MultiplyTranspose(sigma, nodal_forces_vec);
//    SolMat->ColoredAssemble(nodal_forces_vec, nodal_forces_global2);
//
//    //Check the result
//    int rescpu = Norm(nodal_forces_global2 - residual);
//    if(rescpu == 0){
//        std::cout << "\nAssemble done in the CPU is ok." << std::endl;
//    } else {
//        std::cout << "\nAssemble done in the CPU is not ok." << std::endl;
//    }
//
//    #ifdef __HIPCC__
//    int resgpu = Norm(nodal_forces_global1 - residual);
//    if(resgpu == 0){
//        std::cout << "\nAssemble done in the GPU is ok." << std::endl;
//    } else {
//        std::cout << "\nAssemble done in the GPU is not ok." << std::endl;
//    }
//    #endif
}

TPZFMatrix<REAL> Residual(TPZCompMesh *cmesh, TPZCompMesh *cmesh_noboundary) {
//    bool optimizeBandwidth = true;
//    int n_threads = 4;
//
//    TPZAnalysis an_d(cmesh_noboundary, optimizeBandwidth);
//    TPZSymetricSpStructMatrix strskyl(cmesh_noboundary);
//    strskyl.SetNumThreads(n_threads);
//    an_d.SetStructuralMatrix(strskyl);
//
//    TPZStepSolver<STATE> step;
//    step.SetDirect(ELDLt);
//    an_d.SetSolver(step);
//    an_d.Assemble();
//    an_d.Solve();

    TPZFMatrix<STATE> res(cmesh->NEquations(),1,0.);
//    an_d.Solver().Matrix()->Multiply(cmesh->Solution(), res);
    return res;
}
